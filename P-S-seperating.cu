#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include<iostream>
using namespace std;
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
#include<complex>
#include<hipfft/hipfft.h>
#include "hip/hip_complex.h"
#define PI 3.1415926
#define L 200
#define N 6
#define NT 6000
#define Xn 600
#define Zn 600
#define m 2
#define M 13
#define FM 30.0
#define R 1e-3
#define a_1 30
#define a_2 30
#define shot_num 40

void creatmodel(float* vp, float* vs, float* P)
{
	for (int i = L; i < Xn - L; i++){
		for (int j = L; j < Zn - L; j++){
			P[i * Zn + j] = 1;
		}
	}
}

void xiangbian(float* P, float* Vp, float* Vs)
{
	int i, j;
	
	for (i = L; i < Xn - L; i++)
		for (j = 0; j < L; j++)
		{

			P[i * Zn + j] = P[i * Zn + L];
			Vp[i * Zn + j] = Vp[i * Zn + L];
			Vs[i * Zn + j] = Vs[i * Zn + L];
		}

	
	for (i = L; i < Xn - L; i++)
		for (j = Zn - L; j < Zn; j++)
		{

			P[i * Zn + j] = P[i * Zn + Zn - L - 1];
			Vp[i * Zn + j] = Vp[i * Zn + Zn - L - 1];
			Vs[i * Zn + j] = Vs[i * Zn + Zn - L - 1];
		}

	
	for (i = 0; i < L; i++)
		for (j = L; j < Zn - L; j++)
		{

			P[i * Zn + j] = P[L * Zn + j];
			Vp[i * Zn + j] = Vp[L * Zn + j];
			Vs[i * Zn + j] = Vs[L * Zn + j];
		}

	
	for (i = Xn - L; i < Xn; i++)
		for (j = L; j < Zn - L; j++)
		{

			P[i * Zn + j] = P[(Xn - L - 1) * Zn + j];
			Vp[i * Zn + j] = Vp[(Xn - L - 1) * Zn + j];
			Vs[i * Zn + j] = Vs[(Xn - L - 1) * Zn + j];
		}

	
	for (i = 0; i < L; i++)
		for (j = 0; j < L; j++)
		{

			P[i * Zn + j] = P[L * Zn + L];
			Vp[i * Zn + j] = Vp[L * Zn + L];
			Vs[i * Zn + j] = Vs[L * Zn + L];
		}
	
	for (i = Xn - L; i < Xn; i++)
		for (j = 0; j < L; j++)
		{

			P[i * Zn + j] = P[(Xn - L - 1) * Zn + L];
			Vp[i * Zn + j] = Vp[(Xn - L - 1) * Zn + L];
			Vs[i * Zn + j] = Vs[(Xn - L - 1) * Zn + L];
		}

	
	for (i = 0; i < L; i++)
		for (j = Zn - L; j < Zn; j++)
		{

			P[i * Zn + j] = P[L * Zn + Zn - L - 1];
			Vp[i * Zn + j] = Vp[L * Zn + Zn - L - 1];
			Vs[i * Zn + j] = Vs[L * Zn + Zn - L - 1];
		}
	
	for (i = Xn - L; i < Xn; i++)
		for (j = Zn - L; j < Zn; j++)
		{

			P[i * Zn + j] = P[(Xn - L - 1) * Zn + Zn - L - 1];
			Vp[i * Zn + j] = Vp[(Xn - L - 1) * Zn + Zn - L - 1];
			Vs[i * Zn + j] = Vs[(Xn - L - 1) * Zn + Zn - L - 1];
		}


}
__global__
void revise_xiangbian(float* Vp, float* Vs)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	
	for (i = L; i < Xn - L; i++)
		for (j = 0; j < L; j++)
		{


			Vp[i * Zn + j] = Vp[i * Zn + L];
			Vs[i * Zn + j] = Vs[i * Zn + L];
		}

	
	for (i = L; i < Xn - L; i++)
		for (j = Zn - L; j < Zn; j++)
		{


			Vp[i * Zn + j] = Vp[i * Zn + Zn - L - 1];
			Vs[i * Zn + j] = Vs[i * Zn + Zn - L - 1];
		}

	
	for (i = 0; i < L; i++)
		for (j = L; j < Zn - L; j++)
		{


			Vp[i * Zn + j] = Vp[L * Zn + j];
			Vs[i * Zn + j] = Vs[L * Zn + j];
		}

	
	for (i = Xn - L; i < Xn; i++)
		for (j = L; j < Zn - L; j++)
		{


			Vp[i * Zn + j] = Vp[(Xn - L - 1) * Zn + j];
			Vs[i * Zn + j] = Vs[(Xn - L - 1) * Zn + j];
		}

	
	for (i = 0; i < L; i++)
		for (j = 0; j < L; j++)
		{


			Vp[i * Zn + j] = Vp[L * Zn + L];
			Vs[i * Zn + j] = Vs[L * Zn + L];
		}
	
	for (i = Xn - L; i < Xn; i++)
		for (j = 0; j < L; j++)
		{


			Vp[i * Zn + j] = Vp[(Xn - L - 1) * Zn + L];
			Vs[i * Zn + j] = Vs[(Xn - L - 1) * Zn + L];
		}

	
	for (i = 0; i < L; i++)
		for (j = Zn - L; j < Zn; j++)
		{


			Vp[i * Zn + j] = Vp[L * Zn + Zn - L - 1];
			Vs[i * Zn + j] = Vs[L * Zn + Zn - L - 1];
		}
	
	for (i = Xn - L; i < Xn; i++)
		for (j = Zn - L; j < Zn; j++)
		{


			Vp[i * Zn + j] = Vp[(Xn - L - 1) * Zn + Zn - L - 1];
			Vs[i * Zn + j] = Vs[(Xn - L - 1) * Zn + Zn - L - 1];
		}
}

__global__ void forward_u(float* Vx, float* Vz, float* Vpx, float* Vpz, float* Vsx, float* Vsz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* az, float* ax, float dt, float dx, float dz, float Z_receive, float* record_vx, float* record_vz, int t, float* Vpx_now, float* Vpz_now, float* Vsx_now, float* Vsz_now,
	float* Vpx_past, float* Vpz_past, float* Vsx_past, float* Vsz_past, float* O_duz_xdz, float* O_duz_zdx, float* O_dux_zdx, float* O_dux_xdz, float* O_thetax, float* O_thetaz, float* O_omegaz, float* O_omegax) {


	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	float a[N] = { 1.2213364, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;

		O_duz_xdz[i * Zn + j] = O_duz_xdz[i * Zn + j] + (az[i * Zn + j] * duz_xdz - az[i * Zn + j] * O_duz_xdz[i * Zn + j]) * dt;
		O_duz_zdx[i * Zn + j] = O_duz_zdx[i * Zn + j] + (ax[i * Zn + j] * duz_zdx - ax[i * Zn + j] * O_duz_zdx[i * Zn + j]) * dt;
		O_dux_zdx[i * Zn + j] = O_dux_zdx[i * Zn + j] + (ax[i * Zn + j] * dux_zdx - ax[i * Zn + j] * O_dux_zdx[i * Zn + j]) * dt;
		O_dux_xdz[i * Zn + j] = O_dux_xdz[i * Zn + j] + (az[i * Zn + j] * dux_xdz - az[i * Zn + j] * O_dux_xdz[i * Zn + j]) * dt;

		O_thetax[i * Zn + j] = O_thetax[i * Zn + j] + (ax[i * Zn + j] * dthetadx - ax[i * Zn + j] * O_thetax[i * Zn + j]) * dt;
		O_thetaz[i * Zn + j] = O_thetaz[i * Zn + j] + (az[i * Zn + j] * dthetadz - az[i * Zn + j] * O_thetaz[i * Zn + j]) * dt;
		O_omegaz[i * Zn + j] = O_omegaz[i * Zn + j] + (az[i * Zn + j] * domegadz - az[i * Zn + j] * O_omegaz[i * Zn + j]) * dt;
		O_omegax[i * Zn + j] = O_omegax[i * Zn + j] + (ax[i * Zn + j] * domegadx - ax[i * Zn + j] * O_omegax[i * Zn + j]) * dt;

		Vpx[i * Zn + j] = 2 * Vpx_now[i * Zn + j] - Vpx_past[i * Zn + j] + (dt * dt) * (dthetadx - O_thetax[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		Vpz[i * Zn + j] = 2 * Vpz_now[i * Zn + j] - Vpz_past[i * Zn + j] + (dt * dt) * (dthetadz - O_thetaz[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);
		Vsx[i * Zn + j] = 2 * Vsx_now[i * Zn + j] - Vsx_past[i * Zn + j] + (dt * dt) * (domegadz - O_omegaz[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		Vsz[i * Zn + j] = 2 * Vsz_now[i * Zn + j] - Vsz_past[i * Zn + j] + (dt * dt) * (-domegadx + O_omegax[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);

		Vpx_past[i * Zn + j] = Vpx_now[i * Zn + j];	Vpx_now[i * Zn + j] = Vpx[i * Zn + j];
		Vpz_past[i * Zn + j] = Vpz_now[i * Zn + j];	Vpz_now[i * Zn + j] = Vpz[i * Zn + j];
		Vsx_past[i * Zn + j] = Vsx_now[i * Zn + j];	Vsx_now[i * Zn + j] = Vsx[i * Zn + j];
		Vsz_past[i * Zn + j] = Vsz_now[i * Zn + j];	Vsz_now[i * Zn + j] = Vsz[i * Zn + j];


		Vx[i * Zn + j] = Vpx[i * Zn + j] + Vsx[i * Zn + j];
		Vz[i * Zn + j] = Vpz[i * Zn + j] + Vsz[i * Zn + j];

	}

	if (j = Z_receive)
	{
		record_vx[i * NT + t] = Vx[i * Zn + j];
		record_vz[i * NT + t] = Vz[i * Zn + j];
	}


}

__global__ void forward_s(float* Vx, float* Vz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* ax, float* az, float dt, float dx, float dz, int t, int Sx, int Sz, float* Vp, float* Vs, float* source,
	float* F_xx, float* F_zz, float* F_xz, float* F_zx) {
	
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dvxdx = 0.0f;
	float dvzdz = 0.0f;
	float dvxdz = 0.0f;
	float dvzdx = 0.0f;
	float a[N] = { 1.2213364, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };
	float s;
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		if (i == Sx && j == Sz)
		{
			s = source[t];
		}
		else
		{
			s = 0.0;
		}

		dvxdx = (a[0] * (Vx[(i + 0) * Zn + j] - Vx[(i - 1) * Zn + j])
			+ a[1] * (Vx[(i + 1) * Zn + j] - Vx[(i - 2) * Zn + j])
			+ a[2] * (Vx[(i + 2) * Zn + j] - Vx[(i - 3) * Zn + j])
			+ a[3] * (Vx[(i + 3) * Zn + j] - Vx[(i - 4) * Zn + j])
			+ a[4] * (Vx[(i + 4) * Zn + j] - Vx[(i - 5) * Zn + j])
			+ a[5] * (Vx[(i + 5) * Zn + j] - Vx[(i - 6) * Zn + j])) / dx;

		dvxdz = (a[0] * (Vx[(i)*Zn + j + 1] - Vx[(i)*Zn + j - 0])
			+ a[1] * (Vx[(i)*Zn + j + 2] - Vx[(i)*Zn + j - 1])
			+ a[2] * (Vx[(i)*Zn + j + 3] - Vx[(i)*Zn + j - 2])
			+ a[3] * (Vx[(i)*Zn + j + 4] - Vx[(i)*Zn + j - 3])
			+ a[4] * (Vx[(i)*Zn + j + 5] - Vx[(i)*Zn + j - 4])
			+ a[5] * (Vx[(i)*Zn + j + 6] - Vx[(i)*Zn + j - 5])) / dz;

		dvzdz = (a[0] * (Vz[(i)*Zn + j + 0] - Vz[(i)*Zn + j - 1])
			+ a[1] * (Vz[(i)*Zn + j + 1] - Vz[(i)*Zn + j - 2])
			+ a[2] * (Vz[(i)*Zn + j + 2] - Vz[(i)*Zn + j - 3])
			+ a[3] * (Vz[(i)*Zn + j + 3] - Vz[(i)*Zn + j - 4])
			+ a[4] * (Vz[(i)*Zn + j + 4] - Vz[(i)*Zn + j - 5])
			+ a[5] * (Vz[(i)*Zn + j + 5] - Vz[(i)*Zn + j - 6])) / dz;

		dvzdx = (a[0] * (Vz[(i + 1) * Zn + j] - Vz[(i - 0) * Zn + j])
			+ a[1] * (Vz[(i + 2) * Zn + j] - Vz[(i - 1) * Zn + j])
			+ a[2] * (Vz[(i + 3) * Zn + j] - Vz[(i - 2) * Zn + j])
			+ a[3] * (Vz[(i + 4) * Zn + j] - Vz[(i - 3) * Zn + j])
			+ a[4] * (Vz[(i + 5) * Zn + j] - Vz[(i - 4) * Zn + j])
			+ a[5] * (Vz[(i + 6) * Zn + j] - Vz[(i - 5) * Zn + j])) / dx;

		F_xx[i * Zn + j] = F_xx[i * Zn + j] + (ax[i * Zn + j] * dvxdx - ax[i * Zn + j] * F_xx[i * Zn + j]) * dt;
		F_zz[i * Zn + j] = F_zz[i * Zn + j] + (az[i * Zn + j] * dvzdz - az[i * Zn + j] * F_zz[i * Zn + j]) * dt;
		F_xz[i * Zn + j] = F_xz[i * Zn + j] + (az[i * Zn + j] * dvxdz - az[i * Zn + j] * F_xz[i * Zn + j]) * dt;
		F_zx[i * Zn + j] = F_zx[i * Zn + j] + (ax[i * Zn + j] * dvzdx - ax[i * Zn + j] * F_zx[i * Zn + j]) * dt;

		theta[i * Zn + j] = Vp[i * Zn + j] * Vp[i * Zn + j] * (dvxdx + dvzdz - F_xx[i * Zn + j] - F_zz[i * Zn + j]);
		omega[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdz - dvzdx - F_xz[i * Zn + j] + F_zx[i * Zn + j]);
		duzdx[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvzdx - F_zx[i * Zn + j]);
		duzdz[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvzdz - F_zz[i * Zn + j]);
		duxdz[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdz - F_xz[i * Zn + j]);
		duxdx[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdx - F_xx[i * Zn + j]);

		theta[i * Zn + j] = theta[i * Zn + j] + s;		

	}
	
}





__global__
void velocity( float* Vx, float* Vz, float* Vpx, float* Vpz, float* Vsx, float* Vsz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* az, float* ax, float dt, float dx, float dz, float Z_receive, float* record_vx, float* record_vz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	float a[N] = { +1.2213e+0,-9.6931e-2,1.7448e-2,-2.9673e-3,+3.5901e-4,-2.1848e-5 };
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		dthetadx= (a[0] * (theta[(i+1)*Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;




		Vpx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * ((1 - 0.5 * dt * ax[i * Zn + j]) * Vpx[i * Zn + j] + dt * (dthetadx + duz_xdz - duz_zdx));

		Vpz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * ((1 - 0.5 * dt * az[i * Zn + j]) * Vpz[i * Zn + j] + dt * (dthetadz + dux_zdx - dux_xdz));

		Vsx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * ((1 - 0.5 * dt * ax[i * Zn + j]) * Vsx[i * Zn + j] + dt * ( domegadz + duz_xdz - duz_zdx));

		Vsz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * ((1 - 0.5 * dt * az[i * Zn + j]) * Vsz[i * Zn + j] + dt * (-domegadx + dux_zdx - dux_xdz));


		Vx[i * Zn + j] = Vpx[i * Zn + j] + Vsx[i * Zn + j];
		Vz[i * Zn + j] = Vpz[i * Zn + j] + Vsz[i * Zn + j];

	}
	if (j = Z_receive)
	{
		record_vx[i * NT + t] = Vx[i * Zn + j];
		record_vz[i * NT + t] = Vz[i * Zn + j];
	}
}
__global__
void stress(float* P, float* Vx, float* Vz, float* theta, float* omega, float* theta_x, float* omega_x, float* theta_z, float* omega_z, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* ax,float* az, float dt, float dx, float dz, int t, int Sx, int Sz, float* Vp, float* Vs, float* source)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dvxdx = 0.0f;
	float dvzdz = 0.0f;
	float dvxdz = 0.0f;
	float dvzdx = 0.0f;
	float a[N] = { +1.2213e+0,-9.6931e-2,1.7448e-2,-2.9673e-3,+3.5901e-4,-2.1848e-5 };
	float s;
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		if (i == Sx && j == Sz)
		{
			s = source[t];
		}
		else
		{
			s = 0.0;
		}

		dvxdx = (a[0] * (Vx[(i + 0) * Zn + j] - Vx[(i - 1) * Zn + j])
			+ a[1] * (Vx[(i + 1) * Zn + j] - Vx[(i - 2) * Zn + j])
			+ a[2] * (Vx[(i + 2) * Zn + j] - Vx[(i - 3) * Zn + j])
			+ a[3] * (Vx[(i + 3) * Zn + j] - Vx[(i - 4) * Zn + j])
			+ a[4] * (Vx[(i + 4) * Zn + j] - Vx[(i - 5) * Zn + j])
			+ a[5] * (Vx[(i + 5) * Zn + j] - Vx[(i - 6) * Zn + j])) / dx;

		dvxdz = (a[0] * (Vx[(i)*Zn + j + 1] - Vx[(i)*Zn + j - 0])
			+ a[1] * (Vx[(i)*Zn + j + 2] - Vx[(i)*Zn + j - 1])
			+ a[2] * (Vx[(i)*Zn + j + 3] - Vx[(i)*Zn + j - 2])
			+ a[3] * (Vx[(i)*Zn + j + 4] - Vx[(i)*Zn + j - 3])
			+ a[4] * (Vx[(i)*Zn + j + 5] - Vx[(i)*Zn + j - 4])
			+ a[5] * (Vx[(i)*Zn + j + 6] - Vx[(i)*Zn + j - 5])) / dz;

		dvzdz= (a[0] * (Vz[(i)*Zn + j + 0] - Vz[(i)*Zn + j - 1])
			+ a[1] * (Vz[(i)*Zn + j + 1] - Vz[(i)*Zn + j - 2])
			+ a[2] * (Vz[(i)*Zn + j + 2] - Vz[(i)*Zn + j - 3])
			+ a[3] * (Vz[(i)*Zn + j + 3] - Vz[(i)*Zn + j - 4])
			+ a[4] * (Vz[(i)*Zn + j + 4] - Vz[(i)*Zn + j - 5])
			+ a[5] * (Vz[(i)*Zn + j + 5] - Vz[(i)*Zn + j - 6])) / dz;

		dvzdx = (a[0] * (Vz[(i + 1) * Zn + j] - Vz[(i - 0) * Zn + j])
			+ a[1] * (Vz[(i + 2) * Zn + j] - Vz[(i - 1) * Zn + j])
			+ a[2] * (Vz[(i + 3) * Zn + j] - Vz[(i - 2) * Zn + j])
			+ a[3] * (Vz[(i + 4) * Zn + j] - Vz[(i - 3) * Zn + j])
			+ a[4] * (Vz[(i + 5) * Zn + j] - Vz[(i - 4) * Zn + j])
			+ a[5] * (Vz[(i + 6) * Zn + j] - Vz[(i - 5) * Zn + j])) / dx;

		theta_x[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (dt * powf(Vp[i * Zn + j], 2) * dvxdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * theta_x[i * Zn + j]);

		theta_z[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vp[i * Zn + j], 2) * dvzdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * theta_z[i * Zn + j]);
		////
		omega_x[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (-dt * powf(Vs[i * Zn + j], 2) * dvzdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * omega_x[i * Zn + j]);

		omega_z[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvxdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * omega_z[i * Zn + j]);
		////
		duxdx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvxdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * duxdx[i * Zn + j]);

		duxdz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvxdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * duxdz[i * Zn + j]);

		duzdx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvzdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * duzdx[i * Zn + j]);

		duzdz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvzdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * duzdz[i * Zn + j]);

		theta[i * Zn + j] = theta_x[i * Zn + j] + theta_z[i * Zn + j] + s;
		omega[i * Zn + j] = omega_x[i * Zn + j] + omega_z[i * Zn + j];

	}

}

__global__ void rt_u_res(float* Vx, float* Vz, float* Vpx, float* Vpz, float* Vsx, float* Vsz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* az, float* ax, float dt, float dx, float dz, int t, float* Vpx_now, float* Vpz_now, float* Vsx_now, float* Vsz_now,
	float* Vpx_past, float* Vpz_past, float* Vsx_past, float* Vsz_past, float* O_duz_xdz, float* O_duz_zdx, float* O_dux_zdx, float* O_dux_xdz, float* O_thetax, float* O_thetaz, float* O_omegaz, float* O_omegax) {


	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	float a[N] = { 1.2213364, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;

		O_duz_xdz[i * Zn + j] = O_duz_xdz[i * Zn + j] + (az[i * Zn + j] * duz_xdz - az[i * Zn + j] * O_duz_xdz[i * Zn + j]) * dt;
		O_duz_zdx[i * Zn + j] = O_duz_zdx[i * Zn + j] + (ax[i * Zn + j] * duz_zdx - ax[i * Zn + j] * O_duz_zdx[i * Zn + j]) * dt;
		O_dux_zdx[i * Zn + j] = O_dux_zdx[i * Zn + j] + (ax[i * Zn + j] * dux_zdx - ax[i * Zn + j] * O_dux_zdx[i * Zn + j]) * dt;
		O_dux_xdz[i * Zn + j] = O_dux_xdz[i * Zn + j] + (az[i * Zn + j] * dux_xdz - az[i * Zn + j] * O_dux_xdz[i * Zn + j]) * dt;

		O_thetax[i * Zn + j] = O_thetax[i * Zn + j] + (ax[i * Zn + j] * dthetadx - ax[i * Zn + j] * O_thetax[i * Zn + j]) * dt;
		O_thetaz[i * Zn + j] = O_thetaz[i * Zn + j] + (az[i * Zn + j] * dthetadz - az[i * Zn + j] * O_thetaz[i * Zn + j]) * dt;
		O_omegaz[i * Zn + j] = O_omegaz[i * Zn + j] + (az[i * Zn + j] * domegadz - az[i * Zn + j] * O_omegaz[i * Zn + j]) * dt;
		O_omegax[i * Zn + j] = O_omegax[i * Zn + j] + (ax[i * Zn + j] * domegadx - ax[i * Zn + j] * O_omegax[i * Zn + j]) * dt;

		Vpx[i * Zn + j] = 2 * Vpx_now[i * Zn + j] - Vpx_past[i * Zn + j] + (dt * dt) * (dthetadx - O_thetax[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		Vpz[i * Zn + j] = 2 * Vpz_now[i * Zn + j] - Vpz_past[i * Zn + j] + (dt * dt) * (dthetadz - O_thetaz[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);
		Vsx[i * Zn + j] = 2 * Vsx_now[i * Zn + j] - Vsx_past[i * Zn + j] + (dt * dt) * (domegadz - O_omegaz[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		Vsz[i * Zn + j] = 2 * Vsz_now[i * Zn + j] - Vsz_past[i * Zn + j] + (dt * dt) * (-domegadx + O_omegax[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);

		Vpx_past[i * Zn + j] = Vpx_now[i * Zn + j];	Vpx_now[i * Zn + j] = Vpx[i * Zn + j];
		Vpz_past[i * Zn + j] = Vpz_now[i * Zn + j];	Vpz_now[i * Zn + j] = Vpz[i * Zn + j];
		Vsx_past[i * Zn + j] = Vsx_now[i * Zn + j];	Vsx_now[i * Zn + j] = Vsx[i * Zn + j];
		Vsz_past[i * Zn + j] = Vsz_now[i * Zn + j];	Vsz_now[i * Zn + j] = Vsz[i * Zn + j];


		Vx[i * Zn + j] = Vpx[i * Zn + j] + Vsx[i * Zn + j];
		Vz[i * Zn + j] = Vpz[i * Zn + j] + Vsz[i * Zn + j];

	}


}

__global__ void rt_s_res(float* Vx, float* Vz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* ax, float* az, float dt, float dx, float dz, float* Vp, float* Vs, 
	float* F_xx, float* F_zz, float* F_xz, float* F_zx) {

	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dvxdx = 0.0f;
	float dvzdz = 0.0f;
	float dvxdz = 0.0f;
	float dvzdx = 0.0f;
	float a[N] = { 1.2213364, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };
	float s;
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		
		dvxdx = (a[0] * (Vx[(i + 0) * Zn + j] - Vx[(i - 1) * Zn + j])
			+ a[1] * (Vx[(i + 1) * Zn + j] - Vx[(i - 2) * Zn + j])
			+ a[2] * (Vx[(i + 2) * Zn + j] - Vx[(i - 3) * Zn + j])
			+ a[3] * (Vx[(i + 3) * Zn + j] - Vx[(i - 4) * Zn + j])
			+ a[4] * (Vx[(i + 4) * Zn + j] - Vx[(i - 5) * Zn + j])
			+ a[5] * (Vx[(i + 5) * Zn + j] - Vx[(i - 6) * Zn + j])) / dx;

		dvxdz = (a[0] * (Vx[(i)*Zn + j + 1] - Vx[(i)*Zn + j - 0])
			+ a[1] * (Vx[(i)*Zn + j + 2] - Vx[(i)*Zn + j - 1])
			+ a[2] * (Vx[(i)*Zn + j + 3] - Vx[(i)*Zn + j - 2])
			+ a[3] * (Vx[(i)*Zn + j + 4] - Vx[(i)*Zn + j - 3])
			+ a[4] * (Vx[(i)*Zn + j + 5] - Vx[(i)*Zn + j - 4])
			+ a[5] * (Vx[(i)*Zn + j + 6] - Vx[(i)*Zn + j - 5])) / dz;

		dvzdz = (a[0] * (Vz[(i)*Zn + j + 0] - Vz[(i)*Zn + j - 1])
			+ a[1] * (Vz[(i)*Zn + j + 1] - Vz[(i)*Zn + j - 2])
			+ a[2] * (Vz[(i)*Zn + j + 2] - Vz[(i)*Zn + j - 3])
			+ a[3] * (Vz[(i)*Zn + j + 3] - Vz[(i)*Zn + j - 4])
			+ a[4] * (Vz[(i)*Zn + j + 4] - Vz[(i)*Zn + j - 5])
			+ a[5] * (Vz[(i)*Zn + j + 5] - Vz[(i)*Zn + j - 6])) / dz;

		dvzdx = (a[0] * (Vz[(i + 1) * Zn + j] - Vz[(i - 0) * Zn + j])
			+ a[1] * (Vz[(i + 2) * Zn + j] - Vz[(i - 1) * Zn + j])
			+ a[2] * (Vz[(i + 3) * Zn + j] - Vz[(i - 2) * Zn + j])
			+ a[3] * (Vz[(i + 4) * Zn + j] - Vz[(i - 3) * Zn + j])
			+ a[4] * (Vz[(i + 5) * Zn + j] - Vz[(i - 4) * Zn + j])
			+ a[5] * (Vz[(i + 6) * Zn + j] - Vz[(i - 5) * Zn + j])) / dx;

		F_xx[i * Zn + j] = F_xx[i * Zn + j] + (ax[i * Zn + j] * dvxdx - ax[i * Zn + j] * F_xx[i * Zn + j]) * dt;
		F_zz[i * Zn + j] = F_zz[i * Zn + j] + (az[i * Zn + j] * dvzdz - az[i * Zn + j] * F_zz[i * Zn + j]) * dt;
		F_xz[i * Zn + j] = F_xz[i * Zn + j] + (az[i * Zn + j] * dvxdz - az[i * Zn + j] * F_xz[i * Zn + j]) * dt;
		F_zx[i * Zn + j] = F_zx[i * Zn + j] + (ax[i * Zn + j] * dvzdx - ax[i * Zn + j] * F_zx[i * Zn + j]) * dt;

		theta[i * Zn + j] = Vp[i * Zn + j] * Vp[i * Zn + j] * (dvxdx + dvzdz - F_xx[i * Zn + j] - F_zz[i * Zn + j]);
		omega[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdz - dvzdx - F_xz[i * Zn + j] + F_zx[i * Zn + j]);
		duzdx[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvzdx - F_zx[i * Zn + j]);
		duzdz[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvzdz - F_zz[i * Zn + j]);
		duxdz[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdz - F_xz[i * Zn + j]);
		duxdx[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdx - F_xx[i * Zn + j]);

		
	}

}

__global__
void velocity_backward_propagation(float* Vx, float* Vz, float* Vpx, float* Vpz, float* Vsx, float* Vsz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* az, float* ax, float dt, float dx, float dz)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	float a[N] = { +1.2213e+0,-9.6931e-2,1.7448e-2,-2.9673e-3,+3.5901e-4,-2.1848e-5 };
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;




		Vpx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * ((1 - 0.5 * dt * ax[i * Zn + j]) * Vpx[i * Zn + j] + dt * (dthetadx + duz_xdz - duz_zdx));

		Vpz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * ((1 - 0.5 * dt * az[i * Zn + j]) * Vpz[i * Zn + j] + dt * (dthetadz + dux_zdx - dux_xdz));

		Vsx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * ((1 - 0.5 * dt * ax[i * Zn + j]) * Vsx[i * Zn + j] + dt * (domegadz + duz_xdz - duz_zdx));

		Vsz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * ((1 - 0.5 * dt * az[i * Zn + j]) * Vsz[i * Zn + j] + dt * (-domegadx + dux_zdx - dux_xdz));


		Vx[i * Zn + j] = Vpx[i * Zn + j] + Vsx[i * Zn + j];
		Vz[i * Zn + j] = Vpz[i * Zn + j] + Vsz[i * Zn + j];

	}
	
}
__global__
void stress_backward_propagation(float* P, float* Vx, float* Vz, float* theta, float* omega, float* theta_x, float* omega_x, float* theta_z, float* omega_z, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* ax, float* az, float dt, float dx, float dz, int t, float* Vp, float* Vs)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dvxdx = 0.0f;
	float dvzdz = 0.0f;
	float dvxdz = 0.0f;
	float dvzdx = 0.0f;
	float a[N] = { +1.2213e+0,-9.6931e-2,1.7448e-2,-2.9673e-3,+3.5901e-4,-2.1848e-5 };
	float s;
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		

		dvxdx = (a[0] * (Vx[(i + 0) * Zn + j] - Vx[(i - 1) * Zn + j])
			+ a[1] * (Vx[(i + 1) * Zn + j] - Vx[(i - 2) * Zn + j])
			+ a[2] * (Vx[(i + 2) * Zn + j] - Vx[(i - 3) * Zn + j])
			+ a[3] * (Vx[(i + 3) * Zn + j] - Vx[(i - 4) * Zn + j])
			+ a[4] * (Vx[(i + 4) * Zn + j] - Vx[(i - 5) * Zn + j])
			+ a[5] * (Vx[(i + 5) * Zn + j] - Vx[(i - 6) * Zn + j])) / dx;

		dvxdz = (a[0] * (Vx[(i)*Zn + j + 1] - Vx[(i)*Zn + j - 0])
			+ a[1] * (Vx[(i)*Zn + j + 2] - Vx[(i)*Zn + j - 1])
			+ a[2] * (Vx[(i)*Zn + j + 3] - Vx[(i)*Zn + j - 2])
			+ a[3] * (Vx[(i)*Zn + j + 4] - Vx[(i)*Zn + j - 3])
			+ a[4] * (Vx[(i)*Zn + j + 5] - Vx[(i)*Zn + j - 4])
			+ a[5] * (Vx[(i)*Zn + j + 6] - Vx[(i)*Zn + j - 5])) / dz;

		dvzdz = (a[0] * (Vz[(i)*Zn + j + 0] - Vz[(i)*Zn + j - 1])
			+ a[1] * (Vz[(i)*Zn + j + 1] - Vz[(i)*Zn + j - 2])
			+ a[2] * (Vz[(i)*Zn + j + 2] - Vz[(i)*Zn + j - 3])
			+ a[3] * (Vz[(i)*Zn + j + 3] - Vz[(i)*Zn + j - 4])
			+ a[4] * (Vz[(i)*Zn + j + 4] - Vz[(i)*Zn + j - 5])
			+ a[5] * (Vz[(i)*Zn + j + 5] - Vz[(i)*Zn + j - 6])) / dz;

		dvzdx = (a[0] * (Vz[(i + 1) * Zn + j] - Vz[(i - 0) * Zn + j])
			+ a[1] * (Vz[(i + 2) * Zn + j] - Vz[(i - 1) * Zn + j])
			+ a[2] * (Vz[(i + 3) * Zn + j] - Vz[(i - 2) * Zn + j])
			+ a[3] * (Vz[(i + 4) * Zn + j] - Vz[(i - 3) * Zn + j])
			+ a[4] * (Vz[(i + 5) * Zn + j] - Vz[(i - 4) * Zn + j])
			+ a[5] * (Vz[(i + 6) * Zn + j] - Vz[(i - 5) * Zn + j])) / dx;

		theta_x[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (dt * powf(Vp[i * Zn + j], 2) * dvxdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * theta_x[i * Zn + j]);

		theta_z[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vp[i * Zn + j], 2) * dvzdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * theta_z[i * Zn + j]);
		////
		omega_x[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (-dt * powf(Vs[i * Zn + j], 2) * dvzdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * omega_x[i * Zn + j]);

		omega_z[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvxdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * omega_z[i * Zn + j]);
		////
		duxdx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvxdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * duxdx[i * Zn + j]);

		duxdz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvxdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * duxdz[i * Zn + j]);

		duzdx[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * ax[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvzdx + (1.0 - 0.5 * dt * ax[i * Zn + j]) * duzdx[i * Zn + j]);

		duzdz[i * Zn + j] = (1.0 / (1.0 + 0.5 * dt * az[i * Zn + j])) * (dt * powf(Vs[i * Zn + j], 2) * dvzdz + (1.0 - 0.5 * dt * az[i * Zn + j]) * duzdz[i * Zn + j]);

		theta[i * Zn + j] = theta_x[i * Zn + j] + theta_z[i * Zn + j];
		omega[i * Zn + j] = omega_x[i * Zn + j] + omega_z[i * Zn + j];

	}

}


__global__ void reshot_u(float* Vx, float* Vz, float* Vpx, float* Vpz, float* Vsx, float* Vsz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float dt, float dx, float dz, float* Vpx_now, float* Vpz_now, float* Vsx_now, float* Vsz_now,
	float* Vpx_past, float* Vpz_past, float* Vsx_past, float* Vsz_past) {
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	float a[N] = { 1.2213364, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;

		 Vpx[i * Zn + j] = Vpx_now[i * Zn + j];Vpx_now[i * Zn + j] = Vpx_past[i * Zn + j];
		 Vpz[i * Zn + j] = Vpz_now[i * Zn + j];Vpz_now[i * Zn + j] = Vpz_past[i * Zn + j];
		 Vsx[i * Zn + j] = Vsx_now[i * Zn + j];Vsx_now[i * Zn + j] = Vsx_past[i * Zn + j];
		 Vsz[i * Zn + j] = Vsz_now[i * Zn + j];Vsz_now[i * Zn + j] = Vsz_past[i * Zn + j];

		

		Vpx_past[i * Zn + j] = 2 * Vpx_now[i * Zn + j] - Vpx[i * Zn + j] + (dt * dt) * (dthetadx + duz_xdz - duz_zdx);
		Vpz_past[i * Zn + j] = 2 * Vpz_now[i * Zn + j] - Vpz[i * Zn + j] + (dt * dt) * (dthetadz + dux_zdx - dux_xdz);
		Vsx_past[i * Zn + j] = 2 * Vsx_now[i * Zn + j] - Vsx[i * Zn + j] + (dt * dt) * (domegadz + duz_xdz - duz_zdx);
		Vsz_past[i * Zn + j] = 2 * Vsz_now[i * Zn + j] - Vsz[i * Zn + j] + (dt * dt) * (-domegadx + dux_zdx - dux_xdz);



		Vx[i * Zn + j] = Vpx_past[i * Zn + j] + Vsx_past[i * Zn + j];
		Vz[i * Zn + j] = Vpz_past[i * Zn + j] + Vsz_past[i * Zn + j];

	}

}

__global__ void reshot_s(float* Vx, float* Vz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float dt, float dx, float dz, float* Vp, float* Vs) {

	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dvxdx = 0.0f;
	float dvzdz = 0.0f;
	float dvxdz = 0.0f;
	float dvzdx = 0.0f;
	float a[N] = { 1.2213364, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };
	float s;
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		

		dvxdx = (a[0] * (Vx[(i + 0) * Zn + j] - Vx[(i - 1) * Zn + j])
			+ a[1] * (Vx[(i + 1) * Zn + j] - Vx[(i - 2) * Zn + j])
			+ a[2] * (Vx[(i + 2) * Zn + j] - Vx[(i - 3) * Zn + j])
			+ a[3] * (Vx[(i + 3) * Zn + j] - Vx[(i - 4) * Zn + j])
			+ a[4] * (Vx[(i + 4) * Zn + j] - Vx[(i - 5) * Zn + j])
			+ a[5] * (Vx[(i + 5) * Zn + j] - Vx[(i - 6) * Zn + j])) / dx;

		dvxdz = (a[0] * (Vx[(i)*Zn + j + 1] - Vx[(i)*Zn + j - 0])
			+ a[1] * (Vx[(i)*Zn + j + 2] - Vx[(i)*Zn + j - 1])
			+ a[2] * (Vx[(i)*Zn + j + 3] - Vx[(i)*Zn + j - 2])
			+ a[3] * (Vx[(i)*Zn + j + 4] - Vx[(i)*Zn + j - 3])
			+ a[4] * (Vx[(i)*Zn + j + 5] - Vx[(i)*Zn + j - 4])
			+ a[5] * (Vx[(i)*Zn + j + 6] - Vx[(i)*Zn + j - 5])) / dz;

		dvzdz = (a[0] * (Vz[(i)*Zn + j + 0] - Vz[(i)*Zn + j - 1])
			+ a[1] * (Vz[(i)*Zn + j + 1] - Vz[(i)*Zn + j - 2])
			+ a[2] * (Vz[(i)*Zn + j + 2] - Vz[(i)*Zn + j - 3])
			+ a[3] * (Vz[(i)*Zn + j + 3] - Vz[(i)*Zn + j - 4])
			+ a[4] * (Vz[(i)*Zn + j + 4] - Vz[(i)*Zn + j - 5])
			+ a[5] * (Vz[(i)*Zn + j + 5] - Vz[(i)*Zn + j - 6])) / dz;

		dvzdx = (a[0] * (Vz[(i + 1) * Zn + j] - Vz[(i - 0) * Zn + j])
			+ a[1] * (Vz[(i + 2) * Zn + j] - Vz[(i - 1) * Zn + j])
			+ a[2] * (Vz[(i + 3) * Zn + j] - Vz[(i - 2) * Zn + j])
			+ a[3] * (Vz[(i + 4) * Zn + j] - Vz[(i - 3) * Zn + j])
			+ a[4] * (Vz[(i + 5) * Zn + j] - Vz[(i - 4) * Zn + j])
			+ a[5] * (Vz[(i + 6) * Zn + j] - Vz[(i - 5) * Zn + j])) / dx;

		

		theta[i * Zn + j] = Vp[i * Zn + j] * Vp[i * Zn + j] * (dvxdx + dvzdz);
		omega[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdz - dvzdx);
		duzdx[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvzdx);
		duzdz[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvzdz);
		duxdz[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdz);
		duxdx[i * Zn + j] = Vs[i * Zn + j] * Vs[i * Zn + j] * (dvxdx);

		

	}

}
__global__ void PLACE_V( float* Vpx, float* Vpz, float* Vsx, float* Vsz, 
	float* Vpx_now, float* Vpz_now, float* Vsx_now, float* Vsz_now,
	float* Vpx_past, float* Vpz_past, float* Vsx_past, float* Vsz_past) {


	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
	
		Vpx_past[i * Zn + j] = Vpx_now[i * Zn + j];
		Vpz_past[i * Zn + j] = Vpz_now[i * Zn + j];
		Vsx_past[i * Zn + j] = Vsx_now[i * Zn + j];
		Vsz_past[i * Zn + j] = Vsz_now[i * Zn + j];

		Vpx_now[i * Zn + j] = Vpx[i * Zn + j];
		Vpz_now[i * Zn + j] = Vpz[i * Zn + j];
		Vsx_now[i * Zn + j] = Vsx[i * Zn + j];
		Vsz_now[i * Zn + j] = Vsz[i * Zn + j];

		

	}



}

__global__
void velocity_wavefield_reconstruction(float* Vx, float* Vz, float* Vpx, float* Vpz, float* Vsx, float* Vsz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	 float dt, float dx, float dz)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	float a[N] = { +1.2213e+0,-9.6931e-2,1.7448e-2,-2.9673e-3,+3.5901e-4,-2.1848e-5 };
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;




		Vpx[i * Zn + j] = Vpx[i * Zn + j] - dt * (dthetadx + duz_xdz - duz_zdx);

		Vpz[i * Zn + j] = Vpz[i * Zn + j] - dt * (dthetadz + dux_zdx - dux_xdz);

		Vsx[i * Zn + j] = Vsx[i * Zn + j] - dt * (domegadz + duz_xdz - duz_zdx);

		Vsz[i * Zn + j] = Vsz[i * Zn + j] - dt * (-domegadx + dux_zdx - dux_xdz);


		Vx[i * Zn + j] = Vpx[i * Zn + j] + Vsx[i * Zn + j];
		Vz[i * Zn + j] = Vpz[i * Zn + j] + Vsz[i * Zn + j];

	}
	
}
__global__
void stress_wavefield_reconstruction(float* Vx, float* Vz, float* theta, float* omega, float* theta_x, float* omega_x, float* theta_z, float* omega_z, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	 float dt, float dx, float dz, float* Vp, float* Vs)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dvxdx = 0.0f;
	float dvzdz = 0.0f;
	float dvxdz = 0.0f;
	float dvzdx = 0.0f;
	float a[N] = { +1.2213e+0,-9.6931e-2,1.7448e-2,-2.9673e-3,+3.5901e-4,-2.1848e-5 };
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{

		dvxdx = (a[0] * (Vx[(i + 0) * Zn + j] - Vx[(i - 1) * Zn + j])
			+ a[1] * (Vx[(i + 1) * Zn + j] - Vx[(i - 2) * Zn + j])
			+ a[2] * (Vx[(i + 2) * Zn + j] - Vx[(i - 3) * Zn + j])
			+ a[3] * (Vx[(i + 3) * Zn + j] - Vx[(i - 4) * Zn + j])
			+ a[4] * (Vx[(i + 4) * Zn + j] - Vx[(i - 5) * Zn + j])
			+ a[5] * (Vx[(i + 5) * Zn + j] - Vx[(i - 6) * Zn + j])) / dx;

		dvxdz = (a[0] * (Vx[(i)*Zn + j + 1] - Vx[(i)*Zn + j - 0])
			+ a[1] * (Vx[(i)*Zn + j + 2] - Vx[(i)*Zn + j - 1])
			+ a[2] * (Vx[(i)*Zn + j + 3] - Vx[(i)*Zn + j - 2])
			+ a[3] * (Vx[(i)*Zn + j + 4] - Vx[(i)*Zn + j - 3])
			+ a[4] * (Vx[(i)*Zn + j + 5] - Vx[(i)*Zn + j - 4])
			+ a[5] * (Vx[(i)*Zn + j + 6] - Vx[(i)*Zn + j - 5])) / dz;

		dvzdz = (a[0] * (Vz[(i)*Zn + j + 0] - Vz[(i)*Zn + j - 1])
			+ a[1] * (Vz[(i)*Zn + j + 1] - Vz[(i)*Zn + j - 2])
			+ a[2] * (Vz[(i)*Zn + j + 2] - Vz[(i)*Zn + j - 3])
			+ a[3] * (Vz[(i)*Zn + j + 3] - Vz[(i)*Zn + j - 4])
			+ a[4] * (Vz[(i)*Zn + j + 4] - Vz[(i)*Zn + j - 5])
			+ a[5] * (Vz[(i)*Zn + j + 5] - Vz[(i)*Zn + j - 6])) / dz;

		dvzdx = (a[0] * (Vz[(i + 1) * Zn + j] - Vz[(i - 0) * Zn + j])
			+ a[1] * (Vz[(i + 2) * Zn + j] - Vz[(i - 1) * Zn + j])
			+ a[2] * (Vz[(i + 3) * Zn + j] - Vz[(i - 2) * Zn + j])
			+ a[3] * (Vz[(i + 4) * Zn + j] - Vz[(i - 3) * Zn + j])
			+ a[4] * (Vz[(i + 5) * Zn + j] - Vz[(i - 4) * Zn + j])
			+ a[5] * (Vz[(i + 6) * Zn + j] - Vz[(i - 5) * Zn + j])) / dx;

		theta_x[i * Zn + j] = -dt * powf(Vp[i * Zn + j], 2) * dvxdx + theta_x[i * Zn + j];

		theta_z[i * Zn + j] = -dt * powf(Vp[i * Zn + j], 2) * dvzdz + theta_z[i * Zn + j];
		////
		omega_x[i * Zn + j] = dt * powf(Vs[i * Zn + j], 2) * dvzdx + omega_x[i * Zn + j];

		omega_z[i * Zn + j] = -dt * powf(Vs[i * Zn + j], 2) * dvxdz + omega_z[i * Zn + j];
		////
		duxdx[i * Zn + j] = -dt * powf(Vs[i * Zn + j], 2) * dvxdx + duxdx[i * Zn + j];

		duxdz[i * Zn + j] = -dt * powf(Vs[i * Zn + j], 2) * dvxdz + duxdz[i * Zn + j];

		duzdx[i * Zn + j] = -dt * powf(Vs[i * Zn + j], 2) * dvzdx + duzdx[i * Zn + j];

		duzdz[i * Zn + j] = -dt * powf(Vs[i * Zn + j], 2) * dvzdz + duzdz[i * Zn + j];

		theta[i * Zn + j] = theta_x[i * Zn + j] + theta_z[i * Zn + j];
		omega[i * Zn + j] = omega_x[i * Zn + j] + omega_z[i * Zn + j];

	}

}
__global__
void save_wavefiled(float* d_vx_up, float* d_vz_up, float* d_theta_up, float* d_omega_up, float* d_duzdx_up, float* d_duzdz_up, float* d_duxdx_up, float* d_duxdz_up,
	float* d_vx_dn, float* d_vz_dn, float* d_theta_dn, float* d_omega_dn, float* d_duzdx_dn, float* d_duzdz_dn, float* d_duxdx_dn, float* d_duxdz_dn,
	float* d_vx_lf, float* d_vz_lf, float* d_theta_lf, float* d_omega_lf, float* d_duzdx_lf, float* d_duzdz_lf, float* d_duxdx_lf, float* d_duxdz_lf,
	float* d_vx_rt, float* d_vz_rt, float* d_theta_rt, float* d_omega_rt, float* d_duzdx_rt, float* d_duzdz_rt, float* d_duxdx_rt, float* d_duxdz_rt, 
	float* d_vx, float* d_vz, float* d_theta, float* d_omega, float* d_duzdx, float* d_duzdz, float* d_duxdx, float* d_duxdz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= 0 && i < Xn && j >= L - N && j < L + N)
	{
		d_vx_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_vx[i * Zn + j];
		d_vz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_vz[i * Zn + j];
		d_theta_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_theta[i * Zn + j];
		d_omega_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_omega[i * Zn + j];
		d_duzdx_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duzdx[i * Zn + j];
		d_duzdz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duzdz[i * Zn + j];
		d_duxdx_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duxdx[i * Zn + j];
		d_duxdz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duxdz[i * Zn + j];
	}
	if (i >= 0 && i < Xn && j >= Zn - L - N && j < Zn - L + N)
	{
		d_vx_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_vx[i * Zn + j];
		d_vz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_vz[i * Zn + j];
		d_theta_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_theta[i * Zn + j];
		d_omega_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_omega[i * Zn + j];
		d_duzdx_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duzdx[i * Zn + j];
		d_duzdz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duzdz[i * Zn + j];
		d_duxdx_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duxdx[i * Zn + j];
		d_duxdz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duxdz[i * Zn + j];
	}
	if (i >= L - N && i < L + N && j >= 0 && j < Zn)
	{
		d_vx_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_vx[i * Zn + j];
		d_vz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_vz[i * Zn + j];
		d_theta_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_theta[i * Zn + j];
		d_omega_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_omega[i * Zn + j];
		d_duzdx_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duzdx[i * Zn + j];
		d_duzdz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duzdz[i * Zn + j];
		d_duxdx_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duxdx[i * Zn + j];
		d_duxdz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duxdz[i * Zn + j];

	}
	if (i >= Xn - L - N && i < Xn - L + N && j >= 0 && j < Zn)
	{
		d_vx_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_vx[i * Zn + j];
		d_vz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_vz[i * Zn + j];
		d_theta_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_theta[i * Zn + j];
		d_omega_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_omega[i * Zn + j];
		d_duzdx_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duzdx[i * Zn + j];
		d_duzdz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duzdz[i * Zn + j];
		d_duxdx_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duxdx[i * Zn + j];
		d_duxdz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duxdz[i * Zn + j];

	}

}
__global__
void read_last_wavefiled(float* d_theta_s, float* d_omega_s, float* d_Vx_s, float* d_Vz_s, float* d_duzdx_s, float* d_duzdz_s, float* d_duxdz_s, float* d_duxdx_s,
	float* d_theta_x_s, float* d_omega_x_s, float* d_theta_z_s, float* d_omega_z_s, float* d_Vpx_s, float* d_Vpz_s, float* d_Vsx_s, float* d_Vsz_s,
	float* d_theta, float* d_omega, float* d_Vx, float* d_Vz, float* d_duzdx, float* d_duzdz, float* d_duxdz, float* d_duxdx,
	float* d_theta_x, float* d_omega_x, float* d_theta_z, float* d_omega_z, float* d_Vpx, float* d_Vpz, float* d_Vsx, float* d_Vsz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L - N && i < Xn - L + N && j >= L - N && j < Zn - L + N)
	{
		d_theta_s[i * Zn + j] = d_theta[i * Zn + j];
		d_omega_s[i * Zn + j] = d_omega[i * Zn + j];
		d_Vx_s[i * Zn + j] = d_Vx[i * Zn + j];
		d_Vz_s[i * Zn + j] = d_Vz[i * Zn + j];
		d_theta_x_s[i * Zn + j] = d_theta_x[i * Zn + j];
		d_omega_x_s[i * Zn + j] = d_omega_x[i * Zn + j];
		d_theta_z_s[i * Zn + j] = d_theta_z[i * Zn + j];
		d_omega_z_s[i * Zn + j] = d_omega_z[i * Zn + j];
		d_Vpx_s[i * Zn + j] = d_Vpx[i * Zn + j];
		d_Vpz_s[i * Zn + j] = d_Vpz[i * Zn + j];
		d_Vsx_s[i * Zn + j] = d_Vsx[i * Zn + j];
		d_Vsz_s[i * Zn + j] = d_Vsz[i * Zn + j];
		d_duzdx_s[i * Zn + j] = d_duzdx[i * Zn + j];
		d_duzdz_s[i * Zn + j] = d_duzdz[i * Zn + j];
		d_duxdx_s[i * Zn + j] = d_duxdx[i * Zn + j];
		d_duxdz_s[i * Zn + j] = d_duxdz[i * Zn + j];
	}
}

__global__ void read_last_wavefiled2(float* d_theta_s, float* d_omega_s, float* d_Vx_s, float* d_Vz_s, float* d_duzdx_s, float* d_duzdz_s, float* d_duxdz_s, float* d_duxdx_s,
	float* d_Vpx_s, float* d_Vpz_s, float* d_Vsx_s, float* d_Vsz_s,
	float* d_theta, float* d_omega, float* d_Vx, float* d_Vz, float* d_duzdx, float* d_duzdz, float* d_duxdz, float* d_duxdx,
	float* d_Vpx, float* d_Vpz, float* d_Vsx, float* d_Vsz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L - N && i < Xn - L + N && j >= L - N && j < Zn - L + N)
	{
		d_theta_s[i * Zn + j] = d_theta[i * Zn + j];
		d_omega_s[i * Zn + j] = d_omega[i * Zn + j];
		d_Vx_s[i * Zn + j] = d_Vx[i * Zn + j];
		d_Vz_s[i * Zn + j] = d_Vz[i * Zn + j];
		
		d_Vpx_s[i * Zn + j] = d_Vpx[i * Zn + j];
		d_Vpz_s[i * Zn + j] = d_Vpz[i * Zn + j];
		d_Vsx_s[i * Zn + j] = d_Vsx[i * Zn + j];
		d_Vsz_s[i * Zn + j] = d_Vsz[i * Zn + j];
		d_duzdx_s[i * Zn + j] = d_duzdx[i * Zn + j];
		d_duzdz_s[i * Zn + j] = d_duzdz[i * Zn + j];
		d_duxdx_s[i * Zn + j] = d_duxdx[i * Zn + j];
		d_duxdz_s[i * Zn + j] = d_duxdz[i * Zn + j];
	}
}

__global__
void read_wavefiled1(float* d_Vx_up, float* d_Vz_up, float* d_Vx_dn, float* d_Vz_dn, float* d_Vx_lf, float* d_Vz_lf, float* d_Vx_rt, float* d_Vz_rt, 
	float* d_Vx, float* d_Vz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= 0 && i < Xn && j >= L - N && j < L + N)
	{

		d_Vz[i * Zn + j] = d_Vz_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_Vx[i * Zn + j] = d_Vx_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
	}
	if (i >= 0 && i < Xn && j >= Zn - L - N && j < Zn - L + N)
	{

		d_Vz[i * Zn + j] = d_Vz_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_Vx[i * Zn + j] = d_Vx_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
	}
	if (i >= L - N && i < L + N && j >= 0 && j < Zn)
	{

		d_Vz[i * Zn + j] = d_Vz_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_Vx[i * Zn + j] = d_Vx_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
	}
	if (i >= Xn - L - N && i < Xn - L + N && j >= 0 && j < Zn)
	{

		d_Vz[i * Zn + j] = d_Vz_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_Vx[i * Zn + j] = d_Vx_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
	}

}
__global__
void read_wavefiled2(float* d_theta_up, float* d_omega_up, float* d_duxdz_up, float* d_duxdx_up, float* d_duzdz_up, float* d_duzdx_up,
	float* d_theta_dn, float* d_omega_dn, float* d_duxdz_dn, float* d_duxdx_dn, float* d_duzdz_dn, float* d_duzdx_dn,
	float* d_theta_lf, float* d_omega_lf, float* d_duxdz_lf, float* d_duxdx_lf, float* d_duzdz_lf, float* d_duzdx_lf,
	float* d_theta_rt, float* d_omega_rt, float* d_duxdz_rt, float* d_duxdx_rt, float* d_duzdz_rt, float* d_duzdx_rt,
	float* d_theta, float* d_omega, float* d_duxdz, float* d_duxdx, float* d_duzdz, float* d_duzdx, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= 0 && i < Xn && j >= L - N && j < L + N)
	{
		d_theta[i * Zn + j] = d_theta_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_omega[i * Zn + j] = d_omega_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duxdz[i * Zn + j] = d_duxdz_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duxdx[i * Zn + j] = d_duxdx_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duzdz[i * Zn + j] = d_duzdz_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duzdx[i * Zn + j] = d_duzdx_up[(t) * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
	}
	if (i >= 0 && i < Xn && j >= Zn - L - N && j < Zn - L + N)
	{
		d_theta[i * Zn + j] = d_theta_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_omega[i * Zn + j] = d_omega_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duxdz[i * Zn + j] = d_duxdz_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duxdx[i * Zn + j] = d_duxdx_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duzdz[i * Zn + j] = d_duzdz_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duzdx[i * Zn + j] = d_duzdx_dn[(t) * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
	}
	if (i >= L - N && i < L + N && j >= 0 && j < Zn)
	{
		d_theta[i * Zn + j] = d_theta_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_omega[i * Zn + j] = d_omega_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duxdz[i * Zn + j] = d_duxdz_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duxdx[i * Zn + j] = d_duxdx_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duzdz[i * Zn + j] = d_duzdz_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duzdx[i * Zn + j] = d_duzdx_lf[(t) * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
	}
	if (i >= Xn - L - N && i < Xn - L + N && j >= 0 && j < Zn)
	{
		d_theta[i * Zn + j] = d_theta_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_omega[i * Zn + j] = d_omega_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duxdz[i * Zn + j] = d_duxdz_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duxdx[i * Zn + j] = d_duxdx_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duzdz[i * Zn + j] = d_duzdz_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duzdx[i * Zn + j] = d_duzdx_rt[(t) * (2 * N * Zn) + (i - Xn + L) * Zn + j];
	}

}

__global__ void corr_v(float* fenzi_PP, float* fenzi_PS, float* fenmu_P, float* SS_Px, float* SS_Pz, float* PP_Px, float* PP_Pz, float* PP_Sx, float* PP_Sz)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		fenzi_PP[i * Zn + j] += SS_Px[i * Zn + j] * PP_Px[i * Zn + j] + SS_Pz[i * Zn + j] * PP_Pz[i * Zn + j];
		fenmu_P[i * Zn + j] += SS_Px[i * Zn + j] * SS_Px[i * Zn + j] + SS_Pz[i * Zn + j] * SS_Pz[i * Zn + j];
		fenzi_PS[i * Zn + j] += SS_Px[i * Zn + j] * PP_Sx[i * Zn + j] + SS_Pz[i * Zn + j] * PP_Sz[i * Zn + j];

	}
}

__global__ void image_fun(float* fenzi, float* fenmu, float* image)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		//	if (abs(1.0*(i - X0) / (j - Z0)) < aperture)                           //偏移孔径
		image[i * Zn + j] = fenzi[i * Zn + j] / fenmu[i * Zn + j];
		/*	else
		image[i*Zn + j] = 0.0;
		*/
	}
}

__global__ void Laplace(float dh, float* image, float* image_lap)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= L + 2 && i < Xn - L - 2 && j >= L + 2 && j < Zn - L - 2)
	{
		image_lap[i * Zn + j] = (1 / dh / dh) * (image[(i + 1) * Zn + j] + image[(i - 1) * Zn + j] - 2 * image[i * Zn + j] +
			image[i * Zn + j + 1] + image[i * Zn + j - 1] - 2 * image[i * Zn + j]);
	}

}

__global__
void load_record(int reciver, float* vx, float* vz, float* record_vx, float* record_vz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L && i < Xn - L && j == reciver)
	{
		vx[i * Zn + j] = record_vx[i * NT + t];
		vz[i * Zn + j] = record_vz[i * NT + t];
		
	}
}
float adsource(float* res_recordux, float* res_recorduz, float* cur_recordux, float* cur_recorduz, float* ori_recordux, float* ori_recorduz, int t) {

	int i, j, k;
	if (i >= L && i < Xn - L && j >= 0 && j < NT)
	{
			res_recordux[i * NT + t] = cur_recordux[i * NT + t] - ori_recordux[i * NT + t];
			res_recorduz[i * NT + t] = cur_recorduz[i * NT + t] - ori_recorduz[i * NT + t];
		
	}

		

	

}

__global__
void revise_model_end(float* Vp, float* Vs, float* Vp_end, float* Vs_end, float* Grad_Vp, float* Grad_Vs, int k)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float s;
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{


		Vp_end[i * Zn + j] = Vp[i * Zn + j] - a_2 * pow(0.95, k) * Grad_Vp[i * Zn + j];
		Vs_end[i * Zn + j] = Vs[i * Zn + j] - a_2 * pow(0.95, k) * Grad_Vs[i * Zn + j];



	}
}
__global__
void gradfun(float* Grad_Vp, float* Grad_Vs, float* Unx_s, float* Unz_s, float* theta_r, float* omega_r, float* dux_z_r, float* dux_x_r, float* duz_z_r, float* duz_x_r, float* SSS, float* Vp, float* Vs, float* P, float dt, float dx, float dz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float duxdx = 0.0f;
	float duzdz = 0.0f;
	float duzdx = 0.0f;
	float duxdz = 0.0f;
	float druxdx = 0.0f;
	float druzdz = 0.0f;
	float druzdx = 0.0f;
	float druxdz = 0.0f;
	float a[N] = { 1.2213364, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		duxdx = (a[0] * (Unx_s[(i)*Zn + j] - Unx_s[(i - 1) * Zn + j])
			+ a[1] * (Unx_s[(i + 1) * Zn + j] - Unx_s[(i - 2) * Zn + j])
			+ a[2] * (Unx_s[(i + 2) * Zn + j] - Unx_s[(i - 3) * Zn + j])
			+ a[3] * (Unx_s[(i + 3) * Zn + j] - Unx_s[(i - 4) * Zn + j])
			+ a[4] * (Unx_s[(i + 4) * Zn + j] - Unx_s[(i - 5) * Zn + j])
			+ a[5] * (Unx_s[(i + 5) * Zn + j] - Unx_s[(i - 6) * Zn + j])) / dx;
		duzdz = (a[0] * (Unz_s[(i)*Zn + j] - Unz_s[(i)*Zn + j - 1])
			+ a[1] * (Unz_s[(i)*Zn + j + 1] - Unz_s[(i)*Zn + j - 2])
			+ a[2] * (Unz_s[(i)*Zn + j + 2] - Unz_s[(i)*Zn + j - 3])
			+ a[3] * (Unz_s[(i)*Zn + j + 3] - Unz_s[(i)*Zn + j - 4])
			+ a[4] * (Unz_s[(i)*Zn + j + 4] - Unz_s[(i)*Zn + j - 5])
			+ a[5] * (Unz_s[(i)*Zn + j + 5] - Unz_s[(i)*Zn + j - 6])) / dz;
		duzdx = (a[0] * (Unz_s[(i + 1) * Zn + j] - Unz_s[(i)*Zn + j])
			+ a[1] * (Unz_s[(i + 2) * Zn + j] - Unz_s[(i - 1) * Zn + j])
			+ a[2] * (Unz_s[(i + 3) * Zn + j] - Unz_s[(i - 2) * Zn + j])
			+ a[3] * (Unz_s[(i + 4) * Zn + j] - Unz_s[(i - 3) * Zn + j])
			+ a[4] * (Unz_s[(i + 5) * Zn + j] - Unz_s[(i - 4) * Zn + j])
			+ a[5] * (Unz_s[(i + 6) * Zn + j] - Unz_s[(i - 5) * Zn + j])) / dx;
		duxdz = (a[0] * (Unx_s[(i)*Zn + j + 1] - Unx_s[(i)*Zn + j])
			+ a[1] * (Unx_s[(i)*Zn + j + 2] - Unx_s[(i)*Zn + j - 1])
			+ a[2] * (Unx_s[(i)*Zn + j + 3] - Unx_s[(i)*Zn + j - 2])
			+ a[3] * (Unx_s[(i)*Zn + j + 4] - Unx_s[(i)*Zn + j - 6])
			+ a[4] * (Unx_s[(i)*Zn + j + 5] - Unx_s[(i)*Zn + j - 4])
			+ a[5] * (Unx_s[(i)*Zn + j + 6] - Unx_s[(i)*Zn + j - 5])) / dz;




		Grad_Vp[i * Zn + j] -= 2 * P[i * Zn + j] * Vp[i * Zn + j] * ((duxdx + duzdz) * (theta_r[i * Zn + j]));




		Grad_Vs[i * Zn + j] -= 2 * P[i * Zn + j] * Vs[i * Zn + j] * (omega_r[i * Zn + j] * (duxdz - duzdx) + duz_x_r[i * Zn + j] * duzdx + duz_z_r[i * Zn + j] * duzdz
			+ dux_z_r[i * Zn + j] * duxdz + dux_x_r[i * Zn + j] * duxdx);


		SSS[i * Zn + j] += ((Unx_s[(i)*Zn + j] * Unx_s[(i)*Zn + j]) + (Unz_s[(i)*Zn + j] * Unz_s[(i)*Zn + j]));

	}
}
__global__
void caculate_Grad(float* Grad_Vp, float* Grad_Vs, float* Unx_s, float* Unz_s, float* theta_r, float*omega_r, float* dux_z_r, float* dux_x_r, float* duz_z_r, float* duz_x_r, float* SSS, float* Vp, float* Vs, float* P, float dt, float dx, float dz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float duxdx = 0.0f;
	float duzdz = 0.0f;
	float duzdx = 0.0f;
	float duxdz = 0.0f;
	float druxdx = 0.0f;
	float druzdz = 0.0f;
	float druzdx = 0.0f;
	float druxdz = 0.0f;
	float a[N] = { +1.2213e+0,-9.6931e-2,1.7448e-2,-2.9673e-3,+3.5901e-4,-2.1848e-5 };
	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		duxdx = (a[0] * (Unx_s[(i)*Zn + j] - Unx_s[(i - 1) * Zn + j])
			+ a[1] * (Unx_s[(i + 1) * Zn + j] - Unx_s[(i - 2) * Zn + j])
			+ a[2] * (Unx_s[(i + 2) * Zn + j] - Unx_s[(i - 3) * Zn + j])
			+ a[3] * (Unx_s[(i + 3) * Zn + j] - Unx_s[(i - 4) * Zn + j])
			+ a[4] * (Unx_s[(i + 4) * Zn + j] - Unx_s[(i - 5) * Zn + j])
			+ a[5] * (Unx_s[(i + 5) * Zn + j] - Unx_s[(i - 6) * Zn + j])) / dx;
		duzdz = (a[0] * (Unz_s[(i)*Zn + j] - Unz_s[(i)*Zn + j - 1])
			+ a[1] * (Unz_s[(i)*Zn + j + 1] - Unz_s[(i)*Zn + j - 2])
			+ a[2] * (Unz_s[(i)*Zn + j + 2] - Unz_s[(i)*Zn + j - 3])
			+ a[3] * (Unz_s[(i)*Zn + j + 3] - Unz_s[(i)*Zn + j - 4])
			+ a[4] * (Unz_s[(i)*Zn + j + 4] - Unz_s[(i)*Zn + j - 5])
			+ a[5] * (Unz_s[(i)*Zn + j + 5] - Unz_s[(i)*Zn + j - 6])) / dz;
		duzdx = (a[0] * (Unz_s[(i + 1) * Zn + j] - Unz_s[(i)*Zn + j])
			+ a[1] * (Unz_s[(i + 2) * Zn + j] - Unz_s[(i - 1) * Zn + j])
			+ a[2] * (Unz_s[(i + 3) * Zn + j] - Unz_s[(i - 2) * Zn + j])
			+ a[3] * (Unz_s[(i + 4) * Zn + j] - Unz_s[(i - 3) * Zn + j])
			+ a[4] * (Unz_s[(i + 5) * Zn + j] - Unz_s[(i - 4) * Zn + j])
			+ a[5] * (Unz_s[(i + 6) * Zn + j] - Unz_s[(i - 5) * Zn + j])) / dx;
		duxdz = (a[0] * (Unx_s[(i)*Zn + j + 1] - Unx_s[(i)*Zn + j])
			+ a[1] * (Unx_s[(i)*Zn + j + 2] - Unx_s[(i)*Zn + j - 1])
			+ a[2] * (Unx_s[(i)*Zn + j + 3] - Unx_s[(i)*Zn + j - 2])
			+ a[3] * (Unx_s[(i)*Zn + j + 4] - Unx_s[(i)*Zn + j - 6])
			+ a[4] * (Unx_s[(i)*Zn + j + 5] - Unx_s[(i)*Zn + j - 4])
			+ a[5] * (Unx_s[(i)*Zn + j + 6] - Unx_s[(i)*Zn + j - 5])) / dz;




		Grad_Vp[i * Zn + j] -= 2 * P[i * Zn + j] * Vp[i * Zn + j] * ((duxdx + duzdz) * (theta_r[i * Zn + j]));




		Grad_Vs[i * Zn + j] -= 2 * P[i * Zn + j] * Vs[i * Zn + j] * (omega_r[i * Zn + j]*(duxdz- duzdx)+ duz_x_r[i * Zn + j]* duzdx+ duz_z_r[i * Zn + j] * duzdz
			+ dux_z_r[i * Zn + j] * duxdz+ dux_x_r[i * Zn + j] * duxdx);


		SSS[i * Zn + j] += ((Unx_s[(i)*Zn + j] * Unx_s[(i)*Zn + j]) + (Unz_s[(i)*Zn + j] * Unz_s[(i)*Zn + j]));

	}
}
__global__
void remove(int Sx, int Sz, int Z_receive, int t0, float dt, float dh, float* v, float* record_vx, float* record_vz)
{
	int  t;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	float distance;
	if (i >= 0 && i < Xn && j == Z_receive)
	{
		for (t = 0; t < NT; t++)
		{
			distance = sqrtf(float(abs(Sx - i) * abs(Sx - i) + abs(Z_receive - Sz) * abs(Z_receive - Sz)));
			if (t < (2 * t0 + distance * dh * 1.0 / (dt * v[Sx * Zn + j])))
			{
				record_vx[i * NT + t] = 0;
				record_vz[i * NT + t] = 0;

			}


		}

	}
}
int main()
{
	hipSetDevice(2);
	FILE* fp;
	int i, j, t, t0, l;
	int starttime, endtime, caltime;
	char filename[2000];
	int Sx, Sz, Z_receive, viewpoint_x, viewpoint_z;
	int size = Xn * Zn;

	float* Vx, * Vz, * theta, * omega, * duzdx, * duzdz, * duxdx, * duxdz;
	float* Vpx, * Vpz, *Vsx, * Vsz, * theta_x, * omega_x, * theta_z, * omega_z;
	float* Vx_r, * Vz_r;
	float* Vx_s, * Vz_s;
	float* record_vx, * record_vz,* record_vx_r, * record_vz_r;
	float* record_vx_1, * record_vz_1;
	float* record_vx_2, * record_vz_2;
	float* Vp_end, * Vs_end;
	float* d_Vp_end, * d_Vs_end;
	float* Grad_Vp, * Grad_Vs;
	float* d_Grad_Vp, * d_Grad_Vs;
	float* source,* Vp, * Vs, * P;
	float dt, dx, dz, dh;
	float* SSS;
	float* Vpx_r, * Vpz_r, *Vsx_r, * Vsz_r;


	dt = 0.001;
	dx = 10.0;
	dz =10.0;
	dh = 10.0;
	float* begin;
	float* begin1;


	float* d_Vx, * d_Vz, * d_theta, * d_omega, * d_duzdx, * d_duzdz, * d_duxdx, * d_duxdz;
	float* d_Vpx, * d_Vpz, * d_Vsx, * d_Vsz, * d_theta_x, * d_omega_x, * d_theta_z, * d_omega_z;

	float* d_Vx_r, * d_Vz_r, * d_theta_r, * d_omega_r, * d_duzdx_r, * d_duzdz_r, * d_duxdx_r, * d_duxdz_r;
	float* d_Vpx_r, * d_Vpz_r, * d_Vsx_r, * d_Vsz_r, * d_theta_x_r, * d_omega_x_r, * d_theta_z_r, * d_omega_z_r;

	float* d_Vx_s, * d_Vz_s, * d_theta_s, * d_omega_s, * d_duzdx_s, * d_duzdz_s, * d_duxdx_s, * d_duxdz_s;
	float* d_Vpx_s, * d_Vpz_s, * d_Vsx_s, * d_Vsz_s, * d_theta_x_s, * d_omega_x_s, * d_theta_z_s, * d_omega_z_s;


	float* d_Vx_up, * d_Vz_up, * d_theta_up, * d_omega_up, * d_duzdx_up, * d_duzdz_up, * d_duxdx_up, * d_duxdz_up;
	float* d_Vx_dn, * d_Vz_dn, * d_theta_dn, * d_omega_dn, * d_duzdx_dn, * d_duzdz_dn, * d_duxdx_dn, * d_duxdz_dn;
	float* d_Vx_lf, * d_Vz_lf, * d_theta_lf, * d_omega_lf, * d_duzdx_lf, * d_duzdz_lf, * d_duxdx_lf, * d_duxdz_lf;
	float* d_Vx_rt, * d_Vz_rt, * d_theta_rt, * d_omega_rt, * d_duzdx_rt, * d_duzdz_rt, * d_duxdx_rt, * d_duxdz_rt;

	float* d_Vp, * d_Vs, * d_P, * d_source,* ax, * az;
	float* d_SSS;
	float* d_record_vx, * d_record_vz, * d_record_vx_r, * d_record_vz_r;
	float* d_record_vx_1, * d_record_vz_1;
	float* d_record_vx_2, * d_record_vz_2;

	float* O_duz_xdz, * O_duz_zdx, * O_dux_zdx, * O_dux_xdz, * O_thetax, * O_thetaz, * O_omegaz, * O_omegax;
	float* F_xx, * F_zz, * F_xz, * F_zx;
	float* O_duz_xdz_r, * O_duz_zdx_r, * O_dux_zdx_r, * O_dux_xdz_r, * O_thetax_r, * O_thetaz_r, * O_omegaz_r, * O_omegax_r;
	float* F_xx_r, * F_zz_r, * F_xz_r, * F_zx_r;
	float* Vpx_now, * Vpz_now, * Vsx_now, * Vsz_now, * Vpx_past, * Vpz_past, * Vsx_past, * Vsz_past;
	float* Vpx_now_r, * Vpz_now_r, * Vsx_now_r, * Vsz_now_r, * Vpx_past_r, * Vpz_past_r, * Vsx_past_r, * Vsz_past_r;
	float* Vpx_now_s, * Vpz_now_s, * Vsx_now_s, * Vsz_now_s, * Vpx_past_s, * Vpz_past_s, * Vsx_past_s, * Vsz_past_s;
	float* fenzi_PP, *fenzi_PS, *fenmu_P, *d_image_PP, *d_image_PS, * d_image_PP_lap, * d_image_PS_lap;
	float* image_PP, *image_PS, * image_PP_lap, * image_PS_lap;
	float* All_image_PP, *All_image_PS;
	float* All_image_PP_lap, * All_image_PS_lap;
	float* res_recordux, * res_recorduz;
	

	Vx = (float*)malloc(size * sizeof(float));
	Vz = (float*)malloc(size * sizeof(float));
	theta = (float*)malloc(size * sizeof(float));
	omega = (float*)malloc(size * sizeof(float));
	Vpx = (float*)malloc(size * sizeof(float));
	Vpz = (float*)malloc(size * sizeof(float));
	Vsx = (float*)malloc(size * sizeof(float));
	Vsz = (float*)malloc(size * sizeof(float));

	Vx_r = (float*)malloc(size * sizeof(float));
	Vz_r = (float*)malloc(size * sizeof(float));
	Vpx_r = (float*)malloc(size * sizeof(float));
	Vpz_r = (float*)malloc(size * sizeof(float));
	Vsx_r = (float*)malloc(size * sizeof(float));
	Vsz_r = (float*)malloc(size * sizeof(float));

	Vx_s = (float*)malloc(size * sizeof(float));
	Vz_s = (float*)malloc(size * sizeof(float));

	Vp = (float*)malloc(size * sizeof(float));
	Vs = (float*)malloc(size * sizeof(float));
	P = (float*)malloc(size * sizeof(float));
	ax = (float*)malloc(size * sizeof(float));
	az = (float*)malloc(size * sizeof(float));
	image_PP = (float*)calloc(size, sizeof(float));
	image_PS = (float*)calloc(size, sizeof(float));
	All_image_PP = (float*)calloc(size, sizeof(float));
	All_image_PS = (float*)calloc(size, sizeof(float));
	image_PP_lap = (float*)calloc(size, sizeof(float));
	image_PS_lap = (float*)calloc(size, sizeof(float));	
	All_image_PP_lap = (float*)calloc(size, sizeof(float));
	All_image_PS_lap = (float*)calloc(size, sizeof(float));

	source = (float*)malloc(NT * sizeof(float));
	record_vx = (float*)malloc(NT * Xn * sizeof(float));
	record_vz = (float*)malloc(NT * Xn * sizeof(float));
	record_vx_r = (float*)malloc(NT * Xn * sizeof(float));
	record_vz_r = (float*)malloc(NT * Xn * sizeof(float));

	record_vx_1 = (float*)malloc(NT * Xn * sizeof(float));
	record_vz_1 = (float*)malloc(NT * Xn * sizeof(float));
	record_vx_2 = (float*)malloc(NT * Xn * sizeof(float));
	record_vz_2 = (float*)malloc(NT * Xn * sizeof(float));


	begin = (float*)malloc(size * sizeof(float));
	begin1 = (float*)malloc(Xn * NT * sizeof(float));

	Vp_end = (float*)malloc(size * sizeof(float));
	Vs_end = (float*)malloc(size * sizeof(float));
	Grad_Vp = (float*)malloc(size * sizeof(float));
	Grad_Vs = (float*)malloc(size * sizeof(float));
	SSS = (float*)malloc(size * sizeof(float));

	hipMalloc((void**)&d_Vx, size * sizeof(float));
	hipMalloc((void**)&d_Vz, size * sizeof(float));
	hipMalloc((void**)&d_theta, size * sizeof(float));
	hipMalloc((void**)&d_omega, size * sizeof(float));
	hipMalloc((void**)&d_duzdx, size * sizeof(float));
	hipMalloc((void**)&d_duxdx, size * sizeof(float));	
	hipMalloc((void**)&d_duzdz, size * sizeof(float)); 
	hipMalloc((void**)&d_duxdz, size * sizeof(float));
	hipMalloc((void**)&d_Vpx, size * sizeof(float));
	hipMalloc((void**)&d_Vsx, size * sizeof(float));
	hipMalloc((void**)&d_Vpz, size * sizeof(float));
	hipMalloc((void**)&d_Vsz, size * sizeof(float));
	hipMalloc((void**)&d_theta_x, size * sizeof(float));
	hipMalloc((void**)&d_theta_z, size * sizeof(float));
	hipMalloc((void**)&d_omega_x, size * sizeof(float));
	hipMalloc((void**)&d_omega_z, size * sizeof(float));
	hipMalloc((void**)&d_SSS, size * sizeof(float));

	
	hipMalloc((void**)&d_Vx_r, size * sizeof(float));
	hipMalloc((void**)&d_Vz_r, size * sizeof(float));
	hipMalloc((void**)&d_theta_r, size * sizeof(float));
	hipMalloc((void**)&d_omega_r, size * sizeof(float));
	hipMalloc((void**)&d_duzdx_r, size * sizeof(float));
	hipMalloc((void**)&d_duxdx_r, size * sizeof(float));
	hipMalloc((void**)&d_duzdz_r, size * sizeof(float));
	hipMalloc((void**)&d_duxdz_r, size * sizeof(float));
	hipMalloc((void**)&d_Vpx_r, size * sizeof(float));
	hipMalloc((void**)&d_Vsx_r, size * sizeof(float));
	hipMalloc((void**)&d_Vpz_r, size * sizeof(float));
	hipMalloc((void**)&d_Vsz_r, size * sizeof(float));
	hipMalloc((void**)&d_theta_x_r, size * sizeof(float));
	hipMalloc((void**)&d_theta_z_r, size * sizeof(float));
	hipMalloc((void**)&d_omega_x_r, size * sizeof(float));
	hipMalloc((void**)&d_omega_z_r, size * sizeof(float));

	hipMalloc((void**)&d_Vx_s, size * sizeof(float));
	hipMalloc((void**)&d_Vz_s, size * sizeof(float));
	hipMalloc((void**)&d_theta_s, size * sizeof(float));
	hipMalloc((void**)&d_omega_s, size * sizeof(float));
	hipMalloc((void**)&d_duzdx_s, size * sizeof(float));
	hipMalloc((void**)&d_duxdx_s, size * sizeof(float));
	hipMalloc((void**)&d_duzdz_s, size * sizeof(float));
	hipMalloc((void**)&d_duxdz_s, size * sizeof(float));
	hipMalloc((void**)&d_Vpx_s, size * sizeof(float));
	hipMalloc((void**)&d_Vsx_s, size * sizeof(float));
	hipMalloc((void**)&d_Vpz_s, size * sizeof(float));
	hipMalloc((void**)&d_Vsz_s, size * sizeof(float));
	hipMalloc((void**)&d_theta_x_s, size * sizeof(float));
	hipMalloc((void**)&d_theta_z_s, size * sizeof(float));
	hipMalloc((void**)&d_omega_x_s, size * sizeof(float));
	hipMalloc((void**)&d_omega_z_s, size * sizeof(float));

	hipMalloc((void**)&d_Vp, size * sizeof(float));
	hipMalloc((void**)&d_Vs, size * sizeof(float));
	hipMalloc((void**)&d_P, size * sizeof(float));
	hipMalloc((void**)&d_source, NT * sizeof(float));
	hipMalloc((void**)&d_record_vx, Xn * NT * sizeof(float));
	hipMalloc((void**)&d_record_vz, Xn * NT * sizeof(float));
	hipMalloc((void**)&d_record_vx_r, Xn * NT * sizeof(float));
	hipMalloc((void**)&d_record_vz_r, Xn * NT * sizeof(float));

	hipMalloc((void**)&d_record_vx_1, Xn * NT * sizeof(float));
	hipMalloc((void**)&d_record_vz_1, Xn * NT * sizeof(float));
	hipMalloc((void**)&d_record_vx_2, Xn * NT * sizeof(float));
	hipMalloc((void**)&d_record_vz_2, Xn * NT * sizeof(float));
	
	hipMalloc((void**)&res_recordux, Xn* NT * sizeof(float));
	hipMalloc((void**)&res_recorduz, Xn* NT * sizeof(float));

	hipMalloc((void**)&d_Vx_up, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_Vz_up, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_up, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_up, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_up, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_up, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_up, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_up, 2 * N * Xn * (NT - 1) * sizeof(float));

	hipMalloc((void**)&d_Vx_dn, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_Vz_dn, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_dn, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_dn, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_dn, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_dn, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_dn, 2 * N * Xn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_dn, 2 * N * Xn * (NT - 1) * sizeof(float));


	hipMalloc((void**)&d_Vx_lf, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_Vz_lf, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_lf, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_lf, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_lf, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_lf, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_lf, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_lf, 2 * N * Zn * (NT - 1) * sizeof(float));

	hipMalloc((void**)&d_Grad_Vp, size * sizeof(float));
	hipMalloc((void**)&d_Grad_Vs, size * sizeof(float));

	hipMalloc((void**)&d_Vp_end, size * sizeof(float));
	hipMalloc((void**)&d_Vs_end, size * sizeof(float));

	hipMalloc((void**)&d_Vx_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_Vz_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_rt, 2 * N * Zn * (NT - 1) * sizeof(float));
	
	hipMalloc((void**)&O_duz_xdz, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_duz_zdx, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_dux_zdx, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_dux_xdz, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_thetax, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_thetaz, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_omegaz, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_omegax, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_xx, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_zz, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_xz, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_zx, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_duz_xdz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_duz_zdx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_dux_zdx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_dux_xdz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_thetax_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_thetaz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_omegaz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_omegax_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_xx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_zz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_xz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_zx_r, Xn* Zn * sizeof(float));

	
	hipMalloc((void**)&Vpx_now, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpz_now, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsx_now, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsz_now, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpx_past, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpz_past, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsx_past, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsz_past, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpx_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpz_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsx_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsz_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpx_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpz_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsx_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsz_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpx_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpz_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsx_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsz_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpx_past_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vpz_past_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsx_past_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&Vsz_past_s, Xn* Zn * sizeof(float));
	
	hipMalloc((void**)&fenzi_PP, Xn* Zn * sizeof(float));
	hipMalloc((void**)&fenzi_PS, Xn* Zn * sizeof(float));
	hipMalloc((void**)&fenmu_P, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PP, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PS, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PP_lap, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PS_lap, Xn* Zn * sizeof(float));

	float Nk = PI * PI * FM * FM * dt * dt;
	t0 = ceil(1.0 / (FM * dt));
	for (t = 0; t < NT; t++)
	{
		source[t] = (1.0 - 2.0 * Nk * (t - t0) * (t - t0)) * exp(-Nk * (t - t0) * (t - t0));
		for (j = 0; j < Zn; j++)
		{
			begin1[j * NT + t] = 0.0;
		}


	}

	for (i = 0; i < Xn; i++)
		for (j = 0; j < Zn; j++)
		{
			theta[i * Zn + j] = 0.0;
			omega[i * Zn + j] = 0.0;
			Vx[i * Zn + j] = 0.0;
			Vz[i * Zn + j] = 0.0;
			begin[i * Zn + j] = 0.0;
		}
	creatmodel(Vp, Vs, P);

	dim3 dimGrid(ceil(Xn / 8.0), ceil(Zn / 8.0), 1);
	dim3 dimBlock(8, 8, 1);

	int k;
	for (k = 0; k <1; k++)
	{
		for (i = 0; i < Xn; i++)
			for (j = 0; j < Zn; j++)
			{

				Grad_Vp[i * Zn + j] = 0.0;
				Grad_Vs[i * Zn + j] = 0.0;
				SSS[i * Zn + j] = 0.0;
			}

		hipMemcpy(d_Grad_Vp, begin, size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_Grad_Vs, begin, size * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_SSS, begin, size * sizeof(float), hipMemcpyHostToDevice);
		cout << k + 1 << " th iteration:" << endl;


		for (i = L; i < Xn - L; i++)
		{
			for (j = L; j < 50 + L; j++)
			{
				Vp[i * Zn + j] = 3400;

			}
		}
		for (i = L; i < Xn - L; i++)
		{
			for (j = 50 + L; j < 100 + L; j++)
			{
				Vp[i * Zn + j] = 3800;

			}
		}
		for (i = L; i < Xn - L; i++)
		{
			for (j = 100 + L; j < Zn - L; j++)
			{
				Vp[i * Zn + j] = 4500;

			}
		}
		for (i = L; i < Xn - L; i++)
		{
			for (j = L; j < 100 + L; j++)
			{

				Vs[i * Zn + j] = 2900;
				P[i * Zn + j] = 1.0;
			}
		}
		for (i = L; i < Xn - L; i++)
		{
			for (j = 100 + L; j < Zn - L; j++)
			{

				Vs[i * Zn + j] = 3800;
				P[i * Zn + j] = 1.0;
			}
		}

		//sprintf(filename, "./model/Vp.dat");
		//if ((fp = fopen(filename, "wb")) != NULL)
		//{
		//	for (i = L; i < Xn - L; i++)
		//		for (j = L; j < Zn - L; j++)
		//		{
		//			fwrite(&Vp[i * Zn + j], sizeof(float), 1, fp);

		//		}
		//}
		//fclose(fp);
		//sprintf(filename, "./model/Vs.dat");
		//if ((fp = fopen(filename, "wb")) != NULL)
		//{
		//	for (i = L; i < Xn - L; i++)
		//		for (j = L; j < Zn - L; j++)
		//		{
		//			fwrite(&Vs[i * Zn + j], sizeof(float), 1, fp);

		//		}
		//}
		//fclose(fp);

		xiangbian(P, Vp, Vs);

		for (i = 0; i < Xn; i++)
			for (j = 0; j < Zn; j++)
			{
				if (i >= 0 && i < L)
				{
					ax[i * Zn + j] = log10(1.0 / R) * 1.5 * Vp[i * Zn + j] / (L)*pow(1.0 * (L - i) / (L), 4.0);

				}
				else if (i > Xn - L && i < Xn)
				{
					ax[i * Zn + j] = log10(1.0 / R) * 1.5 * Vp[i * Zn + j] / (L)*pow(1.0 * (i - Xn + L) / (L), 4.0);

				}
				else
				{
					ax[i * Zn + j] = 0.0;

				}
			}
		for (i = 0; i < Xn; i++)
			for (j = 0; j < Zn; j++)
			{
				if (j >= 0 && j < L)
				{
					az[i * Zn + j] = log10(1.0 / R) * 1.5 * Vp[i * Zn + j] / (L)*pow(1.0 * (L - j) / (L), 4.0);

				}
				else if (j > Zn - L && j < Zn)
				{
					az[i * Zn + j] = log10(1.0 / R) * 1.5 * Vp[i * Zn + j] / (L)*pow(1.0 * (j - Zn + L) / (L), 4.0);

				}
				else
				{
					az[i * Zn + j] = 0.0;

				}

			}
		float E1 = 0, E2 = 0, E3 = 0;

		for (l = 0; l < shot_num; l++)
		{

			cout << k + 1 << "  ShotNumber:" << l + 1 << endl;
			
			Sx = L + l * 5, Sz = L;
			Z_receive = L;

			float* d_ax, * d_az;
			hipMalloc((void**)&d_ax, size * sizeof(float));
			hipMalloc((void**)&d_az, size * sizeof(float));
			hipMemcpy(d_ax, ax, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_az, az, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_Vx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vpx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vpz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vsx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vsz, begin, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_theta, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_theta_x, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_x, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_theta_z, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_z, begin, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_duzdx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duzdz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duxdx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duxdz, begin, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_Vx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vpx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vpz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vsx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vsz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);


			hipMemcpy(d_theta_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_theta_x_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_x_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_theta_z_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_z_r, begin, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_duzdx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duzdz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duxdx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duxdz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_Vx_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vz_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vpx_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vpz_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vsx_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vsz_s, begin, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_theta_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_theta_x_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_x_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_theta_z_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_omega_z_s, begin, size * sizeof(float), hipMemcpyHostToDevice);


			hipMemcpy(d_duzdx_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duzdz_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duxdx_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_duxdz_s, begin, size * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_record_vz, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_record_vx, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_record_vz_1, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_record_vx_1, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_record_vz_2, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_record_vx_2, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);
			
			hipMemcpy(res_recordux, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(res_recorduz, begin1, NT * Xn * sizeof(float), hipMemcpyHostToDevice);

			hipMemcpy(d_Vp, Vp, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Vs, Vs, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_P, P, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_source, source, NT * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_SSS, SSS, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Grad_Vp, Grad_Vp, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Grad_Vs, Grad_Vs, size * sizeof(float), hipMemcpyHostToDevice);
			
			hipMemcpy(O_duz_xdz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_duz_zdx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_dux_zdx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_dux_xdz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_thetax, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_thetaz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_omegaz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_omegax, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_xx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_zz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_xz, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_zx, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_duz_xdz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_duz_zdx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_dux_zdx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_dux_xdz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_thetax_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_thetaz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_omegaz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(O_omegax_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_xx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_zz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_xz_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(F_zx_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			
			hipMemcpy(Vpx_now, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpz_now, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsx_now, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsz_now, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpx_past, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpz_past, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsx_past, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsz_past, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpx_now_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpz_now_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsx_now_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsz_now_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpx_past_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpz_past_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsx_past_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsz_past_r, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpx_now_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpz_now_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsx_now_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsz_now_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpx_past_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vpz_past_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsx_past_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(Vsz_past_s, begin, size * sizeof(float), hipMemcpyHostToDevice);
			
			hipMemcpy(fenzi_PP, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(fenzi_PS, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(fenmu_P, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_image_PP, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_image_PS, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_image_PP_lap, begin, size * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_image_PS_lap, begin, size * sizeof(float), hipMemcpyHostToDevice);

			printf("forward\n");
			for (t = 0; t < NT; t++)
			{

				forward_s << <dimGrid, dimBlock >> > (d_Vx, d_Vz, d_theta, d_omega, d_duxdz, d_duzdz, d_duxdx, d_duzdx,
					d_ax, d_az, dt, dx, dz, t, Sx, Sz, d_Vp, d_Vs, d_source, F_xx, F_zz, F_xz, F_zx);

				forward_u << <dimGrid, dimBlock >> > (d_Vx, d_Vz, d_Vpx, d_Vpz, d_Vsx, d_Vsz, d_theta, d_omega, d_duxdz, d_duzdz, d_duxdx, d_duzdx,
					d_az, d_ax, dt, dx, dz, Z_receive, d_record_vx, d_record_vz, t, Vpx_now, Vpz_now, Vsx_now, Vsz_now,
					Vpx_past, Vpz_past, Vsx_past, Vsz_past, O_duz_xdz, O_duz_zdx, O_dux_zdx, O_dux_xdz, O_thetax, O_thetaz, O_omegaz, O_omegax);
			

				if (t != NT - 1)
				{

					save_wavefiled << <dimGrid, dimBlock >> > (d_Vx_up, d_Vz_up, d_theta_up, d_omega_up, d_duzdx_up, d_duzdz_up, d_duxdx_up, d_duxdz_up,
						d_Vx_dn, d_Vz_dn, d_theta_dn, d_omega_dn, d_duzdx_dn, d_duzdz_dn, d_duxdx_dn, d_duxdz_dn,
						d_Vx_lf, d_Vz_lf, d_theta_lf, d_omega_lf, d_duzdx_lf, d_duzdz_lf, d_duxdx_lf, d_duxdz_lf,
						d_Vx_rt, d_Vz_rt, d_theta_rt, d_omega_rt, d_duzdx_rt, d_duzdz_rt, d_duxdx_rt, d_duxdz_rt,
						d_Vx, d_Vz, d_theta, d_omega, d_duzdx, d_duzdz, d_duxdx, d_duxdz, t);
				}
				if (t == NT - 1)
				{
					
					read_last_wavefiled2 << <dimGrid, dimBlock >> > (d_theta_s, d_omega_s, d_Vx_s, d_Vz_s, d_duzdx_s, d_duzdz_s, d_duxdz_s, d_duxdx_s,
						d_Vpx_s, d_Vpz_s, d_Vsx_s, d_Vsz_s,	d_theta, d_omega, d_Vx, d_Vz, d_duzdx, d_duzdz, d_duxdz, d_duxdx,
						d_Vpx, d_Vpz, d_Vsx, d_Vsz, t);
					

				}
				

			
			
				
			}
			remove << <dimGrid, dimBlock >> > (Sx, Sz, Z_receive, t0, dt, dh, d_Vp, d_record_vx, d_record_vz);
			hipMemcpy(record_vx, d_record_vx, Xn * NT * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(record_vz, d_record_vz, Xn * NT * sizeof(float), hipMemcpyDeviceToHost);

			


			printf("\n");


			

			printf("backward\n");

			

			hipMemcpy(d_record_vx, record_vx, Xn * NT * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_record_vz, record_vz, Xn * NT * sizeof(float), hipMemcpyHostToDevice);
		

			for (t = NT - 1; t >= 0; t--)
			{
				
				if (t == NT - 1)
				{

					load_record << <dimGrid, dimBlock >> > (Z_receive, d_Vx_r, d_Vz_r, d_record_vx, d_record_vz, t);

					rt_s_res << <dimGrid, dimBlock >> > (d_Vx_r, d_Vz_r, d_theta_r, d_omega_r, d_duxdz_r, d_duzdz_r, d_duxdx_r, d_duzdx_r,
						d_ax, d_az, dt, dx, dz, d_Vp, d_Vs, F_xx_r, F_zz_r, F_xz_r, F_zx_r);

					rt_u_res << <dimGrid, dimBlock >> > (d_Vx_r, d_Vz_r, d_Vpx_r, d_Vpz_r, d_Vsx_r, d_Vsz_r, d_theta_r, d_omega_r, d_duxdz_r, d_duzdz_r, d_duxdx_r, d_duzdx_r,
						d_az, d_ax, dt, dx, dz, t, Vpx_now_r, Vpz_now_r, Vsx_now_r, Vsz_now_r,
						Vpx_past_r, Vpz_past_r, Vsx_past_r, Vsz_past_r, O_duz_xdz_r, O_duz_zdx_r, O_dux_zdx_r, O_dux_xdz_r, O_thetax_r, O_thetaz_r, O_omegaz_r, O_omegax_r);


				}
				
				
				if (t < NT - 1)
				{
					reshot_u << <dimGrid, dimBlock >> > (d_Vx_s, d_Vz_s, d_Vpx_s, d_Vpz_s, d_Vsx_s, d_Vsz_s, d_theta_s, d_omega_s, d_duxdz_s, d_duzdz_s, d_duxdx_s, d_duzdx_s,
						dt, dx, dz, Vpx_now, Vpz_now, Vsx_now, Vsz_now, Vpx_past, Vpz_past, Vsx_past, Vsz_past);

					read_wavefiled1 << <dimGrid, dimBlock >> > (d_Vx_up, d_Vz_up, d_Vx_dn, d_Vz_dn, d_Vx_lf, d_Vz_lf, d_Vx_rt, d_Vz_rt,
						d_Vx_s, d_Vz_s, t);

					reshot_s << <dimGrid, dimBlock >> > (d_Vx_s, d_Vz_s, d_theta_s, d_omega_s, d_duxdz_s, d_duzdz_s, d_duxdx_s, d_duzdx_s,
						dt, dx, dz, d_Vp, d_Vs);

					read_wavefiled2 << <dimGrid, dimBlock >> > (d_theta_up, d_omega_up, d_duxdz_up, d_duxdx_up, d_duzdz_up, d_duzdx_up,
						d_theta_dn, d_omega_dn, d_duxdz_dn, d_duxdx_dn, d_duzdz_dn, d_duzdx_dn,
						d_theta_lf, d_omega_lf, d_duxdz_lf, d_duxdx_lf, d_duzdz_lf, d_duzdx_lf,
						d_theta_rt, d_omega_rt, d_duxdz_rt, d_duxdx_rt, d_duzdz_rt, d_duzdx_rt,
						d_theta_s, d_omega_s, d_duxdz_s, d_duxdx_s, d_duzdz_s, d_duzdx_s, t);
					
					load_record << <dimGrid, dimBlock >> > (Z_receive, d_Vx_r, d_Vz_r, d_record_vx, d_record_vz, t);

					rt_s_res << <dimGrid, dimBlock >> > (d_Vx_r, d_Vz_r, d_theta_r, d_omega_r, d_duxdz_r, d_duzdz_r, d_duxdx_r, d_duzdx_r,
						d_ax, d_az, dt, dx, dz, d_Vp, d_Vs, F_xx_r, F_zz_r, F_xz_r, F_zx_r);

					rt_u_res << <dimGrid, dimBlock >> > (d_Vx_r, d_Vz_r, d_Vpx_r, d_Vpz_r, d_Vsx_r, d_Vsz_r, d_theta_r, d_omega_r, d_duxdz_r, d_duzdz_r, d_duxdx_r, d_duzdx_r,
						d_az, d_ax, dt, dx, dz, t, Vpx_now_r, Vpz_now_r, Vsx_now_r, Vsz_now_r,
						Vpx_past_r, Vpz_past_r, Vsx_past_r, Vsz_past_r, O_duz_xdz_r, O_duz_zdx_r, O_dux_zdx_r, O_dux_xdz_r, O_thetax_r, O_thetaz_r, O_omegaz_r, O_omegax_r);

				}
				

				corr_v << <dimGrid, dimBlock >> > (fenzi_PP, fenzi_PS, fenmu_P, d_Vpx_s, d_Vpz_s, d_Vpx_r, d_Vpz_r, d_Vsx_r, d_Vsz_r);
				
				
			}
			
			
			image_fun << <dimGrid, dimBlock >> > (fenzi_PP, fenmu_P, d_image_PP);
			image_fun << <dimGrid, dimBlock >> > (fenzi_PS, fenmu_P, d_image_PS);

			hipMemcpy(image_PP, d_image_PP, Xn* Zn * sizeof(float), hipMemcpyDeviceToHost);			
			hipMemcpy(image_PS, d_image_PS, Xn* Zn * sizeof(float), hipMemcpyDeviceToHost);
			Laplace << <dimGrid, dimBlock >> > (dx, d_image_PP, d_image_PP_lap);
			Laplace << <dimGrid, dimBlock >> > (dx, d_image_PS, d_image_PS_lap);

			hipMemcpy(image_PP_lap, d_image_PP_lap, Xn* Zn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(image_PS_lap, d_image_PS_lap, Xn* Zn * sizeof(float), hipMemcpyDeviceToHost);

			
			sprintf(filename, "./image2/image_PP_lap_%d_%d_%d.dat", (Xn - 2 * L), (Zn - 2 * L), l);
			if ((fp = fopen(filename, "wb")) != NULL)
			{
				for (i = L; i < Xn - L; i++)
				{
					for (j = L; j < Zn - L; j++)
					{
						fwrite(&image_PP_lap[i * Zn + j], sizeof(float), 1, fp);
					}
				}
			}fclose(fp);
			sprintf(filename, "./image2/image_PS_lap_%d_%d_%d.dat", (Xn - 2 * L), (Zn - 2 * L), l);
			if ((fp = fopen(filename, "wb")) != NULL)
			{
				for (i = L; i < Xn - L; i++)
				{
					for (j = L; j < Zn - L; j++)
					{
						fwrite(&image_PS_lap[i * Zn + j], sizeof(float), 1, fp);
					}
				}
			}fclose(fp);
		}
		
		for (l = 0; l < shot_num; l++)
		{


			sprintf(filename, "./image2/image_PP_lap_%d_%d_%d.dat", (Xn - 2 * L), (Zn - 2 * L), l);
			if ((fp = fopen(filename, "rb")) != NULL)
			{
				float a = 0;
				for (i = L; i < Xn - L; i++)
				{
					for (j = L; j < Zn - L; j++)
					{
						fread(&image_PP_lap[i * Zn + j], sizeof(float), 1, fp);
					}
				}
			}fclose(fp);

			sprintf(filename, "./image2/image_PS_lap_%d_%d_%d.dat", (Xn - 2 * L), (Zn - 2 * L), l);
			if ((fp = fopen(filename, "rb")) != NULL)
			{
				float a = 0;
				for (i = L; i < Xn - L; i++)
				{
					for (j = L; j < Zn - L; j++)
					{
						fread(&image_PS_lap[i * Zn + j], sizeof(float), 1, fp);
					}
				}
			}fclose(fp);


			for (i = L; i < Xn - L; i++)
			{
				for (j = L; j < Zn - L; j++)
				{
					All_image_PP_lap[i * Zn + j] += image_PP_lap[i * Zn + j];
					All_image_PS_lap[i * Zn + j] += image_PS_lap[i * Zn + j];
				}
			}

		}

		
		sprintf(filename, "./image2/All_image_PP_lap.dat");
		if ((fp = fopen(filename, "wb")) != NULL)
		{
			for (i = L; i < Xn - L; i++)
			{
				for (j = L; j < Zn - L; j++)
				{
					fwrite(&All_image_PP_lap[i * Zn + j], sizeof(float), 1, fp);
				}
			}
		}fclose(fp);
		sprintf(filename, "./image2/All_image_PS_lap.dat");
		if ((fp = fopen(filename, "wb")) != NULL)
		{
			for (i = L; i < Xn - L; i++)
			{
				for (j = L; j < Zn - L; j++)
				{
					fwrite(&All_image_PS_lap[i * Zn + j], sizeof(float), 1, fp);
				}
			}
		}fclose(fp);

	}
	





















	return 0;
}
