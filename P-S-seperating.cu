#include "hip/hip_runtime.h"
﻿#define _CRT_SECURE_NO_WARNINGS
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include<iostream>
using namespace std;
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<time.h>
#include<complex>
#include<hipfft/hipfft.h>
#include "hip/hip_complex.h"//cuda核函数复数计算库文件

#define Tn 3000
#define pi 3.141592653	
#define fm 20
#define dt 0.0005
#define dx 5.0
#define dz 5.0
#define N 6
#define pml 100

#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        printf("%s\n",hipGetErrorString( err )); \
        }


__device__ float a[6] = { 1.2213365, -9.6931458e-2, 1.7447662e-2, -2.9672895e-3, 3.5900540e-4, -2.1847812e-5 };

void write_bin(float* dst, char* filename, int xnum, int tn, int L) {
	FILE* binfile;
	binfile = fopen(filename, "wb");
	for (int i = L; i < xnum - L; i++)
		for (int k = L; k < tn - L; k++)
			fwrite(&dst[i * tn + k], sizeof(float), 1, binfile);
	fclose(binfile);
}

void write_bin1(float* dst, char* filename, int xnum, int tn, int L) {
	FILE* binfile;
	binfile = fopen(filename, "wb");
	for (int i = L; i < xnum - L; i++)
		for (int k = 0; k < tn; k++)
			fwrite(&dst[i * tn + k], sizeof(float), 1, binfile);
	fclose(binfile);
}
void write_bin2(float* dst, char* filename, int xnum, int tn) {
	FILE* binfile;
	binfile = fopen(filename, "wb");
	for (int i = 0; i < xnum; i++)
		for (int k = 0; k < tn; k++)
			fwrite(&dst[i * tn + k], sizeof(float), 1, binfile);
	fclose(binfile);
}
void read_bin(float* dst, char* filename, int xnum, int tn, int L) {
	FILE* binfile;
	binfile = fopen(filename, "wb");
	for (int i = L; i < xnum - L; i++)
		for (int k = L; k < tn - L; k++)
			fread(&dst[i * tn + k], sizeof(float), 1, binfile);
	fclose(binfile);
}

void addpml(float* vp, float* vs, float* rou, float* vp_pml, float* vs_pml, float* rou_pml, int Xn, int Zn) {
	int i, j;

	for (i = 0; i < Xn; i++) {
		for (j = 0; j < Zn; j++) {
			vp_pml[i * Zn + j] = vp[i * Zn + j];
			vs_pml[i * Zn + j] = vs[i * Zn + j];
			rou_pml[i * Zn + j] = rou[i * Zn + j];
		}
	}
	//hengxiang
	for (i = 0; i < Xn; i++) {
		for (j = 0; j < pml; j++) {
			vp_pml[i * Zn + j] = vp_pml[i * Zn + pml];
			vs_pml[i * Zn + j] = vs_pml[i * Zn + pml];
			rou_pml[i * Zn + j] = rou_pml[i * Zn + pml];
		}
		for (j = Zn - pml; j < Zn; j++) {
			vp_pml[i * Zn + j] = vp_pml[i * Zn + (Zn - pml - 1)];
			vs_pml[i * Zn + j] = vs_pml[i * Zn + (Zn - pml - 1)];
			rou_pml[i * Zn + j] = rou_pml[i * Zn + (Zn - pml - 1)];
		}
	}
	//zongxiang
	for (j = 0; j < Zn; j++) {
		for (i = 0; i < pml; i++) {
			vp_pml[i * Zn + j] = vp_pml[pml * Zn + j];
			vs_pml[i * Zn + j] = vs_pml[pml * Zn + j];
			rou_pml[i * Zn + j] = rou_pml[pml * Zn + j];
		}
		for (i = Xn - pml; i < Xn; i++) {
			vp_pml[i * Zn + j] = vp_pml[(Xn - pml - 1) * Zn + j];
			vs_pml[i * Zn + j] = vs_pml[(Xn - pml - 1) * Zn + j];
			rou_pml[i * Zn + j] = rou_pml[(Xn - pml - 1) * Zn + j];
		}
	}



}
void dumpingfactor(float* vp_pml, float* ddx, float* ddz, int Xn, int Zn, int xn, int zn) {
	float R = pow((float)10, -6);//·ŽÉäÏµÊý

	int i, j, x, z, l;

	//float af = 10e-6, aa = 0.25, b = 0.75;

	float rr = 0.000001;
	for (i = 0; i < Xn; i++)
		for (j = 0; j < Zn; j++)
		{
			if (i < pml)
			{
				l = pml - i;
				ddx[i * Zn + j] = log10(1 / rr) * (5.0 * vp_pml[i * Zn + j] / (2.0 * pml)) * pow(1.0 * l / pml, 4.0);
			}

			if (i > xn + pml)
			{
				l = i - xn - pml;
				ddx[i * Zn + j] = log10(1 / rr) * (5.0 * vp_pml[i * Zn + j] / (2.0 * pml)) * pow(1.0 * l / pml, 4.0);
			}

			if (j < pml)
			{
				l = pml - j;
				ddz[i * Zn + j] = log10(1 / rr) * (5.0 * vp_pml[i * Zn + j] / (2.0 * pml)) * pow(1.0 * l / pml, 4.0);

			}

			if (j > zn + pml)
			{
				l = j - zn - pml;
				ddz[i * Zn + j] = log10(1 / rr) * (5.0 * vp_pml[i * Zn + j] / (2.0 * pml)) * pow(1.0 * l / pml, 4.0);
			}
		}


}

__global__ void forward_u(int Xn, int Zn, float* ux, float* uz, float* ddx, float* ddz, 
	float* upx_next, float* upx_now, float* upx_past, float* upz_next, float* upz_now, float* upz_past, float* usx_next, float* usx_now, float* usx_past, float* usz_next, float* usz_now, float* usz_past,	
	float* recordupx, float* recordupz, float* recordusx, float* recordusz, float* recordux, float* recorduz, 
	float* theta, float* omega, int receiver_depth, int t, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* O_duz_xdz, float* O_duz_zdx, float* O_dux_zdx, float* O_dux_xdz, float* O_thetax, float* O_thetaz, float* O_omegaz, float* O_omegax) {

	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;

	//printf("a[m] = %f\n", a[3]);
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;

		O_duz_xdz[i * Zn + j] = O_duz_xdz[i * Zn + j] + (ddz[i * Zn + j] * duz_xdz - ddz[i * Zn + j] * O_duz_xdz[i * Zn + j]) * dt;
		O_duz_zdx[i * Zn + j] = O_duz_zdx[i * Zn + j] + (ddx[i * Zn + j] * duz_zdx - ddx[i * Zn + j] * O_duz_zdx[i * Zn + j]) * dt;
		O_dux_zdx[i * Zn + j] = O_dux_zdx[i * Zn + j] + (ddx[i * Zn + j] * dux_zdx - ddx[i * Zn + j] * O_dux_zdx[i * Zn + j]) * dt;
		O_dux_xdz[i * Zn + j] = O_dux_xdz[i * Zn + j] + (ddz[i * Zn + j] * dux_xdz - ddz[i * Zn + j] * O_dux_xdz[i * Zn + j]) * dt;

		O_thetax[i * Zn + j] = O_thetax[i * Zn + j] + (ddx[i * Zn + j] * dthetadx - ddx[i * Zn + j] * O_thetax[i * Zn + j]) * dt;
		O_thetaz[i * Zn + j] = O_thetaz[i * Zn + j] + (ddz[i * Zn + j] * dthetadz - ddz[i * Zn + j] * O_thetaz[i * Zn + j]) * dt;
		O_omegaz[i * Zn + j] = O_omegaz[i * Zn + j] + (ddz[i * Zn + j] * domegadz - ddz[i * Zn + j] * O_omegaz[i * Zn + j]) * dt;
		O_omegax[i * Zn + j] = O_omegax[i * Zn + j] + (ddx[i * Zn + j] * domegadx - ddx[i * Zn + j] * O_omegax[i * Zn + j]) * dt;

		upx_next[i * Zn + j] = 2 * upx_now[i * Zn + j] - upx_past[i * Zn + j] + (dt * dt) * (dthetadx - O_thetax[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		upz_next[i * Zn + j] = 2 * upz_now[i * Zn + j] - upz_past[i * Zn + j] + (dt * dt) * (dthetadz - O_thetaz[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);
		usx_next[i * Zn + j] = 2 * usx_now[i * Zn + j] - usx_past[i * Zn + j] + (dt * dt) * (domegadz - O_omegaz[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		usz_next[i * Zn + j] = 2 * usz_now[i * Zn + j] - usz_past[i * Zn + j] + (dt * dt) * (-domegadx + O_omegax[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);

		if (t < Tn - 1)
		{
			upx_past[i * Zn + j] = upx_now[i * Zn + j];	upx_now[i * Zn + j] = upx_next[i * Zn + j];
			upz_past[i * Zn + j] = upz_now[i * Zn + j];	upz_now[i * Zn + j] = upz_next[i * Zn + j];
			usx_past[i * Zn + j] = usx_now[i * Zn + j];	usx_now[i * Zn + j] = usx_next[i * Zn + j];
			usz_past[i * Zn + j] = usz_now[i * Zn + j];	usz_now[i * Zn + j] = usz_next[i * Zn + j];
		}
		


		ux[i * Zn + j] = upx_next[i * Zn + j] + usx_next[i * Zn + j];
		uz[i * Zn + j] = upz_next[i * Zn + j] + usz_next[i * Zn + j];
	}
	if (j = receiver_depth)
	{
		recordupx[i * Tn + t] = upx_next[i * Zn + j];
		recordupz[i * Tn + t] = upz_next[i * Zn + j];
		recordusx[i * Tn + t] = usx_next[i * Zn + j];
		recordusz[i * Tn + t] = usz_next[i * Zn + j];
		recordux[i * Tn + t] = ux[i * Zn + j];
		recorduz[i * Tn + t] = uz[i * Zn + j];
		
	}

	
}

__global__ void forward_s(int Xn, int Zn, float* vp, float* vs, float* ux, float* uz, float* ddx, float* ddz, float* theta, float* omega,
	int t, int shotx, int shotz, float* source,float* F_xx, float* F_zz, float* F_xz, float* F_zx,
	float* duxdz, float* duzdz, float* duxdx, float* duzdx) {
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	
	float dux_dx = 0.0f;
	float duz_dz = 0.0f;
	float dux_dz = 0.0f;
	float duz_dx = 0.0f;
	
	float s;
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		if (i == shotx && j == shotz)
		{
			s = source[t];
		}
		else
		{
			s = 0.0;
		}

		dux_dx = (a[0] * (ux[(i + 0) * Zn + j] - ux[(i - 1) * Zn + j])
			+ a[1] * (ux[(i + 1) * Zn + j] - ux[(i - 2) * Zn + j])
			+ a[2] * (ux[(i + 2) * Zn + j] - ux[(i - 3) * Zn + j])
			+ a[3] * (ux[(i + 3) * Zn + j] - ux[(i - 4) * Zn + j])
			+ a[4] * (ux[(i + 4) * Zn + j] - ux[(i - 5) * Zn + j])
			+ a[5] * (ux[(i + 5) * Zn + j] - ux[(i - 6) * Zn + j])) / dx;

		dux_dz = (a[0] * (ux[(i)*Zn + j + 1] - ux[(i)*Zn + j - 0])
			+ a[1] * (ux[(i)*Zn + j + 2] - ux[(i)*Zn + j - 1])
			+ a[2] * (ux[(i)*Zn + j + 3] - ux[(i)*Zn + j - 2])
			+ a[3] * (ux[(i)*Zn + j + 4] - ux[(i)*Zn + j - 3])
			+ a[4] * (ux[(i)*Zn + j + 5] - ux[(i)*Zn + j - 4])
			+ a[5] * (ux[(i)*Zn + j + 6] - ux[(i)*Zn + j - 5])) / dz;

		duz_dz = (a[0] * (uz[(i)*Zn + j + 0] - uz[(i)*Zn + j - 1])
			+ a[1] * (uz[(i)*Zn + j + 1] - uz[(i)*Zn + j - 2])
			+ a[2] * (uz[(i)*Zn + j + 2] - uz[(i)*Zn + j - 3])
			+ a[3] * (uz[(i)*Zn + j + 3] - uz[(i)*Zn + j - 4])
			+ a[4] * (uz[(i)*Zn + j + 4] - uz[(i)*Zn + j - 5])
			+ a[5] * (uz[(i)*Zn + j + 5] - uz[(i)*Zn + j - 6])) / dz;

		duz_dx = (a[0] * (uz[(i + 1) * Zn + j] - uz[(i - 0) * Zn + j])
			+ a[1] * (uz[(i + 2) * Zn + j] - uz[(i - 1) * Zn + j])
			+ a[2] * (uz[(i + 3) * Zn + j] - uz[(i - 2) * Zn + j])
			+ a[3] * (uz[(i + 4) * Zn + j] - uz[(i - 3) * Zn + j])
			+ a[4] * (uz[(i + 5) * Zn + j] - uz[(i - 4) * Zn + j])
			+ a[5] * (uz[(i + 6) * Zn + j] - uz[(i - 5) * Zn + j])) / dx;

		F_xx[i * Zn + j] = F_xx[i * Zn + j] + (ddx[i * Zn + j] * dux_dx - ddx[i * Zn + j] * F_xx[i * Zn + j]) * dt;
		F_zz[i * Zn + j] = F_zz[i * Zn + j] + (ddz[i * Zn + j] * duz_dz - ddz[i * Zn + j] * F_zz[i * Zn + j]) * dt;
		F_xz[i * Zn + j] = F_xz[i * Zn + j] + (ddz[i * Zn + j] * dux_dz - ddz[i * Zn + j] * F_xz[i * Zn + j]) * dt;
		F_zx[i * Zn + j] = F_zx[i * Zn + j] + (ddx[i * Zn + j] * duz_dx - ddx[i * Zn + j] * F_zx[i * Zn + j]) * dt;

		theta[i * Zn + j] = vp[i * Zn + j] * vp[i * Zn + j] * (dux_dx + duz_dz - F_xx[i * Zn + j] - F_zz[i * Zn + j]);
		omega[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dz - duz_dx - F_xz[i * Zn + j] + F_zx[i * Zn + j]);
		duzdx[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (duz_dx - F_zx[i * Zn + j]);
		duzdz[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (duz_dz - F_zz[i * Zn + j]);
		duxdz[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dz - F_xz[i * Zn + j]);
		duxdx[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dx - F_xx[i * Zn + j]);

		theta[i * Zn + j] = theta[i * Zn + j] + s;

	}

	
}
__global__ void save_wavefiled(int Xn,int Zn,int L,float* d_ux_up, float* d_uz_up, float* d_theta_up, float* d_omega_up, float* d_duzdx_up, float* d_duzdz_up, float* d_duxdx_up, float* d_duxdz_up,
	float* d_ux_dn, float* d_uz_dn, float* d_theta_dn, float* d_omega_dn, float* d_duzdx_dn, float* d_duzdz_dn, float* d_duxdx_dn, float* d_duxdz_dn,
	float* d_ux_lf, float* d_uz_lf, float* d_theta_lf, float* d_omega_lf, float* d_duzdx_lf, float* d_duzdz_lf, float* d_duxdx_lf, float* d_duxdz_lf,
	float* d_ux_rt, float* d_uz_rt, float* d_theta_rt, float* d_omega_rt, float* d_duzdx_rt, float* d_duzdz_rt, float* d_duxdx_rt, float* d_duxdz_rt,
	float* d_ux, float* d_uz, float* d_theta, float* d_omega, float* d_duzdx, float* d_duzdz, float* d_duxdx, float* d_duxdz, int t,
	float* d_upx_next_up, float* d_upz_next_up, float* d_usx_next_up, float* d_usz_next_up, float* d_upx_now_up, float* d_upz_now_up, float* d_usx_now_up, float* d_usz_now_up,
	float* d_upx_next_dn, float* d_upz_next_dn, float* d_usx_next_dn, float* d_usz_next_dn, float* d_upx_now_dn, float* d_upz_now_dn, float* d_usx_now_dn, float* d_usz_now_dn,
	float* d_upx_next_lf, float* d_upz_next_lf, float* d_usx_next_lf, float* d_usz_next_lf, float* d_upx_now_lf, float* d_upz_now_lf, float* d_usx_now_lf, float* d_usz_now_lf,
	float* d_upx_next_rt, float* d_upz_next_rt, float* d_usx_next_rt, float* d_usz_next_rt, float* d_upx_now_rt, float* d_upz_now_rt, float* d_usx_now_rt, float* d_usz_now_rt,
	float* d_upx_next, float* d_upz_next, float* d_usx_next, float* d_usz_next, float* d_upx_now, float* d_upz_now, float* d_usx_now, float* d_usz_now)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= 0 && i < Xn && j >= L - N && j < L + N)
	{
		d_ux_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_ux[i * Zn + j];
		d_uz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_uz[i * Zn + j];
		d_theta_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_theta[i * Zn + j];
		d_omega_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_omega[i * Zn + j];
		d_duzdx_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duzdx[i * Zn + j];
		d_duzdz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duzdz[i * Zn + j];
		d_duxdx_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duxdx[i * Zn + j];
		d_duxdz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_duxdz[i * Zn + j];
		d_upx_next_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_upx_next[i * Zn + j];
		d_upz_next_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_upz_next[i * Zn + j];
		d_usx_next_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_usx_next[i * Zn + j];
		d_usz_next_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_usz_next[i * Zn + j];
		d_upx_now_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_upx_now[i * Zn + j];
		d_upz_now_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_upz_now[i * Zn + j];
		d_usx_now_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_usx_now[i * Zn + j];
		d_usz_now_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)] = d_usz_now[i * Zn + j];
	}
	if (i >= 0 && i < Xn && j >= Zn - L - N && j < Zn - L + N)
	{
		d_ux_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_ux[i * Zn + j];
		d_uz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_uz[i * Zn + j];
		d_theta_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_theta[i * Zn + j];
		d_omega_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_omega[i * Zn + j];
		d_duzdx_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duzdx[i * Zn + j];
		d_duzdz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duzdz[i * Zn + j];
		d_duxdx_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duxdx[i * Zn + j];
		d_duxdz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_duxdz[i * Zn + j];
		d_upx_next_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_upx_next[i * Zn + j];
		d_upz_next_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_upz_next[i * Zn + j];
		d_usx_next_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_usx_next[i * Zn + j];
		d_usz_next_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_usz_next[i * Zn + j];
		d_upx_now_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_upx_now[i * Zn + j];
		d_upz_now_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_upz_now[i * Zn + j];
		d_usx_now_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_usx_now[i * Zn + j];
		d_usz_now_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)] = d_usz_now[i * Zn + j];
	}
	if (i >= L - N && i < L + N && j >= 0 && j < Zn)
	{
		d_ux_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_ux[i * Zn + j];
		d_uz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_uz[i * Zn + j];
		d_theta_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_theta[i * Zn + j];
		d_omega_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_omega[i * Zn + j];
		d_duzdx_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duzdx[i * Zn + j];
		d_duzdz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duzdz[i * Zn + j];
		d_duxdx_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duxdx[i * Zn + j];
		d_duxdz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_duxdz[i * Zn + j];
		d_upx_next_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_upx_next[i * Zn + j];
		d_upz_next_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_upz_next[i * Zn + j];
		d_usx_next_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_usx_next[i * Zn + j];
		d_usz_next_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_usz_next[i * Zn + j];
		d_upx_now_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_upx_now[i * Zn + j];
		d_upz_now_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_upz_now[i * Zn + j];
		d_usx_now_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_usx_now[i * Zn + j];
		d_usz_now_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j] = d_usz_now[i * Zn + j];

	}
	if (i >= Xn - L - N && i < Xn - L + N && j >= 0 && j < Zn)
	{
		d_ux_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_ux[i * Zn + j];
		d_uz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_uz[i * Zn + j];
		d_theta_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_theta[i * Zn + j];
		d_omega_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_omega[i * Zn + j];
		d_duzdx_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duzdx[i * Zn + j];
		d_duzdz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duzdz[i * Zn + j];
		d_duxdx_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duxdx[i * Zn + j];
		d_duxdz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_duxdz[i * Zn + j];
		d_upx_next_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_upx_next[i * Zn + j];
		d_upz_next_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_upz_next[i * Zn + j];
		d_usx_next_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_usx_next[i * Zn + j];
		d_usz_next_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_usz_next[i * Zn + j];
		d_upx_now_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_upx_now[i * Zn + j];
		d_upz_now_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_upz_now[i * Zn + j];
		d_usx_now_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_usx_now[i * Zn + j];
		d_usz_now_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j] = d_usz_now[i * Zn + j];

	}

}
__global__ void read_last_wavefiled(int Xn, int Zn, int L,float* d_theta_s, float* d_omega_s, float* d_ux_s, float* d_uz_s, float* d_duzdx_s, float* d_duzdz_s, float* d_duxdz_s, float* d_duxdx_s,
	float* d_upx_next_s, float* d_upz_next_s, float* d_usx_next_s, float* d_usz_next_s,
	float* d_theta, float* d_omega, float* d_ux, float* d_uz, float* d_duzdx, float* d_duzdz, float* d_duxdz, float* d_duxdx,
	float* d_upx_next, float* d_upz_next, float* d_usx_next, float* d_usz_next, 
	float* d_upx_now, float* d_upz_now, float* d_usx_now, float* d_usz_now,
	float* d_upx_now_s, float* d_upz_now_s, float* d_usx_now_s, float* d_usz_now_s)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L - N && i < Xn - L + N && j >= L - N && j < Zn - L + N)
	{
		d_theta_s[i * Zn + j] = d_theta[i * Zn + j];
		d_omega_s[i * Zn + j] = d_omega[i * Zn + j];
		d_ux_s[i * Zn + j] = d_ux[i * Zn + j];
		d_uz_s[i * Zn + j] = d_uz[i * Zn + j];

		d_upx_next_s[i * Zn + j] = d_upx_next[i * Zn + j];
		d_upz_next_s[i * Zn + j] = d_upz_next[i * Zn + j];
		d_usx_next_s[i * Zn + j] = d_usx_next[i * Zn + j];
		d_usz_next_s[i * Zn + j] = d_usz_next[i * Zn + j];
		d_duzdx_s[i * Zn + j] = d_duzdx[i * Zn + j];
		d_duzdz_s[i * Zn + j] = d_duzdz[i * Zn + j];
		d_duxdx_s[i * Zn + j] = d_duxdx[i * Zn + j];
		d_duxdz_s[i * Zn + j] = d_duxdz[i * Zn + j];
		d_upx_now_s[i * Zn + j] = d_upx_now[i * Zn + j];
		d_upz_now_s[i * Zn + j] = d_upz_now[i * Zn + j];
		d_usx_now_s[i * Zn + j] = d_usx_now[i * Zn + j];
		d_usz_now_s[i * Zn + j] = d_usz_now[i * Zn + j];
	}
}
__global__ void read_wavefiled_NT1(int Xn, int Zn, int L, 	float* d_upx_past_s, float* d_upz_past_s, float* d_usx_past_s, float* d_usz_past_s,
	float* d_upx_next, float* d_upz_next, float* d_usx_next, float* d_usz_next)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L - N && i < Xn - L + N && j >= L - N && j < Zn - L + N)
	{
		
		d_upx_past_s[i * Zn + j] = d_upx_next[i * Zn + j];
		d_upz_past_s[i * Zn + j] = d_upz_next[i * Zn + j];
		d_usx_past_s[i * Zn + j] = d_usx_next[i * Zn + j];
		d_usz_past_s[i * Zn + j] = d_usz_next[i * Zn + j];
		
	}
}
__global__ void read_wavefiled_NT2(int Xn, int Zn, int L, 
	float* d_upx_past_s, float* d_upz_past_s, float* d_usx_past_s, float* d_usz_past_s,
	float* d_upx_now, float* d_upz_now, float* d_usx_now, float* d_usz_now)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L - N && i < Xn - L + N && j >= L - N && j < Zn - L + N)
	{		
		d_upx_past_s[i * Zn + j] = d_upx_now[i * Zn + j];
		d_upz_past_s[i * Zn + j] = d_upz_now[i * Zn + j];
		d_usx_past_s[i * Zn + j] = d_usx_now[i * Zn + j];
		d_usz_past_s[i * Zn + j] = d_usz_now[i * Zn + j];
	}
}
__global__ void read_wavefiled_NT3(int Xn, int Zn, int L, 
	float* d_upx_next_s, float* d_upz_next_s, float* d_usx_next_s, float* d_usz_next_s,	
	float* d_upx_next, float* d_upz_next, float* d_usx_next, float* d_usz_next,
	float* d_upx_now, float* d_upz_now, float* d_usx_now, float* d_usz_now,
	float* d_upx_now_s, float* d_upz_now_s, float* d_usx_now_s, float* d_usz_now_s, 
	float* d_upx_past, float* d_upz_past, float* d_usx_past, float* d_usz_past,
	float* d_upx_past_s, float* d_upz_past_s, float* d_usx_past_s, float* d_usz_past_s)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L - N && i < Xn - L + N && j >= L - N && j < Zn - L + N)
	{
		
		d_upx_next_s[i * Zn + j] = d_upx_next[i * Zn + j];
		d_upz_next_s[i * Zn + j] = d_upz_next[i * Zn + j];
		d_usx_next_s[i * Zn + j] = d_usx_next[i * Zn + j];
		d_usz_next_s[i * Zn + j] = d_usz_next[i * Zn + j];
	
		d_upx_now_s[i * Zn + j] = d_upx_now[i * Zn + j];
		d_upz_now_s[i * Zn + j] = d_upz_now[i * Zn + j];
		d_usx_now_s[i * Zn + j] = d_usx_now[i * Zn + j];
		d_usz_now_s[i * Zn + j] = d_usz_now[i * Zn + j];

		d_upx_past_s[i * Zn + j] = d_upx_past[i * Zn + j];
		d_upz_past_s[i * Zn + j] = d_upz_past[i * Zn + j];
		d_usx_past_s[i * Zn + j] = d_usx_past[i * Zn + j];
		d_usz_past_s[i * Zn + j] = d_usz_past[i * Zn + j];
	}
}

__global__ void remove(int Xn, int Zn, int Sx, int Sz, int Z_receive, int t0, float dh, float* v, float* record_vx, float* record_vz)
{
	int  t;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	float distance;
	if (i >= 0 && i < Xn && j == Z_receive)
	{
		for (t = 0; t < Tn; t++)
		{
			distance = sqrtf(float(abs(Sx - i) * abs(Sx - i) + abs(Z_receive - Sz) * abs(Z_receive - Sz)));
			if (t < (2 * t0 + distance * dh * 1.0 / (dt * v[Sx * Zn + j])))
			{
				record_vx[i * Tn + t] = 0;
				record_vz[i * Tn + t] = 0;

			}


		}

	}
}
__global__ void load_record(int Xn, int Zn,int L, int reciver, float* vx, float* vz, float* record_vx, float* record_vz, int t)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= L && i < Xn - L && j == reciver)
	{
		vx[i * Zn + j] = record_vx[i * Tn + t];
		vz[i * Zn + j] = record_vz[i * Tn + t];

	}
}
__global__ void reshot_u(int Xn, int Zn, int L, float* ux, float* uz, float* upx_next, float* upz_next, float* usx_next, float* usz_next, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* upx_now, float* upz_now, float* usx_now, float* usz_now,
	float* upx_past, float* upz_past, float* usx_past, float* usz_past) {
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	
	if (i >= L+N && i < Xn - L-N && j >= L+N && j < Zn - L-N)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;

		upx_next[i * Zn + j] = upx_now[i * Zn + j]; upx_now[i * Zn + j] = upx_past[i * Zn + j];
		upz_next[i * Zn + j] = upz_now[i * Zn + j]; upz_now[i * Zn + j] = upz_past[i * Zn + j];
		usx_next[i * Zn + j] = usx_now[i * Zn + j]; usx_now[i * Zn + j] = usx_past[i * Zn + j];
		usz_next[i * Zn + j] = usz_now[i * Zn + j]; usz_now[i * Zn + j] = usz_past[i * Zn + j];



		upx_past[i * Zn + j] = 2 * upx_now[i * Zn + j] - upx_next[i * Zn + j] + (dt * dt) * (dthetadx + duz_xdz - duz_zdx);
		upz_past[i * Zn + j] = 2 * upz_now[i * Zn + j] - upz_next[i * Zn + j] + (dt * dt) * (dthetadz + dux_zdx - dux_xdz);
		usx_past[i * Zn + j] = 2 * usx_now[i * Zn + j] - usx_next[i * Zn + j] + (dt * dt) * (domegadz + duz_xdz - duz_zdx);
		usz_past[i * Zn + j] = 2 * usz_now[i * Zn + j] - usz_next[i * Zn + j] + (dt * dt) * (-domegadx + dux_zdx - dux_xdz);



		ux[i * Zn + j] = upx_past[i * Zn + j] + usx_past[i * Zn + j];
		uz[i * Zn + j] = upz_past[i * Zn + j] + usz_past[i * Zn + j];

	}

}


__global__ void reshot_s(int Xn, int Zn, int L, float* ux, float* uz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* vp, float* vs) {

	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_dx = 0.0f;
	float duz_dz = 0.0f;
	float dux_dz = 0.0f;
	float duz_dx = 0.0f;
	
	float s;
	if (i >= L + N && i < Xn - L - N && j >= L + N && j < Zn - L - N)
	{


		dux_dx = (a[0] * (ux[(i + 0) * Zn + j] - ux[(i - 1) * Zn + j])
			+ a[1] * (ux[(i + 1) * Zn + j] - ux[(i - 2) * Zn + j])
			+ a[2] * (ux[(i + 2) * Zn + j] - ux[(i - 3) * Zn + j])
			+ a[3] * (ux[(i + 3) * Zn + j] - ux[(i - 4) * Zn + j])
			+ a[4] * (ux[(i + 4) * Zn + j] - ux[(i - 5) * Zn + j])
			+ a[5] * (ux[(i + 5) * Zn + j] - ux[(i - 6) * Zn + j])) / dx;

		dux_dz = (a[0] * (ux[(i)*Zn + j + 1] - ux[(i)*Zn + j - 0])
			+ a[1] * (ux[(i)*Zn + j + 2] - ux[(i)*Zn + j - 1])
			+ a[2] * (ux[(i)*Zn + j + 3] - ux[(i)*Zn + j - 2])
			+ a[3] * (ux[(i)*Zn + j + 4] - ux[(i)*Zn + j - 3])
			+ a[4] * (ux[(i)*Zn + j + 5] - ux[(i)*Zn + j - 4])
			+ a[5] * (ux[(i)*Zn + j + 6] - ux[(i)*Zn + j - 5])) / dz;

		duz_dz = (a[0] * (uz[(i)*Zn + j + 0] - uz[(i)*Zn + j - 1])
			+ a[1] * (uz[(i)*Zn + j + 1] - uz[(i)*Zn + j - 2])
			+ a[2] * (uz[(i)*Zn + j + 2] - uz[(i)*Zn + j - 3])
			+ a[3] * (uz[(i)*Zn + j + 3] - uz[(i)*Zn + j - 4])
			+ a[4] * (uz[(i)*Zn + j + 4] - uz[(i)*Zn + j - 5])
			+ a[5] * (uz[(i)*Zn + j + 5] - uz[(i)*Zn + j - 6])) / dz;

		duz_dx = (a[0] * (uz[(i + 1) * Zn + j] - uz[(i - 0) * Zn + j])
			+ a[1] * (uz[(i + 2) * Zn + j] - uz[(i - 1) * Zn + j])
			+ a[2] * (uz[(i + 3) * Zn + j] - uz[(i - 2) * Zn + j])
			+ a[3] * (uz[(i + 4) * Zn + j] - uz[(i - 3) * Zn + j])
			+ a[4] * (uz[(i + 5) * Zn + j] - uz[(i - 4) * Zn + j])
			+ a[5] * (uz[(i + 6) * Zn + j] - uz[(i - 5) * Zn + j])) / dx;



		theta[i * Zn + j] = vp[i * Zn + j] * vp[i * Zn + j] * (dux_dx + duz_dz);
		omega[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dz - duz_dx);
		duzdx[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (duz_dx);
		duzdz[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (duz_dz);
		duxdz[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dz);
		duxdx[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dx);



	}

}
__global__ void read_wavefiled1(int t,int L, int Xn, int Zn,
	float* d_ux_up, float* d_uz_up,float* d_ux_dn, float* d_uz_dn,float* d_ux_lf, float* d_uz_lf, float* d_ux_rt, float* d_uz_rt, float* d_ux, float* d_uz)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= 0 && i < Xn && j >= L - N && j < L + N)
	{		
		d_ux[i * Zn + j] = d_ux_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_uz[i * Zn + j] = d_uz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
	
	}
	if (i >= 0 && i < Xn && j >= Zn - L - N && j < Zn - L + N)
	{
		d_ux[i * Zn + j] = d_ux_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_uz[i * Zn + j] = d_uz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		
	}
	if (i >= L - N && i < L + N && j >= 0 && j < Zn)
	{		
		d_ux[i * Zn + j] = d_ux_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_uz[i * Zn + j] = d_uz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		

	}
	if (i >= Xn - L - N && i < Xn - L + N && j >= 0 && j < Zn)
	{		
		d_ux[i * Zn + j] = d_ux_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_uz[i * Zn + j] = d_uz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		

	}
	

}
__global__ void read_wavefiled2(float* d_theta_up, float* d_omega_up, float* d_duzdx_up, float* d_duzdz_up, float* d_duxdx_up, float* d_duxdz_up,
	float* d_theta_dn, float* d_omega_dn, float* d_duzdx_dn, float* d_duzdz_dn, float* d_duxdx_dn, float* d_duxdz_dn,
	float* d_theta_lf, float* d_omega_lf, float* d_duzdx_lf, float* d_duzdz_lf, float* d_duxdx_lf, float* d_duxdz_lf,
	float* d_theta_rt, float* d_omega_rt, float* d_duzdx_rt, float* d_duzdz_rt, float* d_duxdx_rt, float* d_duxdz_rt,
	float* d_theta, float* d_omega, float* d_duzdx, float* d_duzdz, float* d_duxdx, float* d_duxdz, int t, int L, int Xn, int Zn)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= 0 && i < Xn && j >= L - N && j < L + N)
	{
		d_theta[i * Zn + j] = d_theta_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_omega[i * Zn + j] = d_omega_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duzdx[i * Zn + j] = d_duzdx_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duzdz[i * Zn + j] = d_duzdz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duxdx[i * Zn + j] = d_duxdx_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		d_duxdz[i * Zn + j] = d_duxdz_up[t * (2 * N * Xn) + i * (2 * N) + j - (L - 2 * N)];
		
	}
	if (i >= 0 && i < Xn && j >= Zn - L - N && j < Zn - L + N)
	{
		d_theta[i * Zn + j] = d_theta_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_omega[i * Zn + j] = d_omega_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duzdx[i * Zn + j] = d_duzdx_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duzdz[i * Zn + j] = d_duzdz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duxdx[i * Zn + j] = d_duxdx_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		d_duxdz[i * Zn + j] = d_duxdz_dn[t * (2 * N * Xn) + i * (2 * N) + j - (Zn - L)];
		
	}
	if (i >= L - N && i < L + N && j >= 0 && j < Zn)
	{
		d_theta[i * Zn + j] = d_theta_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_omega[i * Zn + j] = d_omega_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duzdx[i * Zn + j] = d_duzdx_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duzdz[i * Zn + j] = d_duzdz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duxdx[i * Zn + j] = d_duxdx_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		d_duxdz[i * Zn + j] = d_duxdz_lf[t * (2 * N * Zn) + (i - L + 2 * N) * Zn + j];
		

	}
	if (i >= Xn - L - N && i < Xn - L + N && j >= 0 && j < Zn)
	{
		d_theta[i * Zn + j] = d_theta_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_omega[i * Zn + j] = d_omega_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duzdx[i * Zn + j] = d_duzdx_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duzdz[i * Zn + j]= d_duzdz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duxdx[i * Zn + j] = d_duxdx_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		d_duxdz[i * Zn + j] = d_duxdz_rt[t * (2 * N * Zn) + (i - Xn + L) * Zn + j];
		

	}


}
__global__ void rt_s_res(int Xn, int Zn, float* ux, float* uz, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* ddx, float* ddz, float* vp, float* vs,
	float* F_xx, float* F_zz, float* F_xz, float* F_zx) {

	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_dx = 0.0f;
	float duz_dz = 0.0f;
	float dux_dz = 0.0f;
	float duz_dx = 0.0f;
	
	float s;
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{

		dux_dx = (a[0] * (ux[(i + 0) * Zn + j] - ux[(i - 1) * Zn + j])
			+ a[1] * (ux[(i + 1) * Zn + j] - ux[(i - 2) * Zn + j])
			+ a[2] * (ux[(i + 2) * Zn + j] - ux[(i - 3) * Zn + j])
			+ a[3] * (ux[(i + 3) * Zn + j] - ux[(i - 4) * Zn + j])
			+ a[4] * (ux[(i + 4) * Zn + j] - ux[(i - 5) * Zn + j])
			+ a[5] * (ux[(i + 5) * Zn + j] - ux[(i - 6) * Zn + j])) / dx;

		dux_dz = (a[0] * (ux[(i)*Zn + j + 1] - ux[(i)*Zn + j - 0])
			+ a[1] * (ux[(i)*Zn + j + 2] - ux[(i)*Zn + j - 1])
			+ a[2] * (ux[(i)*Zn + j + 3] - ux[(i)*Zn + j - 2])
			+ a[3] * (ux[(i)*Zn + j + 4] - ux[(i)*Zn + j - 3])
			+ a[4] * (ux[(i)*Zn + j + 5] - ux[(i)*Zn + j - 4])
			+ a[5] * (ux[(i)*Zn + j + 6] - ux[(i)*Zn + j - 5])) / dz;

		duz_dz = (a[0] * (uz[(i)*Zn + j + 0] - uz[(i)*Zn + j - 1])
			+ a[1] * (uz[(i)*Zn + j + 1] - uz[(i)*Zn + j - 2])
			+ a[2] * (uz[(i)*Zn + j + 2] - uz[(i)*Zn + j - 3])
			+ a[3] * (uz[(i)*Zn + j + 3] - uz[(i)*Zn + j - 4])
			+ a[4] * (uz[(i)*Zn + j + 4] - uz[(i)*Zn + j - 5])
			+ a[5] * (uz[(i)*Zn + j + 5] - uz[(i)*Zn + j - 6])) / dz;

		duz_dx = (a[0] * (uz[(i + 1) * Zn + j] - uz[(i - 0) * Zn + j])
			+ a[1] * (uz[(i + 2) * Zn + j] - uz[(i - 1) * Zn + j])
			+ a[2] * (uz[(i + 3) * Zn + j] - uz[(i - 2) * Zn + j])
			+ a[3] * (uz[(i + 4) * Zn + j] - uz[(i - 3) * Zn + j])
			+ a[4] * (uz[(i + 5) * Zn + j] - uz[(i - 4) * Zn + j])
			+ a[5] * (uz[(i + 6) * Zn + j] - uz[(i - 5) * Zn + j])) / dx;

		F_xx[i * Zn + j] = F_xx[i * Zn + j] + (ddx[i * Zn + j] * dux_dx - ddx[i * Zn + j] * F_xx[i * Zn + j]) * dt;
		F_zz[i * Zn + j] = F_zz[i * Zn + j] + (ddz[i * Zn + j] * duz_dz - ddz[i * Zn + j] * F_zz[i * Zn + j]) * dt;
		F_xz[i * Zn + j] = F_xz[i * Zn + j] + (ddz[i * Zn + j] * dux_dz - ddz[i * Zn + j] * F_xz[i * Zn + j]) * dt;
		F_zx[i * Zn + j] = F_zx[i * Zn + j] + (ddx[i * Zn + j] * duz_dx - ddx[i * Zn + j] * F_zx[i * Zn + j]) * dt;

		theta[i * Zn + j] = vp[i * Zn + j] * vp[i * Zn + j] * (dux_dx + duz_dz - F_xx[i * Zn + j] - F_zz[i * Zn + j]);
		omega[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dz - duz_dx - F_xz[i * Zn + j] + F_zx[i * Zn + j]);
		duzdx[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (duz_dx - F_zx[i * Zn + j]);
		duzdz[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (duz_dz - F_zz[i * Zn + j]);
		duxdz[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dz - F_xz[i * Zn + j]);
		duxdx[i * Zn + j] = vs[i * Zn + j] * vs[i * Zn + j] * (dux_dx - F_xx[i * Zn + j]);


	}

}
__global__ void rt_u_res(int Xn, int Zn, float* ux, float* uz, float* upx_next, float* upz_next, float* usx_next, float* usz_next, float* theta, float* omega, float* duxdz, float* duzdz, float* duxdx, float* duzdx,
	float* ddz, float* ddx, float* upx_now, float* upz_now, float* usx_now, float* usz_now,
	float* upx_past, float* upz_past, float* usx_past, float* usz_past, float* O_duz_xdz, float* O_duz_zdx, float* O_dux_zdx, float* O_dux_xdz, float* O_thetax, float* O_thetaz, float* O_omegaz, float* O_omegax) {


	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	float dux_xdz = 0.0f;
	float duz_zdx = 0.0f;
	float duz_xdz = 0.0f;
	float dux_zdx = 0.0f;
	float dthetadx = 0.0f;
	float dthetadz = 0.0f;
	float domegadx = 0.0f;
	float domegadz = 0.0f;
	
	if (i >= N && i < Xn - N && j >= N && j < Zn - N)
	{
		dthetadx = (a[0] * (theta[(i + 1) * Zn + j] - theta[(i - 0) * Zn + j])
			+ a[1] * (theta[(i + 2) * Zn + j] - theta[(i - 1) * Zn + j])
			+ a[2] * (theta[(i + 3) * Zn + j] - theta[(i - 2) * Zn + j])
			+ a[3] * (theta[(i + 4) * Zn + j] - theta[(i - 3) * Zn + j])
			+ a[4] * (theta[(i + 5) * Zn + j] - theta[(i - 4) * Zn + j])
			+ a[5] * (theta[(i + 6) * Zn + j] - theta[(i - 5) * Zn + j])) / dx;

		dthetadz = (a[0] * (theta[(i)*Zn + j + 1] - theta[(i)*Zn + j - 0])
			+ a[1] * (theta[(i)*Zn + j + 2] - theta[(i)*Zn + j - 1])
			+ a[2] * (theta[(i)*Zn + j + 3] - theta[(i)*Zn + j - 2])
			+ a[3] * (theta[(i)*Zn + j + 4] - theta[(i)*Zn + j - 3])
			+ a[4] * (theta[(i)*Zn + j + 5] - theta[(i)*Zn + j - 4])
			+ a[5] * (theta[(i)*Zn + j + 6] - theta[(i)*Zn + j - 5])) / dz;

		domegadx = (a[0] * (omega[(i + 0) * Zn + j] - omega[(i - 1) * Zn + j])
			+ a[1] * (omega[(i + 1) * Zn + j] - omega[(i - 2) * Zn + j])
			+ a[2] * (omega[(i + 2) * Zn + j] - omega[(i - 3) * Zn + j])
			+ a[3] * (omega[(i + 3) * Zn + j] - omega[(i - 4) * Zn + j])
			+ a[4] * (omega[(i + 4) * Zn + j] - omega[(i - 5) * Zn + j])
			+ a[5] * (omega[(i + 5) * Zn + j] - omega[(i - 6) * Zn + j])) / dx;

		domegadz = (a[0] * (omega[(i)*Zn + j + 0] - omega[(i)*Zn + j - 1])
			+ a[1] * (omega[(i)*Zn + j + 1] - omega[(i)*Zn + j - 2])
			+ a[2] * (omega[(i)*Zn + j + 2] - omega[(i)*Zn + j - 3])
			+ a[3] * (omega[(i)*Zn + j + 3] - omega[(i)*Zn + j - 4])
			+ a[4] * (omega[(i)*Zn + j + 4] - omega[(i)*Zn + j - 5])
			+ a[5] * (omega[(i)*Zn + j + 5] - omega[(i)*Zn + j - 6])) / dz;


		dux_xdz = (a[0] * (duxdx[(i)*Zn + j + 1] - duxdx[(i)*Zn + j - 0])
			+ a[1] * (duxdx[(i)*Zn + j + 2] - duxdx[(i)*Zn + j - 1])
			+ a[2] * (duxdx[(i)*Zn + j + 3] - duxdx[(i)*Zn + j - 2])
			+ a[3] * (duxdx[(i)*Zn + j + 4] - duxdx[(i)*Zn + j - 3])
			+ a[4] * (duxdx[(i)*Zn + j + 5] - duxdx[(i)*Zn + j - 4])
			+ a[5] * (duxdx[(i)*Zn + j + 6] - duxdx[(i)*Zn + j - 5])) / dz;

		dux_zdx = (a[0] * (duxdz[(i + 0) * Zn + j] - duxdz[(i - 1) * Zn + j])
			+ a[1] * (duxdz[(i + 1) * Zn + j] - duxdz[(i - 2) * Zn + j])
			+ a[2] * (duxdz[(i + 2) * Zn + j] - duxdz[(i - 3) * Zn + j])
			+ a[3] * (duxdz[(i + 3) * Zn + j] - duxdz[(i - 4) * Zn + j])
			+ a[4] * (duxdz[(i + 4) * Zn + j] - duxdz[(i - 5) * Zn + j])
			+ a[5] * (duxdz[(i + 5) * Zn + j] - duxdz[(i - 6) * Zn + j])) / dx;

		duz_xdz = (a[0] * (duzdx[(i)*Zn + j + 0] - duzdx[(i)*Zn + j - 1])
			+ a[1] * (duzdx[(i)*Zn + j + 1] - duzdx[(i)*Zn + j - 2])
			+ a[2] * (duzdx[(i)*Zn + j + 2] - duzdx[(i)*Zn + j - 3])
			+ a[3] * (duzdx[(i)*Zn + j + 3] - duzdx[(i)*Zn + j - 4])
			+ a[4] * (duzdx[(i)*Zn + j + 4] - duzdx[(i)*Zn + j - 5])
			+ a[5] * (duzdx[(i)*Zn + j + 5] - duzdx[(i)*Zn + j - 6])) / dz;

		duz_zdx = (a[0] * (duzdz[(i + 1) * Zn + j] - duzdz[(i - 0) * Zn + j])
			+ a[1] * (duzdz[(i + 2) * Zn + j] - duzdz[(i - 1) * Zn + j])
			+ a[2] * (duzdz[(i + 3) * Zn + j] - duzdz[(i - 2) * Zn + j])
			+ a[3] * (duzdz[(i + 4) * Zn + j] - duzdz[(i - 3) * Zn + j])
			+ a[4] * (duzdz[(i + 5) * Zn + j] - duzdz[(i - 4) * Zn + j])
			+ a[5] * (duzdz[(i + 6) * Zn + j] - duzdz[(i - 5) * Zn + j])) / dx;

		O_duz_xdz[i * Zn + j] = O_duz_xdz[i * Zn + j] + (ddz[i * Zn + j] * duz_xdz - ddz[i * Zn + j] * O_duz_xdz[i * Zn + j]) * dt;
		O_duz_zdx[i * Zn + j] = O_duz_zdx[i * Zn + j] + (ddx[i * Zn + j] * duz_zdx - ddx[i * Zn + j] * O_duz_zdx[i * Zn + j]) * dt;
		O_dux_zdx[i * Zn + j] = O_dux_zdx[i * Zn + j] + (ddx[i * Zn + j] * dux_zdx - ddx[i * Zn + j] * O_dux_zdx[i * Zn + j]) * dt;
		O_dux_xdz[i * Zn + j] = O_dux_xdz[i * Zn + j] + (ddz[i * Zn + j] * dux_xdz - ddz[i * Zn + j] * O_dux_xdz[i * Zn + j]) * dt;

		O_thetax[i * Zn + j] = O_thetax[i * Zn + j] + (ddx[i * Zn + j] * dthetadx - ddx[i * Zn + j] * O_thetax[i * Zn + j]) * dt;
		O_thetaz[i * Zn + j] = O_thetaz[i * Zn + j] + (ddz[i * Zn + j] * dthetadz - ddz[i * Zn + j] * O_thetaz[i * Zn + j]) * dt;
		O_omegaz[i * Zn + j] = O_omegaz[i * Zn + j] + (ddz[i * Zn + j] * domegadz - ddz[i * Zn + j] * O_omegaz[i * Zn + j]) * dt;
		O_omegax[i * Zn + j] = O_omegax[i * Zn + j] + (ddx[i * Zn + j] * domegadx - ddx[i * Zn + j] * O_omegax[i * Zn + j]) * dt;

		upx_next[i * Zn + j] = 2 * upx_now[i * Zn + j] - upx_past[i * Zn + j] + (dt * dt) * (dthetadx - O_thetax[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		upz_next[i * Zn + j] = 2 * upz_now[i * Zn + j] - upz_past[i * Zn + j] + (dt * dt) * (dthetadz - O_thetaz[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);
		usx_next[i * Zn + j] = 2 * usx_now[i * Zn + j] - usx_past[i * Zn + j] + (dt * dt) * (domegadz - O_omegaz[i * Zn + j] + duz_xdz - duz_zdx - O_duz_xdz[i * Zn + j] + O_duz_zdx[i * Zn + j]);
		usz_next[i * Zn + j] = 2 * usz_now[i * Zn + j] - usz_past[i * Zn + j] + (dt * dt) * (-domegadx + O_omegax[i * Zn + j] + dux_zdx - dux_xdz - O_dux_zdx[i * Zn + j] + O_dux_xdz[i * Zn + j]);

		upx_past[i * Zn + j] = upx_now[i * Zn + j];	upx_now[i * Zn + j] = upx_next[i * Zn + j];
		upz_past[i * Zn + j] = upz_now[i * Zn + j];	upz_now[i * Zn + j] = upz_next[i * Zn + j];
		usx_past[i * Zn + j] = usx_now[i * Zn + j];	usx_now[i * Zn + j] = usx_next[i * Zn + j];
		usz_past[i * Zn + j] = usz_now[i * Zn + j];	usz_now[i * Zn + j] = usz_next[i * Zn + j];


		ux[i * Zn + j] = upx_next[i * Zn + j] + usx_next[i * Zn + j];
		uz[i * Zn + j] = upz_next[i * Zn + j] + usz_next[i * Zn + j];

	}


}
__global__ void poynting(int Xn, int Zn, int L,float* upx_r, float* upz_r, float* usx_r, float* usz_r, float* theta_r, float* omega_r, float* upx_s, float* upz_s, float* theta_s, float* omega_s,
	float* Epx_S, float* Epz_S, float* Epx_R, float* Epz_R, float* Esx_R, float* Esz_R,
	float* fenzi_PP, float* fenzi_PS, float* fenmu_P,
	float* RR_upx_u, float* RR_upx_d, float* RR_upx_l, float* RR_upx_r, float* RR_upz_u, float* RR_upz_d, float* RR_upz_l, float* RR_upz_r,
	float* RR_usx_u, float* RR_usx_d, float* RR_usx_l, float* RR_usx_r, float* RR_usz_u, float* RR_usz_d, float* RR_usz_l, float* RR_usz_r,
	float* SS_upx_u, float* SS_upx_d, float* SS_upx_l, float* SS_upx_r, float* SS_upz_u, float* SS_upz_d, float* SS_upz_l, float* SS_upz_r)
{
	int i, j;

	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;


	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		
		Epx_S[i * Zn + j] = -theta_s[i * Zn + j] * upx_s[i * Zn + j];
		Epz_S[i * Zn + j] = -theta_s[i * Zn + j] * upz_s[i * Zn + j];  


	
		Epx_R[i * Zn + j] = -theta_r[i * Zn + j] * upx_r[i * Zn + j];
		Epz_R[i * Zn + j] = -theta_r[i * Zn + j] * upz_r[i * Zn + j];

		Esx_R[i * Zn + j] = omega_r[i * Zn + j] * usz_r[i * Zn + j];
		Esz_R[i * Zn + j] = -omega_r[i * Zn + j] * usx_r[i * Zn + j];





	
		if (Epz_R[i * Zn + j] >= 0)
		{
			RR_upx_u[i * Zn + j] = 0.0;
			RR_upx_d[i * Zn + j] = upx_r[i * Zn + j];
		}
		else
		{
			RR_upx_u[i * Zn + j] = upx_r[i * Zn + j];
			RR_upx_d[i * Zn + j] = 0.0;
		}

		if (Epx_R[i * Zn + j] >= 0)
		{
			RR_upx_l[i * Zn + j] = 0.0;
			RR_upx_r[i * Zn + j] = upx_r[i * Zn + j];
		}
		else
		{
			RR_upx_l[i * Zn + j] = upx_r[i * Zn + j];
			RR_upx_r[i * Zn + j] = 0.0;
		}

	
		if (Epz_R[i * Zn + j] >= 0)
		{
			RR_upz_u[i * Zn + j] = 0.0;
			RR_upz_d[i * Zn + j] = upz_r[i * Zn + j];
		}
		else
		{
			RR_upz_u[i * Zn + j] = upz_r[i * Zn + j];
			RR_upz_d[i * Zn + j] = 0.0;
		}

		if (Epx_R[i * Zn + j] >= 0)
		{
			RR_upz_l[i * Zn + j] = 0.0;
			RR_upz_r[i * Zn + j] = upz_r[i * Zn + j];
		}
		else
		{
			RR_upz_l[i * Zn + j] = upz_r[i * Zn + j];
			RR_upz_r[i * Zn + j] = 0.0;
		}


	
		if (Esz_R[i * Zn + j] >= 0)
		{
			RR_usx_u[i * Zn + j] = 0.0;
			RR_usx_d[i * Zn + j] = usx_r[i * Zn + j];
		}
		else
		{
			RR_usx_u[i * Zn + j] = usx_r[i * Zn + j];
			RR_usx_d[i * Zn + j] = 0.0;
		}

		if (Esx_R[i * Zn + j] >= 0)
		{
			RR_usx_l[i * Zn + j] = 0.0;
			RR_usx_r[i * Zn + j] = usx_r[i * Zn + j];
		}
		else
		{
			RR_usx_l[i * Zn + j] = usx_r[i * Zn + j];
			RR_usx_r[i * Zn + j] = 0.0;
		}

	
		if (Esz_R[i * Zn + j] >= 0)
		{
			RR_usz_u[i * Zn + j] = 0.0;
			RR_usz_d[i * Zn + j] = usz_r[i * Zn + j];
		}
		else
		{
			RR_usz_u[i * Zn + j] = usz_r[i * Zn + j];
			RR_usz_d[i * Zn + j] = 0.0;
		}

		if (Esx_R[i * Zn + j] >= 0)
		{
			RR_usz_l[i * Zn + j] = 0.0;
			RR_usz_r[i * Zn + j] = usz_r[i * Zn + j];
		}
		else
		{
			RR_usz_l[i * Zn + j] = usz_r[i * Zn + j];
			RR_usz_r[i * Zn + j] = 0.0;
		}

		
		if (Epz_S[i * Zn + j] >= 0)
		{
			SS_upx_u[i * Zn + j] = 0.0;
			SS_upx_d[i * Zn + j] = upx_s[i * Zn + j];
		}
		else
		{
			SS_upx_u[i * Zn + j] = upx_s[i * Zn + j];
			SS_upx_d[i * Zn + j] = 0.0;
		}

		if (Epx_S[i * Zn + j] >= 0)
		{
			SS_upx_l[i * Zn + j] = 0.0;
			SS_upx_r[i * Zn + j] = upx_s[i * Zn + j];
		}
		else
		{
			SS_upx_l[i * Zn + j] = upx_s[i * Zn + j];
			SS_upx_r[i * Zn + j] = 0.0;
		}

		
		if (Epz_S[i * Zn + j] >= 0)
		{
			SS_upz_u[i * Zn + j] = 0.0;
			SS_upz_d[i * Zn + j] = upz_s[i * Zn + j];
		}
		else
		{
			SS_upz_u[i * Zn + j] = upz_s[i * Zn + j];
			SS_upz_d[i * Zn + j] = 0.0;
		}

		if (Epx_S[i * Zn + j] >= 0)
		{
			SS_upz_l[i * Zn + j] = 0.0;
			SS_upz_r[i * Zn + j] = upz_s[i * Zn + j];
		}
		else
		{
			SS_upz_l[i * Zn + j] = upz_s[i * Zn + j];
			SS_upz_r[i * Zn + j] = 0.0;
		}


		fenzi_PP[i * Zn + j] += SS_upx_u[i * Zn + j] * RR_upx_d[i * Zn + j] + SS_upz_u[i * Zn + j] * RR_upz_d[i * Zn + j] +
			SS_upx_d[i * Zn + j] * RR_upx_u[i * Zn + j] + SS_upz_d[i * Zn + j] * RR_upz_u[i * Zn + j] +
			SS_upx_r[i * Zn + j] * RR_upx_l[i * Zn + j] + SS_upz_r[i * Zn + j] * RR_upz_l[i * Zn + j] +
			SS_upx_l[i * Zn + j] * RR_upx_r[i * Zn + j] + SS_upz_l[i * Zn + j] * RR_upz_r[i * Zn + j];

		fenzi_PS[i * Zn + j] += SS_upx_u[i * Zn + j] * RR_usx_d[i * Zn + j] + SS_upz_u[i * Zn + j] * RR_usz_d[i * Zn + j] +
			SS_upx_d[i * Zn + j] * RR_usx_u[i * Zn + j] + SS_upz_d[i * Zn + j] * RR_usz_u[i * Zn + j] +
			SS_upx_r[i * Zn + j] * RR_usx_l[i * Zn + j] + SS_upz_r[i * Zn + j] * RR_usz_l[i * Zn + j] +
			SS_upx_l[i * Zn + j] * RR_usx_r[i * Zn + j] + SS_upz_l[i * Zn + j] * RR_usz_r[i * Zn + j];

		fenmu_P[i * Zn + j] += upx_s[i * Zn + j] * upx_s[i * Zn + j] + upz_s[i * Zn + j] * upz_s[i * Zn + j];
	}

}
__global__ void corr_v(int Xn, int Zn, int L, float* fenzi_PP, float* fenzi_PS, float* fenmu_P, float* SS_Px, float* SS_Pz, float* PP_Px, float* PP_Pz, float* PP_Sx, float* PP_Sz)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		fenzi_PP[i * Zn + j] += SS_Px[i * Zn + j] * PP_Px[i * Zn + j] + SS_Pz[i * Zn + j] * PP_Pz[i * Zn + j];
		fenmu_P[i * Zn + j] += SS_Px[i * Zn + j] * SS_Px[i * Zn + j] + SS_Pz[i * Zn + j] * SS_Pz[i * Zn + j];
		fenzi_PS[i * Zn + j] += SS_Px[i * Zn + j] * PP_Sx[i * Zn + j] + SS_Pz[i * Zn + j] * PP_Sz[i * Zn + j];

	}
}
__global__ void image_fun(int Xn, int Zn, int L, float* fenzi, float* fenmu, float* image)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= L && i < Xn - L && j >= L && j < Zn - L)
	{
		
		image[i * Zn + j] = fenzi[i * Zn + j] / fenmu[i * Zn + j];
	
	}
}
__global__ void Laplace(int Xn, int Zn, int L, float dh, float* image, float* image_lap)
{
	int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= L + 2 && i < Xn - L - 2 && j >= L + 2 && j < Zn - L - 2)
	{
		image_lap[i * Zn + j] = (1 / dh / dh) * (image[(i + 1) * Zn + j] + image[(i - 1) * Zn + j] - 2 * image[i * Zn + j] +
			image[i * Zn + j + 1] + image[i * Zn + j - 1] - 2 * image[i * Zn + j]);
	}

}
int main() {
	hipSetDevice(1);
	int i, j, t,t0;
	int xn = 400;
	int zn = 200;
	int shot_num = 80;
	int shotx;
	int shotz;
	
	int receiver_interp = 1;
	int receiver_depth = pml + 4;
	int receiver_num = 400;	
	int shotno;
	
	int Xn;
	int Zn;

	Xn = xn + 2 * pml;
	Zn = zn + 2 * pml;
	
	FILE* fp;	
	char filename[2000];
	int dh;
	dh = 5.0;

	
	float* source = (float*)calloc(Tn, sizeof(float));
	float Nk = pi * pi * fm * fm * dt * dt;
	t0 = ceil(1.0 / (fm * dt));
	for (t = 0; t < Tn; t++)
	{
		source[t] = (1.0 - 2.0 * Nk * (t - t0) * (t - t0)) * exp(-Nk * (t - t0) * (t - t0));
		
	}
	char vpfile[1000], vsfile[1000], denfile[1000];
	float* vp = (float*)calloc(Xn * Zn, sizeof(float));
	float* vs = (float*)calloc(Xn * Zn, sizeof(float));
	float* rou = (float*)calloc(Xn * Zn, sizeof(float));
	float* vp_pml = (float*)calloc(Xn * Zn, sizeof(float));
	float* vs_pml = (float*)calloc(Xn * Zn, sizeof(float));
	float* rou_pml = (float*)calloc(Xn * Zn, sizeof(float));
	float* ddx = (float*)calloc(Xn * Zn, sizeof(float));
	float* ddz = (float*)calloc(Xn * Zn, sizeof(float));

	dim3 dimGrid(ceil(Xn / 8.0), ceil(Zn / 8.0), 1);
	dim3 dimBlock(8, 8, 1);

	
	for (i = pml; i < Xn - pml; i++)
	{
		for (j = pml; j < zn / 2 + pml; j++)
		{
			vp[i * Zn + j] = 3200;
			vs[i * Zn + j] = 1700;
			rou[i * Zn + j] = 1.0;
		}
	}

	for (i = pml; i < Xn - pml; i++)
	{
		for (j = zn / 2 + pml; j < Zn - pml; j++)
		{
			vp[i * Zn + j] = 3900;
			vs[i * Zn + j] = 2300;
			rou[i * Zn + j] = 1.0;
		}
	}

	addpml(vp, vs, rou, vp_pml, vs_pml, rou_pml, Xn, Zn);


	dumpingfactor(vp_pml, ddx, ddz, Xn, Zn, xn, zn);
	sprintf(filename, "./snapshot/vp_%d_%d.dat", xn, zn);
	write_bin(vp_pml, filename, Xn, Zn, pml);
	sprintf(filename, "./snapshot/vs_%d_%d.dat", xn, zn);
	write_bin(vs_pml, filename, Xn, Zn, pml);

	float* frontwaveux = (float*)calloc(Xn * Zn, sizeof(float));
	float* frontwaveuz = (float*)calloc(Xn * Zn, sizeof(float));
	float* ux = (float*)calloc(Xn * Zn, sizeof(float));
	float* uz = (float*)calloc(Xn * Zn, sizeof(float));
	float* upx = (float*)calloc(Xn * Zn, sizeof(float));
	float* upz = (float*)calloc(Xn * Zn, sizeof(float));
	float* usx = (float*)calloc(Xn * Zn, sizeof(float));
	float* usz = (float*)calloc(Xn * Zn, sizeof(float));
	float* record_upx = (float*)calloc(Xn * Tn, sizeof(float));
	float* record_upz = (float*)calloc(Xn * Tn, sizeof(float));
	float* record_usx = (float*)calloc(Xn * Tn, sizeof(float));
	float* record_usz = (float*)calloc(Xn * Tn, sizeof(float));
	float* record_ux = (float*)calloc(Xn * Tn, sizeof(float));
	float* record_uz = (float*)calloc(Xn * Tn, sizeof(float));
	float* ux_s = (float*)calloc(Xn * Zn, sizeof(float));
	float* uz_s = (float*)calloc(Xn * Zn, sizeof(float));
	float* ux_r = (float*)calloc(Xn * Zn, sizeof(float));
	float* uz_r = (float*)calloc(Xn * Zn, sizeof(float));
	
	float* image_PP = (float*)calloc(Xn * Zn, sizeof(float));
	float* image_PS = (float*)calloc(Xn * Zn, sizeof(float));
	float* image_PP_lap = (float*)calloc(Xn * Zn, sizeof(float));
	float* image_PS_lap = (float*)calloc(Xn * Zn, sizeof(float));
	float* image_PP_pyt = (float*)calloc(Xn * Zn, sizeof(float));
	float* image_PS_pyt = (float*)calloc(Xn * Zn, sizeof(float));
	float* All_image_PP = (float*)calloc(Xn * Zn, sizeof(float));
	float* All_image_PS = (float*)calloc(Xn * Zn, sizeof(float));
	float* All_image_PP_lap = (float*)calloc(Xn * Zn, sizeof(float));
	float* All_image_PS_lap = (float*)calloc(Xn * Zn, sizeof(float));
	float* All_image_PP_pyt = (float*)calloc(Xn * Zn, sizeof(float));
	float* All_image_PS_pyt = (float*)calloc(Xn * Zn, sizeof(float));
	

	float* d_ddx, * d_ddz, * d_vp, * d_vs;
	float* d_upx_next, * d_upx_now, * d_upx_past, * d_upz_next, * d_upz_now, * d_upz_past, * d_usx_next, * d_usx_now, * d_usx_past, * d_usz_next, * d_usz_now, * d_usz_past;	
	float* d_recordupx, * d_recordupz, * d_recordusx, * d_recordusz, * d_recordux, * d_recorduz;
	float* O_duz_xdz, * O_duz_zdx, * O_dux_zdx, * O_dux_xdz, * O_thetax, * O_thetaz, * O_omegaz, * O_omegax;
	float* F_xx, * F_zz, * F_xz, * F_zx;
	float* d_ux, * d_uz, * d_theta, * d_omega, * d_duzdx, * d_duzdz, * d_duxdx, * d_duxdz;
	float* d_source;

	float* d_ux_up, * d_uz_up, * d_theta_up, * d_omega_up, * d_duzdx_up, * d_duzdz_up, * d_duxdx_up, * d_duxdz_up;
	float* d_ux_dn, * d_uz_dn, * d_theta_dn, * d_omega_dn, * d_duzdx_dn, * d_duzdz_dn, * d_duxdx_dn, * d_duxdz_dn;
	float* d_ux_lf, * d_uz_lf, * d_theta_lf, * d_omega_lf, * d_duzdx_lf, * d_duzdz_lf, * d_duxdx_lf, * d_duxdz_lf;
	float* d_ux_rt, * d_uz_rt, * d_theta_rt, * d_omega_rt, * d_duzdx_rt, * d_duzdz_rt, * d_duxdx_rt, * d_duxdz_rt;
	float* d_upx_next_up, * d_upz_next_up, * d_usx_next_up, * d_usz_next_up, * d_upx_now_up, * d_upz_now_up, * d_usx_now_up, * d_usz_now_up;
	float* d_upx_next_dn, * d_upz_next_dn, * d_usx_next_dn, * d_usz_next_dn, * d_upx_now_dn, * d_upz_now_dn, * d_usx_now_dn, * d_usz_now_dn;
	float* d_upx_next_lf, * d_upz_next_lf, * d_usx_next_lf, * d_usz_next_lf, * d_upx_now_lf, * d_upz_now_lf, * d_usx_now_lf, * d_usz_now_lf;
	float* d_upx_next_rt, * d_upz_next_rt, * d_usx_next_rt, * d_usz_next_rt, * d_upx_now_rt, * d_upz_now_rt, * d_usx_now_rt, * d_usz_now_rt;

	float* d_ux_s, * d_uz_s, * d_theta_s, * d_omega_s, * d_duzdx_s, * d_duzdz_s, * d_duxdz_s, * d_duxdx_s;
	float* d_upx_next_s, * d_upz_next_s, * d_usx_next_s, * d_usz_next_s;
	float* d_upx_now_s, * d_upz_now_s, * d_usx_now_s, * d_usz_now_s;
	float* d_upx_past_s, * d_upz_past_s, * d_usx_past_s, * d_usz_past_s;

	float* d_ux_r, * d_uz_r, * d_theta_r, * d_omega_r, * d_duxdz_r, * d_duzdz_r, * d_duxdx_r, * d_duzdx_r;
	float* d_upx_next_r, * d_upz_next_r, * d_usx_next_r, * d_usz_next_r;
	float* d_upx_now_r, * d_upz_now_r, * d_usx_now_r, * d_usz_now_r;
	float* d_upx_past_r, * d_upz_past_r, * d_usx_past_r, * d_usz_past_r;
	float* O_duz_xdz_r, * O_duz_zdx_r, * O_dux_zdx_r, * O_dux_xdz_r;
	float* O_thetax_r, * O_thetaz_r, * O_omegaz_r, * O_omegax_r;
	float* F_xx_r, * F_zz_r, * F_xz_r, * F_zx_r;
	
	float* d_Epx_S, * d_Epz_S, * d_Epx_R, * d_Epz_R, * d_Esx_R, * d_Esz_R;	
	float* d_fenzi_PP_pyt, * d_fenzi_PS_pyt, * d_fenmu_P_pyt;
	float* d_fenzi_PP, * d_fenzi_PS, * d_fenmu_P;
	float* d_RR_upx_up, * d_RR_upx_dn, * d_RR_upx_lf, * d_RR_upx_rt;
	float* d_RR_upz_up, * d_RR_upz_dn, * d_RR_upz_lf, * d_RR_upz_rt;
	float* d_RR_usx_up, * d_RR_usx_dn, * d_RR_usx_lf, * d_RR_usx_rt;
	float* d_RR_usz_up, * d_RR_usz_dn, * d_RR_usz_lf, * d_RR_usz_rt;
	float* d_SS_upx_up, * d_SS_upx_dn, * d_SS_upx_lf, * d_SS_upx_rt;
	float* d_SS_upz_up, * d_SS_upz_dn, * d_SS_upz_lf, * d_SS_upz_rt;
	
	float* d_image_PP, * d_image_PS;
	float* d_image_PP_lap, * d_image_PS_lap;
	float* d_image_PP_pyt, * d_image_PS_pyt;

	hipMalloc((void**)&d_ddx, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_ddz, Xn * Zn * sizeof(float));	
	hipMalloc((void**)&d_vp, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_vs, Xn * Zn * sizeof(float));

	hipMalloc((void**)&d_upx_next, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_upx_now, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_upx_past, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_upz_next, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_upz_now, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_upz_past, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_usx_next, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_usx_now, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_usx_past, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_usz_next, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_usz_now, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_usz_past, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_ux, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_uz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_theta, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_omega, Xn * Zn * sizeof(float));
	
	hipMalloc((void**)&O_duz_xdz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&O_duz_zdx, Xn * Zn * sizeof(float));
	hipMalloc((void**)&O_dux_zdx, Xn * Zn * sizeof(float));
	hipMalloc((void**)&O_dux_xdz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&O_thetax, Xn * Zn * sizeof(float));
	hipMalloc((void**)&O_thetaz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&O_omegaz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&O_omegax, Xn * Zn * sizeof(float));	
	hipMalloc((void**)&F_xx, Xn * Zn * sizeof(float));
	hipMalloc((void**)&F_zz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&F_xz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&F_zx, Xn * Zn * sizeof(float));	
	hipMalloc((void**)&d_duzdx, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_duzdz, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_duxdx, Xn * Zn * sizeof(float));
	hipMalloc((void**)&d_duxdz, Xn * Zn * sizeof(float));

	hipMalloc((void**)&d_recordupx, Xn* Tn * sizeof(float));
	hipMalloc((void**)&d_recordupz, Xn* Tn * sizeof(float));
	hipMalloc((void**)&d_recordusx, Xn* Tn * sizeof(float));
	hipMalloc((void**)&d_recordusz, Xn* Tn * sizeof(float));
	hipMalloc((void**)&d_recordux, Xn* Tn * sizeof(float));
	hipMalloc((void**)&d_recorduz, Xn* Tn * sizeof(float));
	hipMalloc((void**)&d_source, Tn * sizeof(float));
	

	hipMalloc((void**)&d_ux_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_uz_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_next_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_next_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_next_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_next_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_now_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_now_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_now_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_now_up, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_ux_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_uz_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_next_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_next_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_next_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_next_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_now_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_now_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_now_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_now_dn, 2 * N * Xn * (Tn - 1) * sizeof(float));
	
	hipMalloc((void**)&d_ux_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_uz_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_next_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_next_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_next_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_next_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_now_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_now_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_now_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_now_lf, 2 * N * Zn * (Tn - 1) * sizeof(float));	
	hipMalloc((void**)&d_ux_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_uz_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_theta_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_omega_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdx_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duzdz_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdx_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_duxdz_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_next_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_next_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_next_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_next_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upx_now_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_upz_now_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usx_now_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMalloc((void**)&d_usz_now_rt, 2 * N * Zn * (Tn - 1) * sizeof(float));
		
	hipMalloc((void**)&d_ux_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_uz_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_theta_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_omega_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duzdx_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duzdz_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duxdz_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duxdx_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upx_next_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upz_next_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usx_next_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usz_next_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upx_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upz_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usx_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usz_now_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upx_past_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upz_past_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usx_past_s, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usz_past_s, Xn* Zn * sizeof(float));
	
	hipMalloc((void**)&d_ux_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_uz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_theta_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_omega_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duxdz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duzdz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duxdx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_duzdx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upx_next_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upz_next_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usx_next_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usz_next_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upx_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upz_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usx_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usz_now_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upx_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_upz_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usx_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_usz_past_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_duz_xdz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_duz_zdx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_dux_zdx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_dux_xdz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_thetax_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_thetaz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_omegaz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&O_omegax_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_xx_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_zz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_xz_r, Xn* Zn * sizeof(float));
	hipMalloc((void**)&F_zx_r, Xn* Zn * sizeof(float));

	hipMalloc((void**)&d_Epx_S, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_Epz_S, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_Epx_R, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_Epz_R, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_Esx_R, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_Esz_R, Xn* Zn * sizeof(float));

	hipMalloc((void**)&d_fenzi_PP_pyt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_fenzi_PS_pyt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_fenmu_P_pyt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_fenzi_PP, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_fenzi_PS, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_fenmu_P, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PP, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PS, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PP_lap, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PS_lap, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PP_pyt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_image_PS_pyt, Xn* Zn * sizeof(float));

	hipMalloc((void**)&d_RR_upx_up, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_upz_up, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usx_up, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usz_up, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upx_up, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upz_up, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_upx_dn, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_upz_dn, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usx_dn, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usz_dn, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upx_dn, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upz_dn, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_upx_lf, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_upz_lf, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usx_lf, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usz_lf, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upx_lf, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upz_lf, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_upx_rt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_upz_rt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usx_rt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_RR_usz_rt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upx_rt, Xn* Zn * sizeof(float));
	hipMalloc((void**)&d_SS_upz_rt, Xn* Zn * sizeof(float));


	hipMemset(d_vp, 0, Xn * Zn * sizeof(float));
	hipMemset(d_vs, 0, Xn * Zn * sizeof(float));	
	hipMemset(d_ddx, 0, Xn * Zn * sizeof(float));
	hipMemset(d_ddz, 0, Xn * Zn * sizeof(float));
	hipMemset(d_upx_next, 0, Xn * Zn * sizeof(float));
	hipMemset(d_upx_now, 0, Xn * Zn * sizeof(float));
	hipMemset(d_upx_past, 0, Xn * Zn * sizeof(float));
	hipMemset(d_upz_next, 0, Xn * Zn * sizeof(float));
	hipMemset(d_upz_now, 0, Xn * Zn * sizeof(float));
	hipMemset(d_upz_past, 0, Xn * Zn * sizeof(float));
	hipMemset(d_usx_next, 0, Xn * Zn * sizeof(float));
	hipMemset(d_usx_now, 0, Xn * Zn * sizeof(float));
	hipMemset(d_usx_past, 0, Xn * Zn * sizeof(float));
	hipMemset(d_usz_next, 0, Xn * Zn * sizeof(float));
	hipMemset(d_usz_now, 0, Xn * Zn * sizeof(float));
	hipMemset(d_usz_past, 0, Xn * Zn * sizeof(float));
	hipMemset(d_ux, 0, Xn * Zn * sizeof(float));
	hipMemset(d_uz, 0, Xn * Zn * sizeof(float));
	hipMemset(d_theta, 0, Xn * Zn * sizeof(float));
	hipMemset(d_omega, 0, Xn * Zn * sizeof(float));
	
	hipMemset(O_duz_xdz, 0, Xn* Zn * sizeof(float));
	hipMemset(O_duz_zdx, 0, Xn* Zn * sizeof(float));
	hipMemset(O_dux_zdx, 0, Xn* Zn * sizeof(float));
	hipMemset(O_dux_xdz, 0, Xn* Zn * sizeof(float));
	hipMemset(O_thetax, 0, Xn* Zn * sizeof(float));
	hipMemset(O_thetaz, 0, Xn* Zn * sizeof(float));
	hipMemset(O_omegaz, 0, Xn* Zn * sizeof(float));
	hipMemset(O_omegax, 0, Xn* Zn * sizeof(float));	
	hipMemset(F_xx, 0, Xn* Zn * sizeof(float));
	hipMemset(F_zz, 0, Xn* Zn * sizeof(float));
	hipMemset(F_xz, 0, Xn* Zn * sizeof(float));
	hipMemset(F_zx, 0, Xn* Zn * sizeof(float));	
	hipMemset(d_duzdx, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duzdz, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duxdx, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duxdz, 0, Xn* Zn * sizeof(float));

	hipMemset(d_recordupx, 0, Xn * Tn * sizeof(float));
	hipMemset(d_recordupz, 0, Xn * Tn * sizeof(float));
	hipMemset(d_recordusx, 0, Xn * Tn * sizeof(float));
	hipMemset(d_recordusz, 0, Xn * Tn * sizeof(float));
	hipMemset(d_recordux, 0, Xn * Tn * sizeof(float));
	hipMemset(d_recorduz, 0, Xn * Tn * sizeof(float));
	

	hipMemset(d_ux_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_uz_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_theta_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_omega_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdx_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdz_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdx_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdz_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_next_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_next_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_next_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_next_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_now_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_now_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_now_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_now_up, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_ux_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_uz_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_theta_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_omega_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdx_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdz_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdx_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdz_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_next_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_next_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_next_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_next_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_now_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_now_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_now_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_now_dn, 0, 2 * N * Xn * (Tn - 1) * sizeof(float));	
	hipMemset(d_ux_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_uz_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_theta_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_omega_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdx_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdz_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdx_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdz_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_next_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_next_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_next_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_next_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_now_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_now_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_now_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_now_lf, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_ux_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_uz_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_theta_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_omega_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdx_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duzdz_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdx_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_duxdz_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_next_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_next_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_next_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_next_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upx_now_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_upz_now_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usx_now_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	hipMemset(d_usz_now_rt, 0, 2 * N * Zn * (Tn - 1) * sizeof(float));
	
	hipMemset(d_ux_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_uz_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_theta_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_omega_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duzdx_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duzdz_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duxdz_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duxdx_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upx_next_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upz_next_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usx_next_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usz_next_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upx_now_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upz_now_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usx_now_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usz_now_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upx_past_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upz_past_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usx_past_s, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usz_past_s, 0, Xn* Zn * sizeof(float));
	
	hipMemset(d_ux_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_uz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_theta_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_omega_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duxdz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duzdz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duxdx_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_duzdx_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upx_next_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upz_next_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usx_next_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usz_next_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upx_now_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upz_now_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usx_now_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usz_now_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upx_past_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_upz_past_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usx_past_r, 0, Xn* Zn * sizeof(float));
	hipMemset(d_usz_past_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_duz_xdz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_duz_zdx_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_dux_zdx_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_dux_xdz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_thetax_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_thetaz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_omegaz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(O_omegax_r, 0, Xn* Zn * sizeof(float));
	hipMemset(F_xx_r, 0, Xn* Zn * sizeof(float));
	hipMemset(F_zz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(F_xz_r, 0, Xn* Zn * sizeof(float));
	hipMemset(F_zx_r, 0, Xn* Zn * sizeof(float));
	
	hipMemset(d_Epx_S, 0, Xn* Zn * sizeof(float));
	hipMemset(d_Epz_S, 0, Xn* Zn * sizeof(float));
	hipMemset(d_Epx_R, 0, Xn* Zn * sizeof(float));
	hipMemset(d_Epz_R, 0, Xn* Zn * sizeof(float));
	hipMemset(d_Esx_R, 0, Xn* Zn * sizeof(float));
	hipMemset(d_Esz_R, 0, Xn* Zn * sizeof(float));	
	hipMemset(d_RR_upx_up, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_upx_dn, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_upx_lf, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_upx_rt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_upz_up, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_upz_dn, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_upz_lf, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_upz_rt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usx_up, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usx_dn, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usx_lf, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usx_rt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usz_up, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usz_dn, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usz_lf, 0, Xn* Zn * sizeof(float));
	hipMemset(d_RR_usz_rt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upx_up, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upx_dn, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upx_lf, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upx_rt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upz_up, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upz_dn, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upz_lf, 0, Xn* Zn * sizeof(float));
	hipMemset(d_SS_upz_rt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_fenzi_PP_pyt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_fenzi_PS_pyt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_fenmu_P_pyt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_fenzi_PP, 0, Xn* Zn * sizeof(float));
	hipMemset(d_fenzi_PS, 0, Xn* Zn * sizeof(float));
	hipMemset(d_fenmu_P, 0, Xn* Zn * sizeof(float));
	hipMemset(d_image_PP, 0, Xn* Zn * sizeof(float));
	hipMemset(d_image_PS, 0, Xn* Zn * sizeof(float));
	hipMemset(d_image_PP_lap, 0, Xn* Zn * sizeof(float));
	hipMemset(d_image_PS_lap, 0, Xn* Zn * sizeof(float));
	hipMemset(d_image_PP_pyt, 0, Xn* Zn * sizeof(float));
	hipMemset(d_image_PS_pyt, 0, Xn* Zn * sizeof(float));

	hipMemcpy(d_ddx, ddx, Xn* Zn * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_ddz, ddz, Xn* Zn * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_vp, vp_pml, Xn* Zn * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_vs, vs_pml, Xn* Zn * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_source, source, Tn * sizeof(float), hipMemcpyHostToDevice);


	int k;
	for (k = 0; k < 1; k++)
	{
		cout << k + 1 << " th iteration:" << endl;

		for (shotno = 0; shotno < shot_num; shotno++) {

			shotx = pml + shotno * 5;
			shotz = pml + 1;
			

			cout << k + 1 << "  shotnumber:" << shotno + 1 << endl;
			hipMemset(d_upx_next, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_now, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_past, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_next, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_now, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_past, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_next, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_now, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_past, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_next, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_now, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_past, 0, Xn * Zn * sizeof(float));
			hipMemset(d_ux, 0, Xn * Zn * sizeof(float));
			hipMemset(d_uz, 0, Xn * Zn * sizeof(float));
			hipMemset(d_theta, 0, Xn * Zn * sizeof(float));
			hipMemset(d_omega, 0, Xn * Zn * sizeof(float));

			hipMemset(O_duz_xdz, 0, Xn * Zn * sizeof(float));
			hipMemset(O_duz_zdx, 0, Xn * Zn * sizeof(float));
			hipMemset(O_dux_zdx, 0, Xn * Zn * sizeof(float));
			hipMemset(O_dux_xdz, 0, Xn * Zn * sizeof(float));
			hipMemset(O_thetax, 0, Xn * Zn * sizeof(float));
			hipMemset(O_thetaz, 0, Xn * Zn * sizeof(float));
			hipMemset(O_omegaz, 0, Xn * Zn * sizeof(float));
			hipMemset(O_omegax, 0, Xn * Zn * sizeof(float));
			hipMemset(F_xx, 0, Xn * Zn * sizeof(float));
			hipMemset(F_zz, 0, Xn * Zn * sizeof(float));
			hipMemset(F_xz, 0, Xn * Zn * sizeof(float));
			hipMemset(F_zx, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duzdx, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duzdz, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duxdx, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duxdz, 0, Xn * Zn * sizeof(float));

			hipMemset(d_recordupx, 0, Xn * Tn * sizeof(float));
			hipMemset(d_recordupz, 0, Xn * Tn * sizeof(float));
			hipMemset(d_recordusx, 0, Xn * Tn * sizeof(float));
			hipMemset(d_recordusz, 0, Xn * Tn * sizeof(float));
			hipMemset(d_recordux, 0, Xn * Tn * sizeof(float));
			hipMemset(d_recorduz, 0, Xn * Tn * sizeof(float));

			hipMemset(d_ux_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_uz_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_theta_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_omega_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duzdx_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duzdz_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duxdz_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duxdx_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_next_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_next_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_next_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_next_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_now_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_now_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_now_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_now_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_past_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_past_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_past_s, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_past_s, 0, Xn * Zn * sizeof(float));

			hipMemset(d_ux_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_uz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_theta_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_omega_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duxdz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duzdz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duxdx_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_duzdx_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_next_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_next_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_next_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_next_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_now_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_now_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_now_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_now_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upx_past_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_upz_past_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usx_past_r, 0, Xn * Zn * sizeof(float));
			hipMemset(d_usz_past_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_duz_xdz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_duz_zdx_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_dux_zdx_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_dux_xdz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_thetax_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_thetaz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_omegaz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(O_omegax_r, 0, Xn * Zn * sizeof(float));
			hipMemset(F_xx_r, 0, Xn * Zn * sizeof(float));
			hipMemset(F_zz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(F_xz_r, 0, Xn * Zn * sizeof(float));
			hipMemset(F_zx_r, 0, Xn * Zn * sizeof(float));
			for (t = 0; t < Tn; t++) {

				forward_s << <dimGrid, dimBlock >> > (Xn, Zn, d_vp, d_vs, d_ux, d_uz, d_ddx, d_ddz, d_theta, d_omega, t, shotx, shotz, d_source,
					F_xx, F_zz, F_xz, F_zx, d_duxdz, d_duzdz, d_duxdx, d_duzdx);

				forward_u << <dimGrid, dimBlock >> > (Xn, Zn, d_ux, d_uz, d_ddx, d_ddz,
					d_upx_next, d_upx_now, d_upx_past, d_upz_next, d_upz_now, d_upz_past, d_usx_next, d_usx_now, d_usx_past, d_usz_next, d_usz_now, d_usz_past,
					d_recordupx, d_recordupz, d_recordusx, d_recordusz, d_recordux, d_recorduz,
					d_theta, d_omega, receiver_depth, t, d_duxdz, d_duzdz, d_duxdx, d_duzdx,
					O_duz_xdz, O_duz_zdx, O_dux_zdx, O_dux_xdz, O_thetax, O_thetaz, O_omegaz, O_omegax);

				if (t != Tn - 1)
				{

					save_wavefiled << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_ux_up, d_uz_up, d_theta_up, d_omega_up, d_duzdx_up, d_duzdz_up, d_duxdx_up, d_duxdz_up,
						d_ux_dn, d_uz_dn, d_theta_dn, d_omega_dn, d_duzdx_dn, d_duzdz_dn, d_duxdx_dn, d_duxdz_dn,
						d_ux_lf, d_uz_lf, d_theta_lf, d_omega_lf, d_duzdx_lf, d_duzdz_lf, d_duxdx_lf, d_duxdz_lf,
						d_ux_rt, d_uz_rt, d_theta_rt, d_omega_rt, d_duzdx_rt, d_duzdz_rt, d_duxdx_rt, d_duxdz_rt,
						d_ux, d_uz, d_theta, d_omega, d_duzdx, d_duzdz, d_duxdx, d_duxdz, t,
						d_upx_next_up, d_upz_next_up, d_usx_next_up, d_usz_next_up, d_upx_now_up, d_upz_now_up, d_usx_now_up, d_usz_now_up,
						d_upx_next_dn, d_upz_next_dn, d_usx_next_dn, d_usz_next_dn, d_upx_now_dn, d_upz_now_dn, d_usx_now_dn, d_usz_now_dn,
						d_upx_next_lf, d_upz_next_lf, d_usx_next_lf, d_usz_next_lf, d_upx_now_lf, d_upz_now_lf, d_usx_now_lf, d_usz_now_lf,
						d_upx_next_rt, d_upz_next_rt, d_usx_next_rt, d_usz_next_rt, d_upx_now_rt, d_upz_now_rt, d_usx_now_rt, d_usz_now_rt,
						d_upx_next, d_upz_next, d_usx_next, d_usz_next, d_upx_now, d_upz_now, d_usx_now, d_usz_now);
				}
				
				
			}
			remove << <dimGrid, dimBlock >> > (Xn, Zn,shotx, shotz, receiver_depth, t0, dh, d_vp, d_recordux, d_recorduz);
			hipMemcpy(record_ux, d_recordux, Xn * Tn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(record_uz, d_recorduz, Xn * Tn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(record_upx, d_recordupx, Xn * Tn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(record_upz, d_recordupz, Xn * Tn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(record_usx, d_recordusx, Xn * Tn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(record_usz, d_recordusz, Xn * Tn * sizeof(float), hipMemcpyDeviceToHost);

			sprintf(filename, "./record/record_ux_%d_%d_%d.dat", xn, t, shotno + 1);
			write_bin1(record_ux, filename, Xn, Tn, pml);
			sprintf(filename, "./record/record_uz_%d_%d_%d.dat", xn, t, shotno + 1);
			write_bin1(record_uz, filename, Xn, Tn, pml);
		
			for (t = Tn - 1; t >= 0; t--)
			{
				
				
				load_record << <dimGrid, dimBlock >> > (Xn, Zn, pml, receiver_depth, d_ux_r, d_uz_r, d_recordux, d_recorduz, t);

				rt_s_res << <dimGrid, dimBlock >> > (Xn, Zn, d_ux_r, d_uz_r, d_theta_r, d_omega_r, d_duxdz_r, d_duzdz_r, d_duxdx_r, d_duzdx_r,
					d_ddx, d_ddz, d_vp, d_vs, F_xx_r, F_zz_r, F_xz_r, F_zx_r);

				rt_u_res << <dimGrid, dimBlock >> > (Xn, Zn, d_ux_r, d_uz_r, d_upx_next_r, d_upz_next_r, d_usx_next_r, d_usz_next_r, d_theta_r, d_omega_r, d_duxdz_r, d_duzdz_r, d_duxdx_r, d_duzdx_r,
					d_ddz, d_ddx, d_upx_now_r, d_upz_now_r, d_usx_now_r, d_usz_now_r,
					d_upx_past_r, d_upz_past_r, d_usx_past_r, d_usz_past_r, O_duz_xdz_r, O_duz_zdx_r, O_dux_zdx_r, O_dux_xdz_r,
					O_thetax_r, O_thetaz_r, O_omegaz_r, O_omegax_r);

				if (t == Tn - 1)
				{
					read_wavefiled_NT1 << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_upx_past_s, d_upz_past_s, d_usx_past_s, d_usz_past_s,
						d_upx_next, d_upz_next, d_usx_next, d_usz_next);

				}
				if (t == Tn - 2)
				{
					read_wavefiled_NT2 << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_upx_past_s, d_upz_past_s, d_usx_past_s, d_usz_past_s,
						d_upx_now, d_upz_now, d_usx_now, d_usz_now);
					

				}
				if (t == Tn - 3)
				{
					


					read_wavefiled_NT3 << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_upx_next_s, d_upz_next_s, d_usx_next_s, d_usz_next_s,
						d_upx_next, d_upz_next, d_usx_next, d_usz_next, d_upx_now, d_upz_now, d_usx_now, d_usz_now,
						d_upx_now_s, d_upz_now_s, d_usx_now_s, d_usz_now_s,
						d_upx_past, d_upz_past, d_usx_past, d_usz_past,
						d_upx_past_s, d_upz_past_s, d_usx_past_s, d_usz_past_s);

					
				}


				if (t < Tn - 3) 
				{
					reshot_u << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_ux_s, d_uz_s, d_upx_next_s, d_upz_next_s, d_usx_next_s, d_usz_next_s, d_theta_s, d_omega_s,
						d_duxdz_s, d_duzdz_s, d_duxdx_s, d_duzdx_s,
						d_upx_now_s, d_upz_now_s, d_usx_now_s, d_usz_now_s,
						d_upx_past_s, d_upz_past_s, d_usx_past_s, d_usz_past_s);

					read_wavefiled1 << <dimGrid, dimBlock >> > (t, pml, Xn, Zn, d_ux_up, d_uz_up, d_ux_dn, d_uz_dn, d_ux_lf, d_uz_lf, d_ux_rt, d_uz_rt, d_ux_s, d_uz_s);
					

					reshot_s << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_ux_s, d_uz_s, d_theta_s, d_omega_s, d_duxdz_s, d_duzdz_s, d_duxdx_s, d_duzdx_s,
						d_vp, d_vs);

					read_wavefiled2 << <dimGrid, dimBlock >> > (d_theta_up, d_omega_up, d_duzdx_up, d_duzdz_up, d_duxdx_up, d_duxdz_up,
						d_theta_dn, d_omega_dn, d_duzdx_dn, d_duzdz_dn, d_duxdx_dn, d_duxdz_dn,
						d_theta_lf, d_omega_lf, d_duzdx_lf, d_duzdz_lf, d_duxdx_lf, d_duxdz_lf,
						d_theta_rt, d_omega_rt, d_duzdx_rt, d_duzdz_rt, d_duxdx_rt, d_duxdz_rt,
						d_theta_s, d_omega_s, d_duzdx_s, d_duzdz_s, d_duxdx_s, d_duxdz_s, t, pml, Xn, Zn);

					

				}
				

				poynting << <dimGrid, dimBlock >> > (Xn, Zn, pml,d_upx_next_r, d_upz_next_r, d_usx_next_r, d_usz_next_r, d_theta_r, d_omega_r, d_upx_past_s, d_upz_past_s, d_theta_s, d_omega_s,
					d_Epx_S, d_Epz_S, d_Epx_R, d_Epz_R, d_Esx_R, d_Esz_R, d_fenzi_PP_pyt, d_fenzi_PS_pyt, d_fenmu_P_pyt,					
					d_RR_upx_up, d_RR_upx_dn, d_RR_upx_lf, d_RR_upx_rt, d_RR_upz_up, d_RR_upz_dn, d_RR_upz_lf, d_RR_upz_rt,
					d_RR_usx_up, d_RR_usx_dn, d_RR_usx_lf, d_RR_usx_rt, d_RR_usz_up, d_RR_usz_dn, d_RR_usz_lf, d_RR_usz_rt,
					d_SS_upx_up, d_SS_upx_dn, d_SS_upx_lf, d_SS_upx_rt, d_SS_upz_up, d_SS_upz_dn, d_SS_upz_lf, d_SS_upz_rt);

				corr_v << <dimGrid, dimBlock >> > (Xn, Zn, pml,d_fenzi_PP, d_fenzi_PS, d_fenmu_P, d_upx_past_s, d_upz_past_s, d_upx_next_r, d_upz_next_r, d_usx_next_r, d_usz_next_r);
				
				if (t % 500 == 0) {
					printf("t=%d\n", t);
				}
			}
			image_fun << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_fenzi_PP, d_fenmu_P, d_image_PP);
			image_fun << <dimGrid, dimBlock >> > (Xn, Zn, pml, d_fenzi_PS, d_fenmu_P, d_image_PS);

			hipMemcpy(image_PP, d_image_PP, Xn * Zn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(image_PS, d_image_PS, Xn * Zn * sizeof(float), hipMemcpyDeviceToHost);

			image_fun << <dimGrid, dimBlock >> > (Xn, Zn, pml,d_fenzi_PP_pyt, d_fenmu_P_pyt, d_image_PP_pyt);
			image_fun << <dimGrid, dimBlock >> > (Xn, Zn, pml,d_fenzi_PS_pyt, d_fenmu_P_pyt, d_image_PS_pyt);
			hipMemcpy(image_PP_pyt, d_image_PP_pyt, Xn * Zn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(image_PS_pyt, d_image_PS_pyt, Xn * Zn * sizeof(float), hipMemcpyDeviceToHost);

			Laplace << <dimGrid, dimBlock >> > (Xn, Zn, pml,dx, d_image_PP_pyt, d_image_PP_lap);
			Laplace << <dimGrid, dimBlock >> > (Xn, Zn, pml,dx, d_image_PS_pyt, d_image_PS_lap);
	
			hipMemcpy(image_PP_lap, d_image_PP_lap, Xn * Zn * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(image_PS_lap, d_image_PS_lap, Xn * Zn * sizeof(float), hipMemcpyDeviceToHost);

	
			sprintf(filename, "./image2/image_PP_lap_%d_%d_%d.dat", xn, zn, shotno + 1);
			write_bin(image_PP_lap, filename, Xn, Zn, pml);
			
			sprintf(filename, "./image2/image_PS_lap_%d_%d_%d.dat", xn, zn, shotno + 1);
			write_bin(image_PS_lap, filename, Xn, Zn, pml);
			
		}
	
		
		for (shotno = 0; shotno < shot_num; shotno++)
		{
			sprintf(filename, "./image2/image_PP_lap_%d_%d_%d.dat", xn, zn, shotno + 1);
			if ((fp = fopen(filename, "rb")) != NULL)
			{
				float a = 0;
				for (i = pml; i < Xn - pml; i++)
				{
					for (j = pml; j < Zn - pml; j++)
					{
						fread(&image_PP_lap[i * Zn + j], sizeof(float), 1, fp);
					}
				}
			}fclose(fp);

			sprintf(filename, "./image2/image_PS_lap_%d_%d_%d.dat", xn, zn, shotno + 1);
			if ((fp = fopen(filename, "rb")) != NULL)
			{
				float a = 0;
				for (i = pml; i < Xn - pml; i++)
				{
					for (j = pml; j < Zn - pml; j++)
					{
						fread(&image_PS_lap[i * Zn + j], sizeof(float), 1, fp);
					}
				}
			}fclose(fp);


			for (i = pml; i < Xn - pml; i++)
			{
				for (j = pml; j < Zn - pml; j++)
				{
					All_image_PP_lap[i * Zn + j] += image_PP_lap[i * Zn + j];
					All_image_PS_lap[i * Zn + j] += image_PS_lap[i * Zn + j];
				}
			}



		}

		sprintf(filename, "./image2/All_image_PP_lap.dat");
		if ((fp = fopen(filename, "wb")) != NULL)
		{
			for (i = pml; i < Xn - pml; i++)
			{
				for (j = pml; j < Zn - pml; j++)
				{
					fwrite(&All_image_PP_lap[i * Zn + j], sizeof(float), 1, fp);
				}
			}
		}fclose(fp);
		sprintf(filename, "./image2/All_image_PS_lap.dat");
		if ((fp = fopen(filename, "wb")) != NULL)
		{
			for (i = pml; i < Xn - pml; i++)
			{
				for (j = pml; j < Zn - pml; j++)
				{
					fwrite(&All_image_PS_lap[i * Zn + j], sizeof(float), 1, fp);
				}
			}
		}fclose(fp);

		
	}
	
	free(vp); free(vs); free(rou); free(vp_pml); free(vs_pml); free(rou_pml); free(ddx); free(ddz); free(frontwaveux); free(frontwaveuz); 
	free(ux); free(uz); free(upx); free(upz); free(usx); free(usz); free(record_upx); free(record_upz); free(record_usx); free(record_usz); free(record_ux); free(record_uz);
	free(ux_s); free(uz_s); free(ux_r); free(uz_r); free(image_PP); free(image_PS); free(image_PP_lap); free(image_PS_lap); free(image_PP_pyt); free(image_PS_pyt); 
	free(All_image_PP); free(All_image_PS); free(All_image_PP_lap); free(All_image_PS_lap); free(All_image_PP_pyt); free(All_image_PS_pyt);

	hipFree(d_ddx); hipFree(d_ddz); hipFree(d_vp); hipFree(d_vs); hipFree(d_upx_next); hipFree(d_upx_now); hipFree(d_upx_past); hipFree(d_upz_next); hipFree(d_upz_now); hipFree(d_upz_past);
	hipFree(d_usx_next); hipFree(d_usx_now); hipFree(d_usx_past);hipFree(d_usz_next); hipFree(d_usz_now); hipFree(d_usz_past); hipFree(d_recordupx); hipFree(d_recordupz); hipFree(d_recordusx); 
	hipFree(d_recordusz); hipFree(d_recordux); hipFree(d_recorduz); hipFree(O_duz_xdz); hipFree(O_duz_zdx); hipFree(O_dux_zdx); hipFree(O_dux_xdz);
	hipFree(O_thetax); hipFree(O_thetaz); hipFree(O_omegaz); hipFree(O_omegax); hipFree(F_xx); hipFree(F_zz); hipFree(F_xz); hipFree(F_zx); hipFree(d_ux); hipFree(d_uz); 
	hipFree(d_theta); hipFree(d_omega); hipFree(d_duzdx); hipFree(d_duzdz); hipFree(d_duxdx); hipFree(d_duxdz); hipFree(d_source); 
	hipFree(d_ux_up); hipFree(d_uz_up); hipFree(d_theta_up); hipFree(d_omega_up); hipFree(d_duzdx_up); hipFree(d_duzdz_up); hipFree(d_duxdx_up); hipFree(d_duxdz_up);
	hipFree(d_ux_dn); hipFree(d_uz_dn); hipFree(d_theta_dn); hipFree(d_omega_dn); hipFree(d_duzdx_dn); hipFree(d_duzdz_dn); hipFree(d_duxdx_dn); hipFree(d_duxdz_dn); 	
	hipFree(d_ux_lf); hipFree(d_uz_lf); hipFree(d_theta_lf); hipFree(d_omega_lf); hipFree(d_duzdx_lf); hipFree(d_duzdz_lf); hipFree(d_duxdx_lf); hipFree(d_duxdz_lf); 
	hipFree(d_ux_rt); hipFree(d_uz_rt); hipFree(d_theta_rt); hipFree(d_omega_rt); hipFree(d_duzdx_rt); hipFree(d_duzdz_rt); hipFree(d_duxdx_rt); hipFree(d_duxdz_rt); 
	hipFree(d_upx_next_up); hipFree(d_upz_next_up); hipFree(d_usx_next_up); hipFree(d_usz_next_up); hipFree(d_upx_now_up); hipFree(d_upz_now_up); hipFree(d_usx_now_up); hipFree(d_usz_now_up); 
	hipFree(d_upx_next_dn); hipFree(d_upz_next_dn); hipFree(d_usx_next_dn); hipFree(d_usz_next_dn); hipFree(d_upx_now_dn);	hipFree(d_upz_now_dn); hipFree(d_usx_now_dn); hipFree(d_usz_now_dn); 
	hipFree(d_upx_next_lf); hipFree(d_upz_next_lf); hipFree(d_usx_next_lf); hipFree(d_usz_next_lf); hipFree(d_upx_now_lf); hipFree(d_upz_now_lf); hipFree(d_usx_now_lf); hipFree(d_usz_now_lf); 
	hipFree(d_upx_next_rt); hipFree(d_upz_next_rt); hipFree(d_usx_next_rt); hipFree(d_usz_next_rt); hipFree(d_upx_now_rt); hipFree(d_upz_now_rt); hipFree(d_usx_now_rt); hipFree(d_usz_now_rt); 
	hipFree(d_ux_s); hipFree(d_uz_s); hipFree(d_theta_s); hipFree(d_omega_s); hipFree(d_duzdx_s); hipFree(d_duzdz_s); hipFree(d_duxdz_s);hipFree(d_duxdx_s); 
	hipFree(d_upx_next_s); hipFree(d_upz_next_s); hipFree(d_usx_next_s); hipFree(d_usz_next_s); hipFree(d_upx_now_s); hipFree(d_upz_now_s); hipFree(d_usx_now_s); hipFree(d_usz_now_s); 
	hipFree(d_upx_past_s); hipFree(d_upz_past_s); hipFree(d_usx_past_s); hipFree(d_usz_past_s);	hipFree(d_ux_r); hipFree(d_uz_r); hipFree(d_theta_r); hipFree(d_omega_r); 
	hipFree(d_duxdz_r); hipFree(d_duzdz_r); hipFree(d_duxdx_r); hipFree(d_duzdx_r); hipFree(d_upx_next_r); hipFree(d_upz_next_r); hipFree(d_usx_next_r); hipFree(d_usz_next_r); 
	hipFree(d_upx_now_r); hipFree(d_upz_now_r); hipFree(d_usx_now_r); hipFree(d_usz_now_r); hipFree(d_upx_past_r); hipFree(d_upz_past_r); hipFree(d_usx_past_r); hipFree(d_usz_past_r); 
	hipFree(O_duz_xdz_r); hipFree(O_duz_zdx_r); hipFree(O_dux_zdx_r); hipFree(O_dux_xdz_r); hipFree(O_thetax_r); hipFree(O_thetaz_r); hipFree(O_omegaz_r); hipFree(O_omegax_r); 
	hipFree(F_xx_r); hipFree(F_zz_r); hipFree(F_xz_r); hipFree(F_zx_r); hipFree(d_Epx_S); hipFree(d_Epz_S); hipFree(d_Epx_R); hipFree(d_Epz_R); hipFree(d_Esx_R); hipFree(d_Esz_R);
	hipFree(d_fenzi_PP_pyt); hipFree(d_fenzi_PS_pyt); hipFree(d_fenmu_P_pyt); hipFree(d_fenzi_PP); hipFree(d_fenzi_PS); hipFree(d_fenmu_P); 
	hipFree(d_RR_upx_up); hipFree(d_RR_upx_dn); hipFree(d_RR_upx_lf); hipFree(d_RR_upx_rt); hipFree(d_RR_upz_up); hipFree(d_RR_upz_dn); hipFree(d_RR_upz_lf); hipFree(d_RR_upz_rt); 
	hipFree(d_RR_usx_up); hipFree(d_RR_usx_dn); hipFree(d_RR_usx_lf); hipFree(d_RR_usx_rt); hipFree(d_RR_usz_up); hipFree(d_RR_usz_dn); hipFree(d_RR_usz_lf); hipFree(d_RR_usz_rt); 
	hipFree(d_SS_upx_up); hipFree(d_SS_upx_dn); hipFree(d_SS_upx_lf); hipFree(d_SS_upx_rt);	hipFree(d_SS_upz_up); hipFree(d_SS_upz_dn); hipFree(d_SS_upz_lf); hipFree(d_SS_upz_rt); 
	hipFree(d_image_PP); hipFree(d_image_PS); hipFree(d_image_PP_lap); hipFree(d_image_PS_lap); hipFree(d_image_PP_pyt); hipFree(d_image_PS_pyt); 


	return 0;
}
